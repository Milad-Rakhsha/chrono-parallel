#include "hip/hip_runtime.h"
#include "ChCCollisionGPU.h"

#include "ChCCollisionGPU.cuh"
__constant__ uint last_active_bin_const;
__constant__ float3 bin_size_vec_const;
__constant__ uint number_of_models_const;

template<class T>
__device__ __host__ inline uint3 Hash(const T &A) {
	return U3(A.x * bin_size_vec_const.x, A.y * bin_size_vec_const.y, A.z * bin_size_vec_const.z);
}
template<class T>
__device__ __host__ inline uint Hash_Index(const T &A) {
	return ((A.x * 73856093) ^ (A.y * 19349663) ^ (A.z * 83492791));
}
__device__ bool TestAABBAABB(const AABB &A, const AABB &B, uint Bin) {
	return (Bin == Hash_Index(Hash(((A.min + A.max) * .5 + (B.min + B.max) * .5) * .5))) && (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
}
__device__ bool PointInAABB(const AABB &A, const float3 &P) {
	if (P.x > A.min.x && P.x < A.max.x && P.y > A.min.y && P.y < A.max.y && P.z > A.min.z && P.z < A.max.z) {
		return true;
	}
	return false;
}
__device__ __host__ bool AABB_Contact_Pt(const AABB& A, const AABB& B, uint &Bin) {
	float3 Pa, Pb;

	//return (F3(A.min+A.max)*.5+F3(B.min+B.max)*.5)*.5;

	bool minX = (A.min.x <= B.min.x && A.max.x >= B.max.x), minY = (A.min.y <= B.min.y && A.max.y >= B.max.y), minZ = (A.min.z <= B.min.z && A.max.z >= B.max.z);

	if (minX && minY && minZ) {
		Pa = (B.min);
		Pb = (B.max);
	} //B inside A

	else if (minY && minZ) {
		if ((A.max.x >= B.min.x && A.max.x <= B.max.x)) {
			Pa = F3(A.max.x, B.max.y, B.max.z);
			Pb = (B.min);
		} //right
		else if ((A.min.x >= B.min.x && A.min.x <= B.max.x)) {
			Pa = F3(A.min.x, B.min.y, B.min.z);
			Pb = (B.max);
		} //left
	} else if (minX && minZ) {
		if ((A.max.y >= B.min.y && A.max.y <= B.max.y)) {
			Pa = F3(B.max.x, A.max.y, B.max.z);
			Pb = (B.min);
		} //top
		else if ((A.min.y >= B.min.y && A.min.y <= B.max.y)) {
			Pa = F3(B.min.x, A.min.y, B.min.z);
			Pb = (B.max);
		} //bottom
	} else if (minY && minX) {
		if ((A.max.z >= B.min.z && A.max.z <= B.max.z)) {
			Pa = F3(B.max.x, B.max.y, A.max.z);
			Pb = (B.min);
		} //front
		else if ((A.min.z >= B.min.z && A.min.z <= B.max.z)) {
			Pa = F3(B.min.x, B.min.y, A.min.z);
			Pb = (B.max);
		} //back
	} else { //corners
		if (A.max.x >= B.min.x || A.max.x <= B.max.x) {
			Pa.x = A.max.x;
			Pb.x = B.min.x;
		} else if (A.min.x <= B.max.x || A.min.x >= B.min.x) {
			Pa.x = A.min.x;
			Pb.x = B.max.x;
		}
		if (A.max.y >= B.min.y || A.max.y <= B.max.y) {
			Pa.y = A.max.y;
			Pb.y = B.min.y;
		} else if (A.min.y <= B.max.y || A.min.y >= B.min.y) {
			Pa.y = A.min.y;
			Pb.y = B.max.y;
		}
		if (A.max.z >= B.min.z || A.max.z <= B.max.z) {
			Pa.z = A.max.z;
			Pb.z = B.min.z;
		} else if (A.min.z <= B.max.z || A.min.z >= B.min.z) {
			Pa.z = A.min.z;
			Pb.z = B.max.z;
		}
	}
	return (Bin == Hash_Index(Hash((Pa + Pb) * .5f))) && (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
}

__device__ int Contact_Type(const int &A, const int &B) {
	if (A == 0 && B == 0) {
		return 0;
	} //sphere-sphere
	if (A == 0 && B == 1) {
		return 1;
	} //sphere-triangle
	if (A == 1 && B == 0) {
		return 2;
	} //triangle-sphere
	if (A == 3 && B == 3) {
		return 3;
	} //ellipsoid-ellipsoid
	return 20;
}
__global__ void AABB_Bins_Count(float3* device_aabb_data, uint* Bins_Intersected) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= number_of_models_const) {
		return;
	}
	uint3 gmin = Hash(device_aabb_data[index]);
	uint3 gmax = Hash(device_aabb_data[index + number_of_models_const]);
	Bins_Intersected[index] = (gmax.x - gmin.x + 1) * (gmax.y - gmin.y + 1) * (gmax.z - gmin.z + 1);
}
__global__ void AABB_Bins(float3* device_aabb_data, uint* Bins_Intersected, uint * bin_number, uint * body_number) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= number_of_models_const) {
		return;
	}
	int count = 0, i, j, k;
	uint3 gmin = Hash(device_aabb_data[index]);
	uint3 gmax = Hash(device_aabb_data[index + number_of_models_const]);
	uint mInd = (index == 0) ? 0 : Bins_Intersected[index - 1];

	for (i = gmin.x; i <= gmax.x; i++) {
		for (j = gmin.y; j <= gmax.y; j++) {
			for (k = gmin.z; k <= gmax.z; k++) {
				bin_number[mInd + count] = Hash_Index(U3(i, j, k));
				body_number[mInd + count] = index;
				count++;
			}
		}
	}

}
__global__ void AABB_AABB_Count(float3* device_aabb_data, int2* device_fam_data, uint * bin_number, uint * body_number, uint * bin_start_index, uint* Num_ContactD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= last_active_bin_const) {
		return;
	}
	uint end = bin_start_index[index], count = 0, i = (!index) ? 0 : bin_start_index[index - 1], Bin = bin_number[index];
	AABB A, B;

	for (; i < end; i++) {
		A.min = device_aabb_data[body_number[i]];
		A.max = device_aabb_data[body_number[i] + number_of_models_const];
		int2 FA = device_fam_data[body_number[i]];
		for (int k = i + 1; k < end; k++) {
			B.min = device_aabb_data[body_number[k]];
			B.max = device_aabb_data[body_number[k] + number_of_models_const];
			int2 FB = device_fam_data[body_number[k]];
			if ((FA.x == FB.y || FB.x == FA.y) == false && AABB_Contact_Pt(A, B, Bin) == true) {
				count++;
			}
		}
	}
	Num_ContactD[index] = count;
}
__global__ void AABB_AABB(float3* device_aabb_data, int3* device_typ_data, int2* device_fam_data, uint * bin_number, uint * body_number, uint * bin_start_index, uint* Num_ContactD, long long* pair) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= last_active_bin_const) {
		return;
	}
	uint end = bin_start_index[index], count = 0, i = (!index) ? 0 : bin_start_index[index - 1], Bin = bin_number[index];
	uint offset = (!index) ? 0 : Num_ContactD[index - 1];
	AABB A, B;
	int3 A_type, B_type;
	for (; i < end; i++) {
		A.min = device_aabb_data[body_number[i]];
		A.max = device_aabb_data[body_number[i] + number_of_models_const];
		A_type = device_typ_data[body_number[i]];
		int2 FA = device_fam_data[body_number[i]];
		for (int k = i + 1; k < end; k++) {
			B.min = device_aabb_data[body_number[k]];
			B.max = device_aabb_data[body_number[k] + number_of_models_const];
			B_type = device_typ_data[body_number[k]];
			int2 FB = device_fam_data[body_number[k]];
			if ((FA.x == FB.y || FB.x == FA.y) == false && AABB_Contact_Pt(A, B, Bin) == true) {
				//int type=Contact_Type(A_aux.x, B_aux.x);
				pair[offset + count] = ((long long) A_type.y << 32 | (long long) B_type.y); //the two indicies of the objects that make up the contact
				count++;
			}
		}
	}
}

__device__ __host__ bool operator ==(const float3 &a, const float3 &b) {
	return ((a.x == b.x) && (a.y == b.y) && (a.z == b.z));
}
void ChCCollisionGPU::Broadphase(float3 &bin_size_vec, gpu_container & gpu_data) {
	uint number_of_models = gpu_data.number_of_models;
	uint last_active_bin = 0, number_of_bin_intersections = 0;
	uint number_of_contacts_possible = 0;
	thrust::device_vector<uint> generic_counter(number_of_models, 0);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(AABB_Bins_Count), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(AABB_Bins), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(AABB_AABB_Count), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(AABB_AABB), hipFuncCachePreferL1);
	COPY_TO_CONST_MEM(bin_size_vec);
	COPY_TO_CONST_MEM(number_of_models);

	AABB_Bins_Count CUDA_KERNEL_DIM(BLOCKS(number_of_models),THREADS)(CASTF3(gpu_data.device_aabb_data), CASTU1(generic_counter));

	Thrust_Inclusive_Scan_Sum(generic_counter, number_of_bin_intersections);

	thrust::device_vector<uint> bin_number(number_of_bin_intersections);
	thrust::device_vector<uint> body_number(number_of_bin_intersections);
	thrust::device_vector<uint> bin_start_index(number_of_bin_intersections);

	AABB_Bins CUDA_KERNEL_DIM(BLOCKS(number_of_models),THREADS)(CASTF3(gpu_data.device_aabb_data), CASTU1(generic_counter), CASTU1(bin_number), CASTU1(body_number));

//	if (gpu_data.OLD_bin_number.size() > 0 && bin_number.size() > 0) {
//		if (gpu_data.OLD_bin_number.size() == bin_number.size()) {
//			if (Thrust_Equal(bin_number,gpu_data.OLD_bin_number)) {
//				gpu_data.device_pair_data = gpu_data.OLD_device_pair_data;
//				return;
//			} else {
//				gpu_data.OLD_bin_number = bin_number;
//			}
//		} else {
//			gpu_data.OLD_bin_number = bin_number;
//		}
//	} else {
//		if (bin_start_index.size() > 0) {
//			gpu_data.OLD_bin_number = bin_number;
//		}
//	}

	Thrust_Sort_By_Key(bin_number, body_number);
	Thrust_Reduce_By_KeyA(last_active_bin, bin_number, bin_start_index);
	Thrust_Inclusive_Scan(bin_start_index);

	generic_counter.resize(last_active_bin);

	COPY_TO_CONST_MEM(last_active_bin);
	AABB_AABB_Count CUDA_KERNEL_DIM(BLOCKS(last_active_bin),THREADS)(CASTF3(gpu_data.device_aabb_data), CASTI2(gpu_data.device_fam_data), CASTU1(bin_number), CASTU1(body_number), CASTU1(bin_start_index), CASTU1(generic_counter));
	Thrust_Inclusive_Scan_Sum(generic_counter, number_of_contacts_possible);
	gpu_data.device_pair_data.resize(number_of_contacts_possible);
	AABB_AABB CUDA_KERNEL_DIM(BLOCKS(last_active_bin),THREADS)(CASTF3(gpu_data.device_aabb_data), CASTI3(gpu_data.device_typ_data), CASTI2(gpu_data.device_fam_data), CASTU1(bin_number), CASTU1(body_number), CASTU1(bin_start_index), CASTU1(generic_counter), CASTLL(gpu_data.device_pair_data));
	gpu_data.number_of_contacts_possible = number_of_contacts_possible;


}
