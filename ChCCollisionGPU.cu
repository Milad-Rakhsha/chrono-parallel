#include "hip/hip_runtime.h"
#include "ChCCollisionGPU.h"
#include "ChCCollisionGPU.cuh"
__constant__ float3 global_origin_const;
__constant__ uint number_of_models_const;
__constant__ float3 bin_size_vec_const;
__constant__ float collision_envelope_const;
using namespace chrono::collision;

__global__ void Compute_AABBs(float3* pos, float4* rot, float3* obA, float3* obB, float3* obC, float4* obR, int3* typ, float3* aabb) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= number_of_models_const) {
		return;
	}
	int3 type = typ[index];

	float3 A = obA[index];
	float3 B = obB[index];
	float3 C = obC[index];

	float3 position = pos[type.z];
	float4 rotation = (mult(rot[type.z], obR[index]));

	float3 temp_min;
	float3 temp_max;

	if (type.x == 0) {
		ComputeAABBSphere(B.x, A + position, temp_min, temp_max);
	} else if (type.x == 5) {
		A = quatRotate(A + position, rotation);
		B = quatRotate(B + position, rotation);
		C = quatRotate(C + position, rotation);

		ComputeAABBTriangle(A, B, C, temp_min, temp_max);
	} else if (type.x == 1 || type.x == 2 || type.x == 3) {
		ComputeAABBBox(B, A + position, rotation, temp_min, temp_max);
	} else {
		return;
	}
	aabb[index] = temp_min;
	aabb[index + number_of_models_const] = temp_max;
}

__global__ void Offset_AABBs(float3* aabb) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= number_of_models_const) {
		return;
	}
	float3 temp_min = aabb[index];
	float3 temp_max = aabb[index + number_of_models_const];
	aabb[index] = temp_min - F3(collision_envelope_const) + global_origin_const;
	aabb[index + number_of_models_const] = temp_max + F3(collision_envelope_const) + global_origin_const;
}

void ChCCollisionGPU::ComputeAABB_HOST(ChGPUDataManager * data_container) {
	uint number_of_models = data_container->number_of_models;
	data_container->host_aabb_data.resize(number_of_models * 2);

	for (int index = 0; index < number_of_models; index++) {
		int3 type = data_container->host_typ_data[index];

		float3 A = data_container->host_ObA_data[index];
		float3 B = data_container->host_ObB_data[index];
		float3 C = data_container->host_ObC_data[index];

		float3 position = data_container->host_pos_data[type.z];
		float4 rotation = normalize(data_container->host_rot_data[type.z] + data_container->host_ObR_data[index]);

		float3 temp_min;
		float3 temp_max;

		if (type.x == 0) {
			ComputeAABBSphere(B.x, A + position, temp_min, temp_max);
		} else if (type.x == 5) {
			A = quatRotate(A + position, rotation);
			B = quatRotate(B + position, rotation);
			C = quatRotate(C + position, rotation);

			ComputeAABBTriangle(A, B, C, temp_min, temp_max);
		} else if (type.x == 1 || type.x == 2 || type.x == 3) {
			ComputeAABBBox(B, A + position, rotation, temp_min, temp_max);
		} else {
			return;
		}
		data_container->host_aabb_data[index] = temp_min;
		data_container->host_aabb_data[index + number_of_models] = temp_max;
	}
}
void ChCCollisionGPU::ComputeBounds_HOST(ChGPUDataManager * data_container) {
	uint number_of_models = data_container->number_of_models;
	bbox init = bbox(data_container->host_aabb_data[0], data_container->host_aabb_data[0]);
	bbox_transformation unary_op;
	bbox_reduction binary_op;
	bbox result = thrust::transform_reduce(data_container->host_aabb_data.begin(), data_container->host_aabb_data.end(), unary_op, init, binary_op);
	data_container->min_bounding_point = result.first;
	data_container->max_bounding_point = result.second;
}
void ChCCollisionGPU::UpdateAABB_HOST(float3 & bin_size_vec, float & max_dimension, float & collision_envelope, ChGPUDataManager * data_container) {
	uint number_of_models = data_container->number_of_models;
	float3 global_origin = fabs(data_container->min_bounding_point); //Determine Global Origin
	max_dimension = max3(global_origin + fabs(data_container->max_bounding_point)); //Determine Max point in space
	bin_size_vec = (global_origin + fabs(data_container->max_bounding_point)) / (powf(number_of_models * 2, 1 / 3.0));
	bin_size_vec = 1.0 / bin_size_vec;

	for (int index = 0; index < number_of_models; index++) {
		float3 temp_min = data_container->host_aabb_data[index];
		float3 temp_max = data_container->host_aabb_data[index + number_of_models];
		data_container->host_aabb_data[index] = temp_min - F3(collision_envelope) + global_origin;
		data_container->host_aabb_data[index + number_of_models] = temp_max + F3(collision_envelope) + global_origin;
	}
}

void ChCCollisionGPU::ComputeAABB(gpu_container & gpu_data) {
	uint number_of_models = gpu_data.number_of_models;
	COPY_TO_CONST_MEM(number_of_models);
	gpu_data.device_aabb_data.resize(number_of_models * 2);
	Compute_AABBs CUDA_KERNEL_DIM(BLOCKS(number_of_models),THREADS)(CASTF3(gpu_data.device_pos_data), CASTF4(gpu_data.device_rot_data), CASTF3(gpu_data.device_ObA_data),
			CASTF3(gpu_data.device_ObB_data), CASTF3(gpu_data.device_ObC_data), CASTF4(gpu_data.device_ObR_data), CASTI3(gpu_data.device_typ_data), CASTF3(gpu_data.device_aabb_data));

	bbox init = bbox(gpu_data.device_aabb_data[0], gpu_data.device_aabb_data[0]);
	bbox_transformation unary_op;
	bbox_reduction binary_op;
	bbox result = thrust::transform_reduce(gpu_data.device_aabb_data.begin(), gpu_data.device_aabb_data.end(), unary_op, init, binary_op);
	gpu_data.min_bounding_point = result.first;
	gpu_data.max_bounding_point = result.second;

	float3 global_origin=fabs(gpu_data.min_bounding_point);
	float collision_envelope=0;
	COPY_TO_CONST_MEM(collision_envelope);
	COPY_TO_CONST_MEM(global_origin);
	Offset_AABBs CUDA_KERNEL_DIM(BLOCKS(number_of_models),THREADS)(CASTF3(gpu_data.device_aabb_data));
}
void ChCCollisionGPU::ComputeBounds(gpu_container & gpu_data) {
	uint number_of_models = gpu_data.number_of_models;
	COPY_TO_CONST_MEM(number_of_models);
	bbox init = bbox(gpu_data.device_aabb_data[0], gpu_data.device_aabb_data[0]);
	bbox_transformation unary_op;
	bbox_reduction binary_op;
	bbox result = thrust::transform_reduce(gpu_data.device_aabb_data.begin(), gpu_data.device_aabb_data.end(), unary_op, init, binary_op);
	gpu_data.min_bounding_point = result.first;
	gpu_data.max_bounding_point = result.second;
	//DBG("ComputeBounds");
}

void ChCCollisionGPU::UpdateAABB(float & collision_envelope, gpu_container & gpu_data, float3 global_origin) {
	uint number_of_models = gpu_data.number_of_models;
	COPY_TO_CONST_MEM(number_of_models);
	COPY_TO_CONST_MEM(collision_envelope);
	COPY_TO_CONST_MEM(global_origin);
	Offset_AABBs CUDA_KERNEL_DIM(BLOCKS(number_of_models),THREADS)(CASTF3(gpu_data.device_aabb_data));

	//	data_container->gpu_data[i].device_bin_data.resize(number_of_models * 2);
	//FindGrid<<<BLOCKS(number_of_models),THREADS>>>(CASTF3(data_container->gpu_data[i].device_aabb_data), CASTU3(data_container->gpu_data[i].device_bin_data));
}
