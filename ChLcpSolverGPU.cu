#include "hip/hip_runtime.h"
#include "ChLcpSolverGPU.h"
#include "ChThrustLinearAlgebra.cuh"
#include "solver/ChSolverGPU.cuh"
#include "ChJacobianGPU.h"
#include "ChIntegratorGPU.h"
#include "ChComputeRHSGPU.h"

using namespace chrono;
__constant__ uint number_of_objects_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ uint number_of_updates_const;
__constant__ real step_size_const;

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__host__ __device__ void function_addForces(uint& index, bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	if (active[index] != 0) {
		// v += m_inv * h * f
		vel[index] += forces[index] * mass[index];
		// w += J_inv * h * c
		omega[index] += torques[index] * inertia[index];
	}
}
__global__ void device_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
}

void ChLcpSolverGPU::host_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
#pragma omp parallel for

	for (uint index = 0; index < number_of_objects; index++) {
		function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
	}
}

__host__ __device__ void function_ComputeGyro(uint& index, real3* omega, real3* inertia, real3* gyro, real3* torque) {
	real3 body_inertia = inertia[index];
	body_inertia = R3(1.0 / body_inertia.x, 1.0 / body_inertia.y, 1.0 / body_inertia.z);
	real3 body_omega = omega[index];
	real3 gyr = cross(body_omega, body_inertia * body_omega);
	gyro[index] = gyr;
}

__global__ void device_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_ComputeGyro(index, omega, inertia, gyro, torque);
}

void ChLcpSolverGPU::host_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
#pragma omp parallel for

	for (uint index = 0; index < number_of_objects; index++) {
		function_ComputeGyro(index, omega, inertia, gyro, torque);
	}
}

void ChLcpSolverGPU::Preprocess() {
	data_container->number_of_updates = 0;
	data_container->host_data.gyr_data.resize(number_of_objects);

#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(number_of_objects);

	device_ComputeGyro CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
			CASTR3(data_container->device_data.device_omg_data),
			CASTR3(data_container->device_data.device_inr_data),
			CASTR3(data_container->device_data.device_gyr_data),
			CASTR3(data_container->device_data.device_trq_data));
	device_addForces CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
			CASTB1(data_container->device_data.device_active_data),
			CASTR1(data_container->device_data.device_mass_data),
			CASTR3(data_container->device_data.device_inr_data),
			CASTR3(data_container->device_data.device_frc_data),
			CASTR3(data_container->device_data.device_trq_data),
			CASTR3(data_container->device_data.device_vel_data),
			CASTR3(data_container->device_data.device_omg_data));
#else
	host_ComputeGyro(
			data_container->host_data.omg_data.data(),
			data_container->host_data.inr_data.data(),
			data_container->host_data.gyr_data.data(),
			data_container->host_data.trq_data.data());

	host_addForces(
			data_container->host_data.active_data.data(),
			data_container->host_data.mass_data.data(),
			data_container->host_data.inr_data.data(),
			data_container->host_data.frc_data.data(),
			data_container->host_data.trq_data.data(),
			data_container->host_data.vel_data.data(),
			data_container->host_data.omg_data.data());
#endif
}

void ChLcpSolverGPU::RunTimeStep(real step) {
	step_size = step;
	data_container->step_size = step;

	number_of_constraints = data_container->number_of_contacts * 3 + data_container->number_of_bilaterals;
	number_of_contacts = data_container->number_of_contacts;
	number_of_bilaterals = 0; ///data_container->number_of_bilaterals;
	number_of_objects = data_container->number_of_objects;
	//cout << number_of_constraints << " " << number_of_contacts << " " << number_of_bilaterals << " " << number_of_objects << endl;
#ifdef PRINT_DEBUG_GPU
	cout << "Preprocess: " << endl;
#endif
	Preprocess();
#ifdef PRINT_DEBUG_GPU
	cout << "Jacobians: " << endl;
#endif
	ChJacobianGPU jacobian_compute;
	jacobian_compute.ComputeJacobians(data_container);

	ChComputeRHSGPU rhs_compute;
	rhs_compute.ComputeRHS(data_container);

#ifdef PRINT_DEBUG_GPU
	cout << "Solve: " << endl;
#endif

		ChSolverGPU solver;
		solver.SetMaxIterations(max_iteration);
		solver.SetTolerance(tolerance);
		solver.SetComplianceParameters(alpha, compliance, complianceT);
		solver.SetContactRecoverySpeed(contact_recovery_speed);
		solver.lcp_omega_bilateral = lcp_omega_bilateral;
		solver.lcp_omega_contact = lcp_omega_contact;
		solver.do_stab = do_stab;
		solver.Solve(solver_type, step, data_container);

		tot_iterations = solver.GetIteration();
		residual = solver.GetResidual();
		rhs = data_container->host_data.rhs_data;
		lambda = data_container->host_data.gam_data;

		for (int i = 0; i < solver.iter_hist.size(); i++) {
			AtIterationEnd(solver.maxd_hist[i], solver.maxdeltalambda_hist[i], solver.iter_hist[i]);
		}
	//stabilization  code
	//jacobian_compute.ComputeJacobians(data_container);
	//rhs_compute.ComputeRHS(data_container);

	//solver.VelocityStabilization(data_container);

#ifdef PRINT_DEBUG_GPU
	cout << "Solve Done: "<<residual << endl;
#endif
	//ChIntegratorGPU integrator;
	//integrator.IntegrateSemiImplicit(step, data_container->gpu_data);
}

