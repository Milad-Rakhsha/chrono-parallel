#include "hip/hip_runtime.h"
#include "ChLcpSolverGPU.h"

using namespace chrono;

__constant__ real lcp_omega_bilateral_const;
__constant__ real lcp_omega_contact_const;
__constant__ real lcp_contact_factor_const;
__constant__ uint number_of_objects_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ uint number_of_updates_const;
__constant__ real step_size_const;
__constant__ real compliance_const;
__constant__ real complianceT_const;
__constant__ real alpha_const; // [R]=alpha*[K]

//__constant__ real force_factor_const;
//__constant__ real negated_recovery_speed_const;
////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void inline Compute_Jacobian(
    const real4 &quaternion_rotation,
    const real3 &normal,
    const real3 &tangent_u,
    const real3 &tangent_w,
    const real3 &point,
    real3 &T3,
    real3 &T4,
    real3 &T5
) {
    real t1 = quaternion_rotation.y * quaternion_rotation.z;
    real t2 = quaternion_rotation.x * quaternion_rotation.w;
    real t3 = 2 * t1 - 2 * t2;
    real t4 = quaternion_rotation.y * quaternion_rotation.w;
    real t5 = quaternion_rotation.x * quaternion_rotation.z;
    real t6 = 2 * t4 + 2 * t5;
    real t7 = quaternion_rotation.z * quaternion_rotation.w;
    real t8 = quaternion_rotation.x * quaternion_rotation.y;
    real t9 = 2 * t7 - 2 * t8;
    real t10 = quaternion_rotation.x * quaternion_rotation.x;
    real t11 = quaternion_rotation.y * quaternion_rotation.y;
    real t12 = quaternion_rotation.w * quaternion_rotation.w;
    real t13 = quaternion_rotation.z * quaternion_rotation.z;
    real t14 = t10 - t11 - t13 + t12;
    real t15 = t6 * point.x + t9 * point.y + t14 * point.z;
    real t16 = t10 - t11 + t13 - t12;
    t7 = 2 * t7 + 2 * t8;
    t8 = -t3 * point.x - t16 * point.y - t7 * point.z;
    real t17 = t3 * t15 + t6 * t8;
    real t18 = t16 * t15 + t9 * t8;
    real t19 = t7 * t15 + t14 * t8;
    t10 = t10 + t11 - t13 - t12;
    t11 = -t15;
    t1 = 2 * t1 + 2 * t2;
    t2 = 2 * t4 - 2 * t5;
    t4 = t10 * point.x + t1 * point.y + t2 * point.z;
    t5 = t10 * t11 + t6 * t4;
    t6 = t1 * t11 + t9 * t4;
    t9 = t2 * t11 + t14 * t4;
    t8 = -t8;
    t4 = -t4;
    t3 = t10 * t8 + t3 * t4;
    t1 = t1 * t8 + t16 * t4;
    t2 = t2 * t8 + t7 * t4;
    T3.x = normal.x * t17 + normal.y * t18 + normal.z * t19;
    T3.y = normal.x * t5 + normal.y * t6 + normal.z * t9;
    T3.z = normal.x * t3 + normal.y * t1 + normal.z * t2;
    T4.x = tangent_u.x * t17 + tangent_u.y * t18 + tangent_u.z * t19;
    T4.y = tangent_u.x * t5 + tangent_u.y * t6 + tangent_u.z * t9;
    T4.z = tangent_u.x * t3 + tangent_u.y * t1 + tangent_u.z * t2;
    T5.x = tangent_w.x * t17 + tangent_w.y * t18 + tangent_w.z * t19;
    T5.y = tangent_w.x * t5 + tangent_w.y * t6 + tangent_w.z * t9;
    T5.z = tangent_w.x * t3 + tangent_w.y * t1 + tangent_w.z * t2;
}

//  Kernel for a single iteration of the LCP over all contacts
//  Version 2.0 - Tasora
//  Version 2.2- Hammad (optimized, cleaned etc)
__global__ void LCP_Iteration_Contacts(
    real3 *norm,
    real3 *ptA,
    real3 *ptB,
    real *contactDepth,
    int2 *ids,
    real3 *G,
    real *dG,
    real3 *aux,
    real3 *inertia,
    real4 *rot,
    real3 *vel,
    real3 *omega,
    real3 *pos,
    real3 *updateV,
    real3 *updateO,
    uint *offset) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
    real3 gamma, T3, T4, T5, T6, T7, T8;
    int2 temp_id = ids[index];
    real depth = -fabs(contactDepth[index]);
    int B1_i = temp_id.x;
    int B2_i = temp_id.y;
    real3 N = norm[index]; //assume: normalized, and if depth=0 norm=(1,0,0)
    real3 W = fabs(N); //Gramm Schmidt; work with the global axis that is "most perpendicular" to the contact normal vector;effectively this means looking for the smallest component (in abs) and using the corresponding direction to carry out cross product.
    real3 U = real3(0, N.z, -N.y); //it turns out that X axis is closest to being perpendicular to contact vector;

    if (W.x > W.y) {
        U = real3(-N.z, 0, N.x);
    } //it turns out that Y axis is closest to being perpendicular to contact vector;

    if (W.y > W.z) {
        U = real3(N.y, -N.x, 0);
    } //it turns out that Z axis is closest to being perpendicular to contact vector;

    U = normalize(U); //normalize the local contact Y,Z axis
    W = cross(N, U); //carry out the last cross product to find out the contact local Z axis : multiply the contact normal by the local Y component
    real3 sbar = ptA[index] - pos[B1_i]; //Contact Point on A - Position of A
    real4 E1 = rot[B1_i]; //bring in the Euler parameters associated with body 1;
    Compute_Jacobian(E1, N, U, W, sbar, T3, T4, T5); //A_i,p'*A_A*(sbar~_i,A)
    sbar = ptB[index] - pos[B2_i]; //Contact Point on B - Position of B
    real4 E2 = rot[B2_i]; //bring in the Euler parameters associated with body 2;
    Compute_Jacobian(E2, N, U, W, sbar, T6, T7, T8); //A_i,p'*A_B*(sbar~_i,B)
    T6 = -T6;
    T7 = -T7;
    T8 = -T8;
//  real normV = dot(N, ((V2 - V1)));
//  normV = (normV > 0) ? 0 : normV;
    real cfm = 0, cfmT = 0;
//  if (compliance_const) {
//      real h = step_size_const;
//      real inv_hpa = 1.0 / (h + alpha_const); // 1/(h+a)
//      real inv_hhpa = 1.0 / (h * (h + alpha_const)); // 1/(h*(h+a))
//      bi = inv_hpa * depth;
//      cfm = inv_hhpa * compliance_const;
//      cfmT = inv_hhpa * complianceT_const;
//  } else {
    real bi = fmaxf((depth / (step_size_const)), -lcp_contact_factor_const);
//  }
    //c_i = [Cq_i]*q + b_i + cfm_i*l_i
    real3 W1 = omega[B1_i];
    real3 W2 = omega[B2_i];
    real3 V1 = vel[B1_i];
    real3 V2 = vel[B2_i];
    gamma.x = dot(T3, W1) - dot(N, V1) + dot(T6, W2) + dot(N, V2) + bi /*+ cfm * gamma_old.x*/; //+bi
    gamma.y = dot(T4, W1) - dot(U, V1) + dot(T7, W2) + dot(U, V2) /*+ cfmT * gamma_old.y*/;
    gamma.z = dot(T5, W1) - dot(W, V1) + dot(T8, W2) + dot(W, V2) /*+ cfmT * gamma_old.z*/;
    real3 In1 = inertia[B1_i]; // bring in the inertia attributes; to be used to compute \eta
    real3 In2 = inertia[B2_i]; // bring in the inertia attributes; to be used to compute \eta
    real3 aux1 = aux[B1_i];
    real3 aux2 = aux[B2_i];
    real eta = dot(T3 * T3, In1) + dot(T4 * T4, In1) + dot(T5 * T5, In1); // update expression of eta
    eta += dot(T6 * T6, In2) + dot(T7 * T7, In2) + dot(T8 * T8, In2);
    eta += (dot(N, N) + dot(U, U) + dot(W, W)) * (aux1.z + aux2.z); // multiply by inverse of mass matrix of B1 and B2, add contribution from mass and matrix A_c.
    eta = lcp_omega_contact_const / eta; // final value of eta
    real3 gamma_old = G[index];
    gamma = eta * gamma; // perform gamma *= omega*eta
    gamma = gamma_old - gamma; // perform gamma = gamma_old - gamma ;  in place.
    /// ---- perform projection of 'a8' onto friction cone  --------
    real f_tang = sqrtf(gamma.y * gamma.y + gamma.z * gamma.z);
    real mu = (aux1.y + aux2.y) * .5f;

    if (f_tang > (mu * gamma.x)) { // inside upper cone? keep untouched!
        if ((f_tang) < -(1.0 / mu) * gamma.x || (fabsf(gamma.x) < 0)) { // inside lower cone? reset  normal,u,v to zero!
            gamma = R3(0);
        } else { // remaining case: project orthogonally to generator segment of upper cone
            gamma.x = (f_tang * mu + gamma.x) / (mu * mu + 1);
            real tproj_div_t = (gamma.x * mu) / f_tang; //  reg = tproj_div_t
            gamma.y *= tproj_div_t;
            gamma.z *= tproj_div_t;
        }
    } else if (mu == 0) {
        gamma.y = gamma.z = 0;
    }

    G[index] = gamma; // store gamma_new
    gamma -= gamma_old; // compute delta_gamma = gamma_new - gamma_old   = delta_gamma.
    dG[index] = length(gamma);
    real3 vB = N * gamma.x + U * gamma.y + W * gamma.z;
    int offset1 = offset[index];
    int offset2 = offset[index + number_of_contacts_const];
    updateV[offset1] = real3(-vB * aux1.z); // compute and store dv1
    updateO[offset1] = real3((T3 * gamma.x + T4 * gamma.y + T5 * gamma.z) * In1); // compute dw1 =  Inert.1' * J1w^ * deltagamma  and store  dw1
    updateV[offset2] = real3(vB * aux2.z); // compute and store dv2
    updateO[offset2] = real3((T6 * gamma.x + T7 * gamma.y + T8 * gamma.z) * In2); // compute dw2 =  Inert.2' * J2w^ * deltagamma  and store  dw2
}
///////////////////////////////////////////////////////////////////////////////////
// Kernel for a single iteration of the LCP over all scalar bilateral contacts
// (a bit similar to the ChKernelLCPiteration above, but without projection etc.)
// Version 2.0 - Tasora
//

__global__ void LCP_Iteration_Bilaterals(
    real4 *bilaterals,
    real3 *aux,
    real3 *inertia,
    real4 *rot,
    real3 *vel,
    real3 *omega,
    real3 *pos,
    real3 *updateV,
    real3 *updateO,
    uint *offset,
    real *dG) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_bilaterals_const);
    real4 vA;
    real3 vB;
    real gamma_new = 0, gamma_old = 0;
    int B1_index = 0, B2_index = 0;
    B1_index = bilaterals[index].w;
    B2_index = bilaterals[index + number_of_bilaterals_const].w;
    real3 aux1 = aux[B1_index];
    real3 aux2 = aux[B2_index];
    // ---- perform   gamma_new = ([J1 J2] {v1 | v2}^ + b)
    {
        vA = bilaterals[index]; // line 0
        vB = vel[B1_index]; // v1
        gamma_new += dot3(vA, vB);
        vA = bilaterals[index + 2 * number_of_bilaterals_const]; // line 2
        vB = omega[B1_index]; // w1
        gamma_new += dot3(vA, vB);
    }
    {
        vA = bilaterals[index + number_of_bilaterals_const]; // line 1
        vB = vel[B2_index]; // v2
        gamma_new += dot3(vA, vB);
        vA = bilaterals[index + 3 * number_of_bilaterals_const]; // line 3
        vB = omega[B2_index]; // w2
        gamma_new += dot3(vA, vB);
    }
    vA = bilaterals[index + 4 * number_of_bilaterals_const]; // line 4   (eta, b, gamma, 0)
    gamma_new += vA.y; // add known term     + b
    gamma_old = vA.z; // old gamma
    /// ---- perform gamma_new *= omega/g_i
    gamma_new *= lcp_omega_bilateral_const; // lcp_omega_const is in constant memory
    gamma_new *= vA.x; // eta = 1/g_i;
    /// ---- perform gamma_new = gamma_old - gamma_new ; in place.
    gamma_new = gamma_old - gamma_new;

    /// ---- perform projection of 'a' (only if simple unilateral behavior C>0 is requested)
    if (vA.w && gamma_new < 0.) {
        gamma_new = 0.;
    }

    // ----- store gamma_new
    vA.z = gamma_new;
    bilaterals[index + 4 * number_of_bilaterals_const] = vA;
    /// ---- compute delta in multipliers: gamma_new = gamma_new - gamma_old   = delta_gamma    , in place.
    gamma_new -= gamma_old;
    dG[number_of_contacts_const + index] = (gamma_new);
    /// ---- compute dv1 =  invInert.18 * J1^ * deltagamma
    vB = inertia[B1_index]; // iJ iJ iJ im
    vA = (bilaterals[index]) * aux1.z * gamma_new; // line 0: J1(x)
    int offset1 = offset[2 * number_of_contacts_const + index];
    int offset2 = offset[2 * number_of_contacts_const + index + number_of_bilaterals_const];
    updateV[offset1] = make_real3(vA); //  ---> store  v1 vel. in reduction buffer
    updateO[offset1] = make_real3((bilaterals[index + 2 * number_of_bilaterals_const]) * make_real4(vB) * gamma_new); // line 2:  J1(w)// ---> store  w1 vel. in reduction buffer
    vB = inertia[B2_index]; // iJ iJ iJ im
    vA = (bilaterals[index + number_of_bilaterals_const]) * aux2.z * gamma_new; // line 1: J2(x)
    updateV[offset2] = make_real3(vA); //  ---> store  v2 vel. in reduction buffer
    updateO[offset2] = make_real3((bilaterals[index + 3 * number_of_bilaterals_const]) * make_real4(vB) * gamma_new); // line 3:  J2(w)// ---> store  w2 vel. in reduction buffer
}

__device__ __host__ inline real4 computeRot_dt(real3 &omega, real4 &rot) {
    return mult(R4(0, omega.x, omega.y, omega.z), rot) * .5;
}
__device__ __host__ real3 RelPoint_AbsSpeed(real3 &vel, real3 &omega, real4 &rot, real3 &point) {
    real4 q = mult(computeRot_dt(omega, rot), mult(R4(0, point.x, point.y, point.z), inv(rot)));
    return vel + ((R3(q.y, q.z, q.w)) * 2);
}

__global__ void DEM_Contacts(
    real3 *norm,
    real3 *ptA,
    real3 *ptB,
    real *contactDepth,
    int2 *ids,
    real3 *aux,
    real3 *inertia,
    real4 *rot,
    real3 *vel,
    real3 *omega,
    real3 *pos,
    real3 *updateV,
    real3 *updateO,
    uint *offset) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
    real mkn = 3924, mgn = 420, mgt = 420, mkt = 2.0f / 7.0f * 3924;
    //long long id=ids[index];
    int2 temp_id = ids[index];
    int B1_i = int(temp_id.x);
    int B2_i = int(temp_id.y);
    real3 vN = norm[index]; //normal
    real3 pA = ptA[index];
    real3 pB = ptB[index];
    real3 B1 = vel[B1_i]; //vel B1
    real3 B2 = vel[B2_i]; //vel B2
    real3 oA = omega[B1_i];
    real3 oB = omega[B2_i];
    real4 rotA = rot[B1_i];
    real4 rotB = rot[B2_i];
    real3 aux1 = aux[B1_i];
    real3 aux2 = aux[B2_i];
    real3 posA = pos[B1_i];
    real3 posB = pos[B2_i];
    real3 f_n = mkn * -fabs(contactDepth[index]) * vN;
    real3 local_pA = quatRotate(pA - posA, inv(rotA));
    real3 local_pB = quatRotate(pB - posB, inv(rotB));
    real3 v_BA = (RelPoint_AbsSpeed(B2, oB, rotB, local_pB)) - (RelPoint_AbsSpeed(B1, oA, rotA, local_pA));
    real3 v_n = normalize(dot(v_BA, vN) * vN);
    real m_eff = (1.0 / aux1.z) * (1.0 / aux2.z) / (1.0 / aux1.z + 1.0 / aux2.z);
    f_n += mgn * m_eff * v_n;
    real mu = (aux1.y + aux2.y) * .5;
    real3 v_t = v_BA - v_n;
    real3 f_t = (mgt * m_eff * v_t) + (mkt * (v_t * step_size_const));

    if (length(f_t) > mu * length(f_n)) {
        f_t = f_t * mu * length(f_n) / length(f_t);
    }

    real3 f_r = f_n + f_t;
    int offset1 = offset[index];
    int offset2 = offset[index + number_of_contacts_const];
    real3 force1_loc = quatRotate(f_r, inv(rotA));
    real3 force2_loc = quatRotate(f_r, inv(rotB));
    real3 trq1 = cross(local_pA, force1_loc);
    real3 trq2 = cross(local_pB, -force2_loc);
    f_r = f_r * step_size_const;
    updateV[offset1] = (f_r) * aux1.z;
    updateV[offset2] = (f_r) * -aux2.z;
    updateO[offset1] = (trq1 * step_size_const) * (inertia[B1_i]);
    updateO[offset2] = (trq2 * step_size_const) * (inertia[B2_i]);
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__global__ void ChKernelLCPaddForces(real3 *aux, real3 *inertia, real3 *forces, real3 *torques, real3 *vel, real3 *omega) {
    // Compute the index values used to access data inside the large
    // array using pointer arithmetic.
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
    real3 temp_aux = aux[index];

    if (temp_aux.x != 0) {
        real3 mF, minvMasses = inertia[index];
        // v += m_inv * h * f
        mF = forces[index]; // vector with f (force)
        //mF *= 1;//step_size_const;
        mF = mF * temp_aux.z;
        vel[index] += mF;
        // w += J_inv * h * c
        mF = torques[index]; // vector with f (torque)
        //mF *= 1;//step_size_const;
        mF.x *= minvMasses.x;
        mF.y *= minvMasses.y;
        mF.z *= minvMasses.z;
        omega[index] += mF;
    }
}
////////////////////////////////////////////////////////////////////////////////////////////////////
// Updates the speeds in the body buffer with values accumulated in the
// reduction buffer:   V_new = V_old + delta_speeds

__global__ void LCP_Reduce_Speeds(real3 *aux, real3 *vel, real3 *omega, real3 *updateV, real3 *updateO, uint *d_body_num, uint *counter, real3 *fap) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_updates_const);
    int start = (index == 0) ? 0 : counter[index - 1], end = counter[index];
    int id = d_body_num[end - 1], j;
    real3 auxd = aux[id];

    if (auxd.x == 0) {
        return;
    }

    real3 mUpdateV = real3(0);
    real3 mUpdateO = real3(0);

    for (j = 0; j < end - start; j++) {
        mUpdateV = mUpdateV + updateV[j + start];
        mUpdateO = mUpdateO + updateO[j + start];
    }

    fap[id] += (mUpdateV / auxd.z) / step_size_const;
    vel[id] += (mUpdateV);
    omega[id] += (mUpdateO);
}
//  Kernel for performing the time step integration (with 1st o;rder Eulero)
//  on the body data stream.
//
//  number of registers: 12 (suggested 320 threads/block)

__global__ void LCP_Integrate_Timestep(real3 *aux, real3 *acc, real4 *rot, real3 *vel, real3 *omega, real3 *pos, real3 *lim) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
    real3 velocity = vel[index];
    real3 aux1 = aux[index];

    if (aux1.x == 0) {
        return;
    }

    // Do 1st order integration of quaternion position as q[t+dt] = qw_abs^(dt) * q[dt] = q[dt] * qw_local^(dt)
    // where qw^(dt) is the quaternion { cos(0.5|w|), wx/|w| sin(0.5|w|), wy/|w| sin(0.5|w|), wz/|w| sin(0.5|w|)}^dt
    // that is argument of sine and cosines are multiplied by dt.
    real3 omg = omega[index];
    real3 limits = lim[index];
    real wlen = length(omg);

    if (limits.x == 1) {
        real w = 2.0 * wlen;

        if (w > limits.z) {
            omg = omg * limits.z / w;
            wlen = sqrtf(dot3(omg, omg));
        }

        real v = length(velocity);

        if (v > limits.y) {
            velocity = velocity * limits.y / v;
        }

        vel[index] = velocity;
        omega[index] = omg;
    }

    pos[index] = pos[index] + velocity * step_size_const; // Do 1st order integration of linear speeds
    real4 Rw = (fabs(wlen) > 10e-10) ? Q_from_AngAxis(step_size_const * wlen, omg / wlen) : R4(1., 0, 0, 0); // to avoid singularity for near zero angular speed
    real4 mq = mult(rot[index], Rw);
    mq =mq* rsqrtf(dot(mq, mq));
    rot[index] = mq;
    acc[index] = (velocity - acc[index]) / step_size_const;
}
__global__ void LCP_ComputeGyro(real3 *omega, real3 *inertia, real3 *gyro, real3 *torque) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
    real3 body_inertia = inertia[index];
    body_inertia = R3(1 / body_inertia.x, 1 / body_inertia.y, 1 / body_inertia.z);
    real3 body_omega = omega[index];
    real3 gyr = cross(body_omega, body_inertia * body_omega);
    gyro[index] = gyr;
}

__global__ void ChKernelOffsets(int2 *ids, real4 *bilaterals, uint *Body) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < number_of_contacts_const) {
        int2 temp_id = ids[index];
        Body[index] = temp_id.x;
        Body[index + number_of_contacts_const] = temp_id.y;
    }

    if (index < number_of_bilaterals_const) {
        Body[2 * number_of_contacts_const + index] = bilaterals[index].w;
        Body[2 * number_of_contacts_const + index + number_of_bilaterals_const] = bilaterals[index + number_of_bilaterals_const].w;
    }
}

void ChLcpSolverGPU::WarmContact(const int &index) {
}

void ChLcpSolverGPU::Preprocess(gpu_container &gpu_data) {
    gpu_data.number_of_updates = 0;
    gpu_data.device_gyr_data.resize(number_of_objects);
    thrust::device_vector<uint> body_num;
    thrust::device_vector<uint> update_number;
    LCP_ComputeGyro CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
        CASTR3(gpu_data.device_omg_data),
        CASTR3(gpu_data.device_inr_data),
        CASTR3(gpu_data.device_gyr_data),
        CASTR3(gpu_data.device_trq_data));
    ChKernelLCPaddForces CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
        CASTR3(gpu_data.device_aux_data),
        CASTR3(gpu_data.device_inr_data),
        CASTR3(gpu_data.device_frc_data),
        CASTR3(gpu_data.device_trq_data),
        CASTR3(gpu_data.device_vel_data),
        CASTR3(gpu_data.device_omg_data));
    gpu_data.device_fap_data.resize(number_of_objects);
    Thrust_Fill(gpu_data.device_fap_data, R3(0));

    if (number_of_constraints > 0) {
        update_number.resize((number_of_constraints) * 2, 0);
        gpu_data.offset_counter.resize((number_of_constraints) * 2, 0);
        gpu_data.update_offset.resize((number_of_constraints) * 2, 0);
        body_num.resize((number_of_constraints) * 2, 0);
        gpu_data.device_dgm_data.resize((number_of_constraints));
        Thrust_Fill(gpu_data.device_dgm_data, 1);
        gpu_data.vel_update.resize((number_of_constraints) * 2);
        gpu_data.omg_update.resize((number_of_constraints) * 2);
        ChKernelOffsets CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(CASTI2(gpu_data.device_bids_data), CASTR4(gpu_data.device_bilateral_data), CASTU1(body_num));
        Thrust_Sequence(update_number);
        Thrust_Sequence(gpu_data.update_offset);
        Thrust_Fill(gpu_data.offset_counter, 0);
        Thrust_Sort_By_Key(body_num, update_number);
        Thrust_Sort_By_Key(update_number, gpu_data.update_offset);
        gpu_data.body_number = body_num;
        Thrust_Reduce_By_KeyB(gpu_data.number_of_updates, body_num, update_number, gpu_data.offset_counter);
        Thrust_Inclusive_Scan(gpu_data.offset_counter);
    }
}

void ChLcpSolverGPU::Iterate(gpu_container &gpu_data) {
}
void ChLcpSolverGPU::Reduce(gpu_container &gpu_data) {
}
void ChLcpSolverGPU::Integrate(gpu_container &gpu_data) {
    LCP_Integrate_Timestep CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
        CASTR3(gpu_data.device_aux_data),
        CASTR3(gpu_data.device_acc_data),
        CASTR4(gpu_data.device_rot_data),
        CASTR3(gpu_data.device_vel_data),
        CASTR3(gpu_data.device_omg_data),
        CASTR3(gpu_data.device_pos_data),
        CASTR3(gpu_data.device_lim_data));
}
void ChLcpSolverGPU::RunTimeStep(real step, gpu_container &gpu_data) {
    DBG("");
    lcp_omega_contact = omega;
    step_size = step;
    number_of_constraints = gpu_data.number_of_contacts + gpu_data.number_of_bilaterals;
    number_of_contacts = gpu_data.number_of_contacts;
    number_of_bilaterals = gpu_data.number_of_bilaterals;
    number_of_objects = gpu_data.number_of_objects;
    COPY_TO_CONST_MEM(number_of_contacts);
    COPY_TO_CONST_MEM(number_of_bilaterals);
    COPY_TO_CONST_MEM(number_of_objects);
    COPY_TO_CONST_MEM(step_size);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_ComputeGyro), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ChKernelLCPaddForces), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ChKernelOffsets), hipFuncCachePreferL1);
    Preprocess(gpu_data);
    number_of_updates = gpu_data.number_of_updates;
    COPY_TO_CONST_MEM(compliance);
    COPY_TO_CONST_MEM(complianceT);
    COPY_TO_CONST_MEM(alpha);
    COPY_TO_CONST_MEM(lcp_omega_bilateral);
    COPY_TO_CONST_MEM(lcp_omega_contact);
    COPY_TO_CONST_MEM(lcp_contact_factor);
    COPY_TO_CONST_MEM(number_of_updates);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Iteration_Contacts), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Iteration_Bilaterals), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Reduce_Speeds), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Integrate_Timestep), hipFuncCachePreferL1);
    real old_gam = 1;

    if (number_of_constraints != 0) {
        for (iteration_number = 0; iteration_number < max_iterations; iteration_number++) {
            LCP_Iteration_Contacts CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(
                CASTR3(gpu_data.device_norm_data),
                CASTR3(gpu_data.device_cpta_data),
                CASTR3(gpu_data.device_cptb_data),
                CASTR1(gpu_data.device_dpth_data),
                CASTI2(gpu_data.device_bids_data),
                CASTR3(gpu_data.device_gam_data),
                CASTR1(gpu_data.device_dgm_data),
                CASTR3(gpu_data.device_aux_data),
                CASTR3(gpu_data.device_inr_data),
                CASTR4(gpu_data.device_rot_data),
                CASTR3(gpu_data.device_vel_data),
                CASTR3(gpu_data.device_omg_data),
                CASTR3(gpu_data.device_pos_data),
                CASTR3(gpu_data.vel_update),
                CASTR3(gpu_data.omg_update),
                CASTU1(gpu_data.update_offset));
            LCP_Iteration_Bilaterals CUDA_KERNEL_DIM(BLOCKS(number_of_bilaterals), THREADS)(
                CASTR4(gpu_data.device_bilateral_data),
                CASTR3(gpu_data.device_aux_data),
                CASTR3(gpu_data.device_inr_data),
                CASTR4(gpu_data.device_rot_data),
                CASTR3(gpu_data.device_vel_data),
                CASTR3(gpu_data.device_omg_data),
                CASTR3(gpu_data.device_pos_data),
                CASTR3(gpu_data.vel_update),
                CASTR3(gpu_data.omg_update),
                CASTU1(gpu_data.update_offset),
                CASTR1(gpu_data.device_dgm_data));
            LCP_Reduce_Speeds CUDA_KERNEL_DIM(BLOCKS(number_of_updates), THREADS)(
                CASTR3(gpu_data.device_aux_data),
                CASTR3(gpu_data.device_vel_data),
                CASTR3(gpu_data.device_omg_data),
                CASTR3(gpu_data.vel_update),
                CASTR3(gpu_data.omg_update),
                CASTU1(gpu_data.body_number),
                CASTU1(gpu_data.offset_counter),
                CASTR3(gpu_data.device_fap_data));

            if (tolerance != 0) {
                if (iteration_number > 20 && iteration_number % 20 == 0) {
                    real gam = Max_DeltaGamma(gpu_data.device_dgm_data);

                    if (fabsf(old_gam - gam) < tolerance) {
                        break;
                    }

                    old_gam = gam;
                }
            }
        }
    }

    Integrate(gpu_data);
}
real ChLcpSolverGPU::Max_DeltaGamma(device_vector<real> &device_dgm_data) {
    return Thrust_Max(device_dgm_data);
}
real ChLcpSolverGPU::Min_DeltaGamma(device_vector<real> &device_dgm_data) {
    return Thrust_Min(device_dgm_data);
}
real ChLcpSolverGPU::Avg_DeltaGamma(uint number_of_constraints, device_vector<real> &device_dgm_data) {
    real gamma = (Thrust_Total(device_dgm_data)) / real(number_of_constraints);
    //cout << gamma << endl;
    return gamma;
}
__global__ void Compute_KE(real3 *vel, real3 *aux, real *ke) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
    real3 velocity = vel[index];
    ke[index] = .5 / aux[index].z * dot(velocity, velocity);
}
real ChLcpSolverGPU::Total_KineticEnergy(gpu_container &gpu_data) {
    thrust::device_vector<real> device_ken_data;
    device_ken_data.resize(gpu_data.number_of_objects);
    Compute_KE CUDA_KERNEL_DIM(BLOCKS(gpu_data.number_of_objects), THREADS)(CASTR3(gpu_data.device_vel_data), CASTR3(gpu_data.device_aux_data), CASTR1(device_ken_data));
    return (Thrust_Total(device_ken_data));
}




