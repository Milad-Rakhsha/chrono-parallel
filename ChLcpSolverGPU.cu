#include "hip/hip_runtime.h"
#include "ChLcpSolverGPU.h"
#include "ChThrustLinearAlgebra.cuh"
#include "solver/ChSolverGPU.cuh"
#include "ChJacobianGPU.h"
#include "ChIntegratorGPU.h"
#include "ChComputeRHSGPU.h"
#include "constraints/ChConstraintRigidRigid.h"
#include "constraints/ChConstraintBilateral.h"

using namespace chrono;
__constant__ uint number_of_objects_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ uint number_of_updates_const;
__constant__ real step_size_const;

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__host__ __device__ void function_addForces(uint& index, bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	if (active[index] != 0) {
		// v += m_inv * h * f
		vel[index] += forces[index] * mass[index];
		// w += J_inv * h * c
		omega[index] += torques[index] * inertia[index];
	}
}
__global__ void device_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
}

void ChLcpSolverGPU::host_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
#pragma omp parallel for

	for (uint index = 0; index < number_of_objects; index++) {
		function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
	}
}

__host__ __device__ void function_ComputeGyro(uint& index, real3* omega, real3* inertia, real3* gyro, real3* torque) {
	real3 body_inertia = inertia[index];
	body_inertia = R3(1.0 / body_inertia.x, 1.0 / body_inertia.y, 1.0 / body_inertia.z);
	real3 body_omega = omega[index];
	real3 gyr = cross(body_omega, body_inertia * body_omega);
	gyro[index] = gyr;
}

__global__ void device_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_ComputeGyro(index, omega, inertia, gyro, torque);
}

void ChLcpSolverGPU::host_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
#pragma omp parallel for

	for (uint index = 0; index < number_of_objects; index++) {
		function_ComputeGyro(index, omega, inertia, gyro, torque);
	}
}

void ChLcpSolverGPU::Preprocess() {
	data_container->number_of_updates = 0;
	data_container->host_data.gyr_data.resize(number_of_objects);

#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(number_of_rigid);

	device_ComputeGyro CUDA_KERNEL_DIM(BLOCKS(number_of_rigid), THREADS)(
			CASTR3(data_container->device_data.device_omg_data),
			CASTR3(data_container->device_data.device_inr_data),
			CASTR3(data_container->device_data.device_gyr_data),
			CASTR3(data_container->device_data.device_trq_data));
	device_addForces CUDA_KERNEL_DIM(BLOCKS(number_of_rigid), THREADS)(
			CASTB1(data_container->device_data.device_active_data),
			CASTR1(data_container->device_data.device_mass_data),
			CASTR3(data_container->device_data.device_inr_data),
			CASTR3(data_container->device_data.device_frc_data),
			CASTR3(data_container->device_data.device_trq_data),
			CASTR3(data_container->device_data.device_vel_data),
			CASTR3(data_container->device_data.device_omg_data));
#else
//	host_ComputeGyro(
//			data_container->host_data.omg_data.data(),
//			data_container->host_data.inr_data.data(),
//			data_container->host_data.gyr_data.data(),
//			data_container->host_data.trq_data.data());

	host_addForces(
			data_container->host_data.active_data.data(),
			data_container->host_data.mass_data.data(),
			data_container->host_data.inr_data.data(),
			data_container->host_data.frc_data.data(),
			data_container->host_data.trq_data.data(),
			data_container->host_data.vel_data.data(),
			data_container->host_data.omg_data.data());
#endif
}

void ChLcpSolverGPU::RunTimeStep(real step) {
	step_size = step;
	data_container->step_size = step;

	number_of_constraints = data_container->number_of_rigid_rigid * 6 + data_container->number_of_bilaterals;
	number_of_bilaterals = 0;     ///data_container->number_of_bilaterals;
	number_of_objects = data_container->number_of_rigid;
	//cout << number_of_constraints << " " << number_of_contacts << " " << number_of_bilaterals << " " << number_of_objects << endl;
#ifdef PRINT_DEBUG_GPU
	cout << "Preprocess: " << endl;
#endif
	Preprocess();
#ifdef PRINT_DEBUG_GPU
	cout << "Jacobians: " << endl;
#endif
//	ChJacobianGPU jacobian_compute;
//	jacobian_compute.ComputeJacobians(data_container);
//
//	ChComputeRHSGPU rhs_compute;
//	rhs_compute.ComputeRHS(data_container);

	data_container->host_data.rhs_data.resize(number_of_constraints);
	data_container->host_data.diag.resize(number_of_constraints);

	//bilateral.Diag();

	data_container->host_data.gamma_data.resize((number_of_constraints));

#pragma omp parallel for
	for (int i = 0; i < number_of_constraints; i++) {
		data_container->host_data.gamma_data[i] = 0;
	}
	if (warm_start) {
		//cout<<"WARM"<<endl;
		RunWarmStartPreprocess();
	}

	ChConstraintRigidRigid rigid_rigid(data_container);
	ChConstraintBilateral bilateral(data_container);

	ChSolverGPU solver;
	solver.SetMaxIterations(max_iteration);
	solver.SetTolerance(tolerance);

	solver.SetComplianceParameters(alpha, compliance, complianceT);
	solver.SetContactRecoverySpeed(contact_recovery_speed);
	solver.lcp_omega_bilateral = lcp_omega_bilateral;
	solver.rigid_rigid = &rigid_rigid;
	solver.bilateral = &bilateral;
	solver.lcp_omega_contact = lcp_omega_contact;
	solver.do_stab = do_stab;
	solver.Initial(step, data_container);
//solve initial
	//solver.SetComplianceParameters(.2, 1e-3, 1e-3);
	rigid_rigid.solve_sliding = true;
	rigid_rigid.ComputeJacobians();
	//rigid_rigid.ComputeRHS();
	bilateral.ComputeJacobians();
	bilateral.ComputeRHS();
	//solver.SetMaxIterations(5);
	//solver.Solve(solver_type);

//solve normal
	solver.SetMaxIterations(max_iteration);
	solver.SetComplianceParameters(alpha, compliance, complianceT);
	rigid_rigid.solve_sliding = false;
	rigid_rigid.solve_spinning = false;
	rigid_rigid.ComputeRHS();
	solver.Solve(solver_type);

	//solve full
	solver.SetMaxIterations(max_iteration/2.0);
	rigid_rigid.solve_sliding = true;
	rigid_rigid.solve_spinning = false;
	rigid_rigid.ComputeRHS();
	solver.Solve(solver_type);

	solver.SetMaxIterations(max_iteration/2.0);
	rigid_rigid.solve_sliding = true;
	rigid_rigid.solve_spinning = true;
	rigid_rigid.ComputeRHS();
	solver.Solve(solver_type);
	solver.ComputeImpulses();

	tot_iterations = solver.GetIteration();
	residual = solver.GetResidual();
	data_container->host_data.old_gamma_data = data_container->host_data.gamma_data;
	//rhs = data_container->host_data.rhs_data;
	//lambda = data_container->host_data.gam_data;

	for (int i = 0; i < solver.iter_hist.size(); i++) {
		AtIterationEnd(solver.maxd_hist[i], solver.maxdeltalambda_hist[i], solver.iter_hist[i]);
	}
	if (warm_start) {
		//RunWarmStartPostProcess();
	}
	//determine binning

	//stabilization  code
	//jacobian_compute.ComputeJacobians(data_container);
	//rhs_compute.ComputeRHS(data_container);

	//solver.VelocityStabilization(data_container);

#ifdef PRINT_DEBUG_GPU
	cout << "Solve Done: "<<residual << endl;
#endif
	//ChIntegratorGPU integrator;
	//integrator.IntegrateSemiImplicit(step, data_container->gpu_data);
}

void ChLcpSolverGPU::RunWarmStartPostProcess() {
	if (data_container->number_of_rigid_rigid == 0) {
		return;
	}

	num_bins_per_axis = I3(20, 40, 20);
	int l = num_bins_per_axis.x;
	int h = num_bins_per_axis.y;
	int w = num_bins_per_axis.z;
	uint N = data_container->number_of_rigid_rigid;

	thrust::host_vector<real3> points(N);
	thrust::host_vector<uint> counter((l + 1) * (w + 1) * (h + 1), 0);

	data_container->host_data.bin_number.resize((l + 1) * (w + 1) * (h + 1));
	thrust::fill(data_container->host_data.bin_number.begin(), data_container->host_data.bin_number.end(), 0);

	points = (data_container->host_data.cpta_rigid_rigid + data_container->host_data.cptb_rigid_rigid);
	points = .5 * points;

	real3 max_point = R3(-FLT_MAX);
	real3 min_point = R3(FLT_MAX);
	real3 origin;

	for (int i = 0; i < N; i++) {
		max_point.x = max(points[i].x, max_point.x);
		max_point.y = max(points[i].y, max_point.y);
		max_point.z = max(points[i].z, max_point.z);

		min_point.x = min(points[i].x, min_point.x);
		min_point.y = min(points[i].y, min_point.y);
		min_point.z = min(points[i].z, min_point.z);
	}

	origin = min_point;
	real3 bin_size_vec = (fabs(max_point - origin));
	bin_size_vec.x = bin_size_vec.x / real(l);
	bin_size_vec.y = bin_size_vec.y / real(h);
	bin_size_vec.z = bin_size_vec.z / real(w);

	for (int i = 0; i < N; i++) {
		points[i] = points[i] - origin;

		int3 temp;
		temp.x = floor(points[i].x / bin_size_vec.x);
		temp.y = floor(points[i].y / bin_size_vec.y);
		temp.z = floor(points[i].z / bin_size_vec.z);
		data_container->host_data.bin_number[temp.x + temp.y * w + temp.z * w * h] += data_container->host_data.gamma_data[i];
		counter[temp.x + temp.y * w + temp.z * w * h]++;
	}
	for (int i = 0; i < counter.size(); i++) {
		if (counter[i] > 0) {
			data_container->host_data.bin_number[i] = data_container->host_data.bin_number[i] / counter[i];
		}
	}
}
void ChLcpSolverGPU::RunWarmStartPreprocess() {
	if (data_container->number_of_rigid_rigid == 0) {
		return;
	}

	if (data_container->host_data.old_pair_rigid_rigid.size() != 0) {

//		cout << "SORTEDN: " << thrust::is_sorted(data_container->host_data.pair_rigid_rigid.begin(), data_container->host_data.pair_rigid_rigid.end())<<endl;
//		cout << "SORTEDO: " << thrust::is_sorted(data_container->host_data.old_pair_rigid_rigid.begin(), data_container->host_data.old_pair_rigid_rigid.end())<<endl;
//		return;

		thrust::host_vector<long long> res1(data_container->host_data.pair_rigid_rigid.size());

		uint numP = thrust::set_intersection(
				thrust::omp::par,
				data_container->host_data.pair_rigid_rigid.begin(),
				data_container->host_data.pair_rigid_rigid.end(),
				data_container->host_data.old_pair_rigid_rigid.begin(),
				data_container->host_data.old_pair_rigid_rigid.end(),
				res1.begin()) - res1.begin();     //list of persistent contacts

		if (numP > 0) {
			res1.resize(numP);
			thrust::host_vector<uint> temporaryA(numP);
			thrust::host_vector<uint> temporaryB(numP);

			thrust::lower_bound(
					thrust::omp::par,
					data_container->host_data.old_pair_rigid_rigid.begin(),
					data_container->host_data.old_pair_rigid_rigid.end(),
					res1.begin(),
					res1.end(),
					temporaryA.begin());     //return index of common new contact
			thrust::lower_bound(thrust::omp::par, data_container->host_data.pair_rigid_rigid.begin(), data_container->host_data.pair_rigid_rigid.end(), res1.begin(), res1.end(), temporaryB.begin());     //return index of common new contact

			uint number_of_rigid_rigid = data_container->number_of_rigid_rigid;
			uint old_number_of_rigid_rigid = data_container->old_number_of_rigid_rigid;
#pragma omp parallel for
			for (int i = 0; i < numP; i++) {

				M33 contact_plane_old;

				{
					real3 U = data_container->host_data.old_norm_rigid_rigid[temporaryA[i]];

					if (U == R3(0, 0, 0)) {
						U = R3(1, 0, 0);
					} else {
						U = normalize(U);
					}
					U = normalize(U);
					real3 W = cross(U, R3(0, 1, 0));
					real mzlen = length(W);

					if (mzlen < 0.0001) {     // was near singularity? change singularity reference vector!
						real3 mVsingular = R3(0, 1, 0);
						if (mVsingular.x < 0.9) {
							mVsingular = R3(0, 0, 1);
						}
						if (mVsingular.y < 0.9) {
							mVsingular = R3(0, 1, 0);
						}
						if (mVsingular.z < 0.9) {
							mVsingular = R3(1, 0, 0);
						}

						W = cross(U, mVsingular);
						mzlen = length(W);
					}
					W = W * 1.0 / mzlen;
					real3 V = cross(W, U);

					contact_plane_old.U = U;
					contact_plane_old.V = V;
					contact_plane_old.W = W;
				}

				real3 old_gamma = R3(0), old_gamma_spinning = R3(0);
				old_gamma.x = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 0];
				old_gamma.y = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 1];
				old_gamma.z = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 2];

				old_gamma_spinning.x = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 3];
				old_gamma_spinning.y = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 4];
				old_gamma_spinning.z = data_container->host_data.old_gamma_data[temporaryA[i] + old_number_of_rigid_rigid * 5];

				real3 global_gamma = MatMult(contact_plane_old, old_gamma);
				real3 global_gamma_spin = MatMult(contact_plane_old, old_gamma_spinning);

				M33 contact_plane_new;

				{
					real3 U = data_container->host_data.norm_rigid_rigid[temporaryB[i]];

					if (U == R3(0, 0, 0)) {
						U = R3(1, 0, 0);
					} else {
						U = normalize(U);
					}
					U = normalize(U);
					real3 W = cross(U, R3(0, 1, 0));
					real mzlen = length(W);

					if (mzlen < 0.0001) {     // was near singularity? change singularity reference vector!
						real3 mVsingular = R3(0, 1, 0);
						if (mVsingular.x < 0.9) {
							mVsingular = R3(0, 0, 1);
						}
						if (mVsingular.y < 0.9) {
							mVsingular = R3(0, 1, 0);
						}
						if (mVsingular.z < 0.9) {
							mVsingular = R3(1, 0, 0);
						}

						W = cross(U, mVsingular);
						mzlen = length(W);
					}
					W = W * 1.0 / mzlen;
					real3 V = cross(W, U);

					contact_plane_new.U = U;
					contact_plane_new.V = V;
					contact_plane_new.W = W;
				}
				real3 new_gamma = MatTMult(contact_plane_new, global_gamma);
				real3 new_gamma_spin = MatTMult(contact_plane_new, global_gamma_spin);

				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 0] = new_gamma.x * .4;
				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 1] = new_gamma.y * .4;
				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 2] = new_gamma.z * .4;

				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 3] = new_gamma_spin.x * .4;
				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 4] = new_gamma_spin.y * .4;
				data_container->host_data.gamma_data[temporaryB[i] + number_of_rigid_rigid * 5] = new_gamma_spin.z * .4;

			}
		}
	}

//
//	num_bins_per_axis = I3(20,40,20);
//	int l = num_bins_per_axis.x;
//	int h = num_bins_per_axis.y;
//	int w = num_bins_per_axis.z;
//	uint N = data_container->number_of_rigid_rigid;
//
//	thrust::host_vector<real3> points(N);
//
//	points = (data_container->host_data.cpta_rigid_rigid + data_container->host_data.cptb_rigid_rigid);
//	points = .5 * points;
//
//	real3 max_point = R3(-FLT_MAX);
//	real3 min_point = R3(FLT_MAX);
//
//	for (int i = 0; i < N; i++) {
//		max_point.x = max(points[i].x, max_point.x);
//		max_point.y = max(points[i].y, max_point.y);
//		max_point.z = max(points[i].z, max_point.z);
//
//		min_point.x = min(points[i].x, min_point.x);
//		min_point.y = min(points[i].y, min_point.y);
//		min_point.z = min(points[i].z, min_point.z);
//	}
//
//	origin = min_point;
//	bin_size_vec = (fabs(max_point - origin));
//	bin_size_vec.x = bin_size_vec.x / real(l);
//	bin_size_vec.y = bin_size_vec.y / real(h);
//	bin_size_vec.z = bin_size_vec.z / real(w);
//
//	for (int i = 0; i < N; i++) {
//		points[i] = points[i] - origin;
//
//		int3 temp;
//		temp.x = floor(points[i].x / bin_size_vec.x);
//		temp.y = floor(points[i].y / bin_size_vec.y);
//		temp.z = floor(points[i].z / bin_size_vec.z);
//		data_container->host_data.gamma_data[i] = data_container->host_data.bin_number[temp.x + temp.y * w + temp.z * w * h];
//
//	}

}

