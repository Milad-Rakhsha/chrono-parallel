#include "hip/hip_runtime.h"
#include "ChLcpSolverGPU.h"
#include "ChThrustLinearAlgebra.cuh"

using namespace chrono;

__constant__ real lcp_omega_bilateral_const;
__constant__ real lcp_omega_contact_const;
__constant__ real lcp_contact_factor_const;
__constant__ uint number_of_objects_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ uint number_of_updates_const;
__constant__ real step_size_const;
__constant__ real compliance_const;
__constant__ real complianceT_const;
__constant__ real alpha_const; // [R]=alpha*[K]

//__constant__ real force_factor_const;
//__constant__ real negated_recovery_speed_const;
////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__ void inline Compute_Jacobian(const real4& quaternion_rotation, const real3& normal, const real3& tangent_u, const real3& tangent_w, const real3& point, real3& T3, real3& T4,
		real3& T5) {
	real t1 = quaternion_rotation.x * quaternion_rotation.y;
	real t2 = quaternion_rotation.w * quaternion_rotation.z;
	real t3 = 2 * t1 - 2 * t2;
	real t4 = quaternion_rotation.x * quaternion_rotation.x;
	real t5 = quaternion_rotation.y * quaternion_rotation.y;
	real t6 = quaternion_rotation.w * quaternion_rotation.w;
	real t7 = quaternion_rotation.z * quaternion_rotation.z;
	real t8 = t6 - t4 + t5 - t7;
	real t9 = quaternion_rotation.y * quaternion_rotation.z;
	real t10 = quaternion_rotation.w * quaternion_rotation.x;
	real t11 = 2 * t9 + 2 * t10;
	real t12 = normal.x * t3 + normal.y * t8 + normal.z * t11;
	real t13 = quaternion_rotation.x * quaternion_rotation.z;
	real t14 = quaternion_rotation.w * quaternion_rotation.y;
	real t15 = 2 * t13 + 2 * t14;
	t9 = 2 * t9 - 2 * t10;
	t10 = t6 - t4 - t5 + t7;
	real t16 = t15 * point.x + t9 * point.y + t10 * point.z;
	real t17 = normal.x * t15 + normal.y * t9 + normal.z * t10;
	real t18 = -t3 * point.x - t8 * point.y - t11 * point.z;
	t4 = t6 + t4 - t5 - t7;
	t1 = 2 * t1 + 2 * t2;
	t2 = 2 * t13 - 2 * t14;
	t5 = normal.x * t4 + normal.y * t1 + normal.z * t2;
	t6 = -t16;
	t7 = t4 * point.x + t1 * point.y + t2 * point.z;
	t13 = -t18;
	t14 = -t7;
	real t19 = tangent_u.x * t3 + tangent_u.y * t8 + tangent_u.z * t11;
	real t20 = tangent_u.x * t15 + tangent_u.y * t9 + tangent_u.z * t10;
	real t21 = tangent_u.x * t4 + tangent_u.y * t1 + tangent_u.z * t2;
	t3 = tangent_w.x * t3 + tangent_w.y * t8 + tangent_w.z * t11;
	t8 = tangent_w.x * t15 + tangent_w.y * t9 + tangent_w.z * t10;
	t1 = tangent_w.x * t4 + tangent_w.y * t1 + tangent_w.z * t2;
	T3.x = t12 * t16 + t17 * t18;
	T3.y = t5 * t6 + t17 * t7;
	T3.z = t5 * t13 + t12 * t14;
	T4.x = t19 * t16 + t20 * t18;
	T4.y = t21 * t6 + t20 * t7;
	T4.z = t21 * t13 + t19 * t14;
	T5.x = t3 * t16 + t8 * t18;
	T5.y = t1 * t6 + t8 * t7;
	T5.z = t1 * t13 + t3 * t14;
}
__host__ __device__ void inline function_ContactJacobians(uint& index, uint& num_contacts, real3* norm, real3* ptA, real3* ptB, real3* pos, real4* rot, int2* ids, real3* JXYZA, real3* JXYZB,
		real3* JUVWA, real3* JUVWB) {
	real3 U = norm[index];
	real3 W = fabs(U);
	real3 V = R3(0, U.z, -U.y);

	if (W.x > W.y) {
		V = R3(-U.z, 0, U.x);
	}

	if (W.y > W.z) {
		V = R3(U.y, -U.x, 0);
	}

	V = normalize(V);
	W = cross(U, V);

	JXYZA[index + num_contacts * 0] = -U;
	JXYZA[index + num_contacts * 1] = -V;
	JXYZA[index + num_contacts * 2] = -W;

	JXYZB[index + num_contacts * 0] = U;
	JXYZB[index + num_contacts * 1] = V;
	JXYZB[index + num_contacts * 2] = W;

	int2 body_id = ids[index];
	real3 T3, T4, T5, T6, T7, T8;

	real3 sbar = ptA[index] - pos[body_id.x];
	real4 E1 = rot[body_id.x];
	Compute_Jacobian(E1, U, V, W, sbar, T3, T4, T5);

	sbar = ptB[index] - pos[body_id.y];
	real4 E2 = rot[body_id.y];
	Compute_Jacobian(E2, U, V, W, sbar, T6, T7, T8);
	T6 = -T6;
	T7 = -T7;
	T8 = -T8;

	JUVWA[index + num_contacts * 0] = T3;
	JUVWA[index + num_contacts * 1] = T4;
	JUVWA[index + num_contacts * 2] = T5;

	JUVWB[index + num_contacts * 0] = T6;
	JUVWB[index + num_contacts * 1] = T7;
	JUVWB[index + num_contacts * 2] = T8;

//    std::cout << "[" << N.x << " " << N.y << " " << N.z << "]" << "[" << -N.x << " " << -N.y << " " << -N.z << "]" << std::endl;
//    std::cout << "[" << U.x << " " << U.y << " " << U.z << "]" << "[" << -U.x << " " << -U.y << " " << -U.z << "]" << std::endl;
//    std::cout << "[" << W.x << " " << W.y << " " << W.z << "]" << "[" << -W.x << " " << -W.y << " " << -W.z << "]" << std::endl;
//    std::cout << "------" << std::endl;
//    std::cout << "[" << T3.x << " " << T3.y << " " << T3.z << "]" << "[" << T6.x << " " << T6.y << " " << T6.z << "]" << std::endl;
//    std::cout << "[" << T4.x << " " << T4.y << " " << T4.z << "]" << "[" << T7.x << " " << T7.y << " " << T7.z << "]" << std::endl;
//    std::cout << "[" << T5.x << " " << T5.y << " " << T5.z << "]" << "[" << T8.x << " " << T8.y << " " << T8.z << "]" << std::endl;
//    std::cout << "****************" << std::endl;
}

__global__ void device_ContactJacobians(real3* norm, real3* ptA, real3* ptB, int2* ids, real4* rot, real3* pos, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
	function_ContactJacobians(index, number_of_contacts_const, norm, ptA, ptB, pos, rot, ids, JXYZA, JXYZB, JUVWA, JUVWB);
}
void ChLcpSolverGPU::host_ContactJacobians(real3* norm, real3* ptA, real3* ptB, int2* ids, real4* rot, real3* pos, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB) {
	// #pragma omp parallel for schedule(guided)
	for (uint index = 0; index < number_of_contacts; index++) {
		function_ContactJacobians(index, number_of_contacts, norm, ptA, ptB, pos, rot, ids, JXYZA, JXYZB, JUVWA, JUVWB);
	}
}
__host__ __device__ void function_shurA(uint& index, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real * gamma, int2* ids, real3 * QXYZ, real3 * QUVW) {
	real q_x, q_y, q_z, q_u, q_v, q_w;
	real temp = 0, m;
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;

	if (index != b1) {
		q_x = JXYZA[index].x * gamma[index];
		q_y = JXYZA[index].y * gamma[index];
		q_z = JXYZA[index].z * gamma[index];
		q_u = JUVWA[index].x * gamma[index];
		q_v = JUVWA[index].y * gamma[index];
		q_w = JUVWA[index].z * gamma[index];

		QXYZ[b1] += R3(q_x, q_y, q_z);
		QUVW[b1] += R3(q_u, q_v, q_w);

	}

	if (index != b2) {
		q_x = JXYZB[index].x * gamma[index];
		q_y = JXYZB[index].y * gamma[index];
		q_z = JXYZB[index].z * gamma[index];
		q_u = JUVWB[index].x * gamma[index];
		q_v = JUVWB[index].y * gamma[index];
		q_w = JUVWB[index].z * gamma[index];

		QXYZ[b2] += R3(q_x, q_y, q_z);
		QUVW[b2] += R3(q_u, q_v, q_w);
	}
}
__host__ __device__ void function_shurB(uint& index, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real * gamma, real * inv_mass, real3 * inv_inertia, int2* ids, real3 * QXYZ, real3 * QUVW,
		real * AX) {
	real temp = 0, m;
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;
	real m1 = inv_mass[b1];
	real m2 = inv_mass[b2];

	real3 inertia1 = inv_inertia[b1];
	real3 inertia2 = inv_inertia[b2];
	if (index != b1) {
		temp += QXYZ[b1].x * JXYZA[index].x * m1;
		temp += QXYZ[b1].y * JXYZA[index].y * m1;
		temp += QXYZ[b1].z * JXYZA[index].z * m1;
		temp += QUVW[b1].x * JUVWA[index].x * inertia1.x;
		temp += QUVW[b1].y * JUVWA[index].y * inertia1.y;
		temp += QUVW[b1].z * JUVWA[index].z * inertia1.z;
	}

	if (index != b2) {
		temp += QXYZ[b2].x * JXYZB[index].x * m1;
		temp += QXYZ[b2].y * JXYZB[index].y * m1;
		temp += QXYZ[b2].z * JXYZB[index].z * m1;
		temp += QUVW[b2].x * JUVWB[index].x * inertia1.x;
		temp += QUVW[b2].y * JUVWB[index].y * inertia1.y;
		temp += QUVW[b2].z * JUVWB[index].z * inertia1.z;
	}

	AX[index] += temp;
}
__host__ __device__ void function_RHS(uint &index, int2* ids, real * inv_mass, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real3* vel, real3* omega, real* correction, real & step_size,
		real * rhs) {
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;
	real m1 = inv_mass[b1];
	real m2 = inv_mass[b2];

	real temp = 0;
	temp -= JXYZA[index].x * vel[b1].x;
	temp -= JXYZA[index].y * vel[b1].y;
	temp -= JXYZA[index].z * vel[b1].z;
	temp -= JUVWA[index].x * omega[b1].x;
	temp -= JUVWA[index].y * omega[b1].y;
	temp -= JUVWA[index].z * omega[b1].z;

	temp -= JXYZB[index].x * vel[b2].x;
	temp -= JXYZB[index].y * vel[b2].y;
	temp -= JXYZB[index].z * vel[b2].z;
	temp -= JUVWB[index].x * omega[b2].x;
	temp -= JUVWB[index].y * omega[b2].y;
	temp -= JUVWB[index].z * omega[b2].z;

	rhs[index] = temp - fmaxf(1 / step_size * correction[index], 0);

}
__host__ __device__ void function_InvNDiag(uint index, int b1, int b2,int2* ids, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real * inv_mass, real3 * inv_inertia, real & inv_n_diag) {

	real Jx1 = JXYZA[index].x;
	real Jy1 = JXYZA[index].y;
	real Jz1 = JXYZA[index].z;
	real Ju1 = JUVWA[index].x;
	real Jv1 = JUVWA[index].y;
	real Jw1 = JUVWA[index].z;
	//
	real Jx2 = JXYZB[index].x;
	real Jy2 = JXYZB[index].y;
	real Jz2 = JXYZB[index].z;
	real Ju2 = JUVWB[index].x;
	real Jv2 = JUVWB[index].y;
	real Jw2 = JUVWB[index].z;
	//
	real m1 = inv_mass[b1];
	real m2 = inv_mass[b2];
	//
	real3 i1 = inv_inertia[b1];
	real3 i2 = inv_inertia[b2];

	real part1 = dot(JXYZA[index], JXYZA[index]) * m1;
	real part2 = dot(JUVWA[index] * JUVWA[index], i1);
	real part3 = dot(JXYZB[index], JXYZB[index]) * m2;
	real part4 = dot(JUVWB[index] * JUVWB[index], i2);
	inv_n_diag = (part1 + part2 + part3 + part4);
}
__host__ __device__ void function_Project(uint & index, real3 & gamma, real* fric, int2* ids) {
	int2 body_id = ids[index];

	real f_tang = sqrtf(gamma.y * gamma.y + gamma.z * gamma.z);
	real mu = (fric[body_id.x] + fric[body_id.y]) * .5f;

	if (f_tang > (mu * gamma.x)) { // inside upper cone? keep untouched!
		if ((f_tang) < -(1.0 / mu) * gamma.x || (fabsf(gamma.x) < 0)) { // inside lower cone? reset  normal,u,v to zero!
			gamma = R3(0);
		} else { // remaining case: project orthogonally to generator segment of upper cone
			gamma.x = (f_tang * mu + gamma.x) / (mu * mu + 1);
			real tproj_div_t = (gamma.x * mu) / f_tang; //  reg = tproj_div_t
			gamma.y *= tproj_div_t;
			gamma.z *= tproj_div_t;
		}
	} else if (mu == 0) {
		gamma.y = gamma.z = 0;
	}

}

__host__ __device__ void function_process_contacts(uint &index, real& step_size, real& lcp_contact_factor, uint& number_of_contacts, real& lcp_omega_contact, real3* JXYZA, real3* JXYZB, real3* JUVWA,
		real3* JUVWB, real* contactDepth, int2* ids, real3* G, real* dG, real* inv_mass, real* fric, real3* inv_inertia, real4* rot, real3* vel, real3* omega, real3* pos, real3* updateV,
		real3* updateO, uint* offset) {

	int2 body_id = ids[index];
	real depth = -fabs(contactDepth[index]);

	real bi = depth / (step_size);//fmaxf((), -lcp_contact_factor);
	real3 W1 = omega[body_id.x];
	real3 W2 = omega[body_id.y];
	real3 V1 = vel[body_id.x];
	real3 V2 = vel[body_id.y];
	real3 gamma;
	gamma.x = dot(JUVWA[index + number_of_contacts * 0], W1) + dot(JXYZA[index + number_of_contacts * 0], V1) + dot(JUVWB[index + number_of_contacts * 0], W2)
			+ dot(JXYZB[index + number_of_contacts * 0], V2) + bi /*+ cfm * gamma_old.x*/; //+bi

	gamma.y = dot(JUVWA[index + number_of_contacts * 1], W1) + dot(JXYZA[index + number_of_contacts * 1], V1) + dot(JUVWB[index + number_of_contacts * 1], W2)
			+ dot(JXYZB[index + number_of_contacts * 1], V2) /*+ cfmT * gamma_old.y*/;

	gamma.z = dot(JUVWA[index + number_of_contacts * 2], W1) + dot(JXYZA[index + number_of_contacts * 2], V1) + dot(JUVWB[index + number_of_contacts * 2], W2)
			+ dot(JXYZB[index + number_of_contacts * 2], V2) /*+ cfmT * gamma_old.z*/;
	real3 In1 = inv_inertia[body_id.x]; // bring in the inertia attributes; to be used to compute \eta
	real3 In2 = inv_inertia[body_id.y]; // bring in the inertia attributes; to be used to compute \eta

  	real eta =
			dot(JUVWA[index + number_of_contacts * 0] * JUVWA[index + number_of_contacts * 0], In1) +
			dot(JUVWA[index + number_of_contacts * 1] * JUVWA[index + number_of_contacts * 1], In1)+
			dot(JUVWA[index + number_of_contacts * 2] * JUVWA[index + number_of_contacts * 2], In1); // update expression of eta

	eta += dot(JUVWB[index + number_of_contacts * 0] * JUVWB[index + number_of_contacts * 0], In2) +
		   dot(JUVWB[index + number_of_contacts * 1] * JUVWB[index + number_of_contacts * 1], In2)+
		   dot(JUVWB[index + number_of_contacts * 2] * JUVWB[index + number_of_contacts * 2], In2);

	eta += dot(JXYZA[index + number_of_contacts * 0], JXYZA[index + number_of_contacts * 0])* inv_mass[body_id.x] +
			dot(JXYZA[index + number_of_contacts * 1], JXYZA[index + number_of_contacts * 1])* inv_mass[body_id.x]+
			dot(JXYZA[index + number_of_contacts * 2], JXYZA[index + number_of_contacts * 2])* inv_mass[body_id.x] ;

	eta += dot(JXYZB[index + number_of_contacts * 0], JXYZB[index + number_of_contacts * 0])* inv_mass[body_id.y] +
			dot(JXYZB[index + number_of_contacts * 1], JXYZB[index + number_of_contacts * 1])* inv_mass[body_id.y]+
			dot(JXYZB[index + number_of_contacts * 2], JXYZB[index + number_of_contacts * 2])* inv_mass[body_id.y] ;

	eta = 3*lcp_omega_contact / eta; // final value of eta
	real3 gamma_old = G[index];
	gamma = eta * gamma; // perform gamma *= omega*eta
	gamma = gamma_old - gamma; // perform gamma = gamma_old - gamma ;  in place.
	/// ---- perform projection of 'a8' onto friction cone  --------

	function_Project(index, gamma, fric, ids);

	G[index] = gamma; // store gamma_new
	gamma -= gamma_old; // compute delta_gamma = gamma_new - gamma_old   = delta_gamma.
	dG[index] = length(gamma);
	real3 vB1 = JXYZA[index + number_of_contacts * 0] * gamma.x + JXYZA[index + number_of_contacts * 1] * gamma.y + JXYZA[index + number_of_contacts * 2] * gamma.z;
	real3 vB2 = JXYZB[index + number_of_contacts * 0] * gamma.x + JXYZB[index + number_of_contacts * 1] * gamma.y + JXYZB[index + number_of_contacts * 2] * gamma.z;

	int offset1 = offset[index];
	int offset2 = offset[index + number_of_contacts];
	updateV[offset1] = real3(vB1 * inv_mass[body_id.x]); // compute and store dv1
	updateO[offset1] = real3((JUVWA[index + number_of_contacts * 0] * gamma.x + JUVWA[index + number_of_contacts * 1] * gamma.y + JUVWA[index + number_of_contacts * 2] * gamma.z) * In1); // compute dw1 =  Inert.1' * J1w^ * deltagamma  and store  dw1
	updateV[offset2] = real3(vB2 * inv_mass[body_id.y]); // compute and store dv2
	updateO[offset2] = real3((JUVWB[index + number_of_contacts * 0] * gamma.x + JUVWB[index + number_of_contacts * 1] * gamma.y + JUVWB[index + number_of_contacts * 2] * gamma.z) * In2); // compute dw2 =  Inert.2' * J2w^ * deltagamma  and store  dw2
}

//  Kernel for a single iteration of the LCP over all contacts
//  Version 2.0 - Tasora
//  Version 2.2- Hammad (optimized, cleaned etc)
__global__ void device_process_contacts(real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real* contactDepth, int2* ids, real3* G, real* dG, real* mass, real* fric, real3* inertia, real4* rot,
		real3* vel, real3* omega, real3* pos, real3* updateV, real3* updateO, uint* offset) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
	function_process_contacts(index, step_size_const, lcp_contact_factor_const, number_of_contacts_const, lcp_omega_contact_const, JXYZA, JXYZB, JUVWA, JUVWB, contactDepth, ids, G, dG, mass, fric,
			inertia, rot, vel, omega, pos, updateV, updateO, offset);
}

void ChLcpSolverGPU::host_process_contacts(real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real* contactDepth, int2* ids, real3* G, real* dG, real* mass, real* fric, real3* inertia,
		real4* rot, real3* vel, real3* omega, real3* pos, real3* updateV, real3* updateO, uint* offset) {
#pragma omp parallel for schedule(guided)

	for (uint index = 0; index < number_of_contacts; index++) {
		function_process_contacts(index, step_size, lcp_contact_factor, number_of_contacts, lcp_omega_contact, JXYZA, JXYZB, JUVWA, JUVWB, contactDepth, ids, G, dG, mass, fric, inertia, rot, vel,
				omega, pos, updateV, updateO, offset);
	}
}

///////////////////////////////////////////////////////////////////////////////////
// Kernel for a single iteration of the LCP over all scalar bilateral contacts
// (a bit similar to the ChKernelLCPiteration above, but without projection etc.)
// Version 2.0 - Tasora
//

__host__ __device__ void function_Bilaterals(uint& index, uint& number_of_bilaterals, uint& number_of_contacts, real& lcp_omega_bilateral, real4* bilaterals, real* mass, real3* inertia, real4* rot,
		real3* vel, real3* omega, real3* pos, real3* updateV, real3* updateO, uint* offset, real* dG) {
	real4 vA;
	real3 vB;
	real gamma_new = 0, gamma_old = 0;
	int B1_index = 0, B2_index = 0;
	B1_index = bilaterals[index].w;
	B2_index = bilaterals[index + number_of_bilaterals].w;
	real mass1 = mass[B1_index];
	real mass2 = mass[B2_index];
	// ---- perform   gamma_new = ([J1 J2] {v1 | v2}^ + b)
	{
		vA = bilaterals[index]; // line 0
		vB = vel[B1_index]; // v1
		gamma_new += dot3(vA, vB);
		vA = bilaterals[index + 2 * number_of_bilaterals]; // line 2
		vB = omega[B1_index]; // w1
		gamma_new += dot3(vA, vB);
	}
	{
		vA = bilaterals[index + number_of_bilaterals]; // line 1
		vB = vel[B2_index]; // v2
		gamma_new += dot3(vA, vB);
		vA = bilaterals[index + 3 * number_of_bilaterals]; // line 3
		vB = omega[B2_index]; // w2
		gamma_new += dot3(vA, vB);
	}
	vA = bilaterals[index + 4 * number_of_bilaterals]; // line 4   (eta, b, gamma, 0)
	gamma_new += vA.y; // add known term     + b
	gamma_old = vA.z; // old gamma
	/// ---- perform gamma_new *= omega/g_i
	gamma_new *= lcp_omega_bilateral; // lcp_omega_const is in constant memory
	gamma_new *= vA.x; // eta = 1/g_i;
	/// ---- perform gamma_new = gamma_old - gamma_new ; in place.
	gamma_new = gamma_old - gamma_new;

	/// ---- perform projection of 'a' (only if simple unilateral behavior C>0 is requested)
	if (vA.w && gamma_new < 0.) {
		gamma_new = 0.;
	}

	// ----- store gamma_new
	vA.z = gamma_new;
	bilaterals[index + 4 * number_of_bilaterals] = vA;
	/// ---- compute delta in multipliers: gamma_new = gamma_new - gamma_old   = delta_gamma    , in place.
	gamma_new -= gamma_old;
	dG[number_of_contacts + index] = (gamma_new);
	/// ---- compute dv1 =  invInert.18 * J1^ * deltagamma
	vB = inertia[B1_index]; // iJ iJ iJ im
	vA = (bilaterals[index]) * mass1 * gamma_new; // line 0: J1(x)
	int offset1 = offset[2 * number_of_contacts + index];
	int offset2 = offset[2 * number_of_contacts + index + number_of_bilaterals];
	updateV[offset1] = make_real3(vA); //  ---> store  v1 vel. in reduction buffer
	updateO[offset1] = make_real3((bilaterals[index + 2 * number_of_bilaterals]) * R4(vB) * gamma_new); // line 2:  J1(w)// ---> store  w1 vel. in reduction buffer
	vB = inertia[B2_index]; // iJ iJ iJ im
	vA = (bilaterals[index + number_of_bilaterals]) * mass2 * gamma_new; // line 1: J2(x)
	updateV[offset2] = make_real3(vA); //  ---> store  v2 vel. in reduction buffer
	updateO[offset2] = make_real3((bilaterals[index + 3 * number_of_bilaterals]) * R4(vB) * gamma_new); // line 3:  J2(w)// ---> store  w2 vel. in reduction buffer
}

__global__ void device_Bilaterals(real4* bilaterals, real* mass, real3* inertia, real4* rot, real3* vel, real3* omega, real3* pos, real3* updateV, real3* updateO, uint* offset, real* dG) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_bilaterals_const);
	function_Bilaterals(index, number_of_bilaterals_const, number_of_contacts_const, lcp_omega_bilateral_const, bilaterals, mass, inertia, rot, vel, omega, pos, updateV, updateO, offset, dG);
}
void ChLcpSolverGPU::host_Bilaterals(real4* bilaterals, real* mass, real3* inertia, real4* rot, real3* vel, real3* omega, real3* pos, real3* updateV, real3* updateO, uint* offset, real* dG) {
	for (uint index = 0; index < number_of_bilaterals; index++) {
		function_Bilaterals(index, number_of_bilaterals, number_of_contacts, lcp_omega_bilateral, bilaterals, mass, inertia, rot, vel, omega, pos, updateV, updateO, offset, dG);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__host__ __device__ void function_addForces(uint& index, bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	if (active[index] != 0) {
		// v += m_inv * h * f
		vel[index] += forces[index] * mass[index];
		// w += J_inv * h * c
		//omega[index] += torques[index] * inertia[index];
	}
}
__global__ void device_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
}

void ChLcpSolverGPU::host_addForces(bool* active, real* mass, real3* inertia, real3* forces, real3* torques, real3* vel, real3* omega) {
#pragma omp parallel for schedule(guided)

	for (uint index = 0; index < number_of_objects; index++) {
		function_addForces(index, active, mass, inertia, forces, torques, vel, omega);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// Updates the speeds in the body buffer with values accumulated in the
// reduction buffer:   V_new = V_old + delta_speeds

__host__ __device__ void function_Reduce_Speeds(uint& index, bool* active, real* mass, real3* vel, real3* omega, real3* updateV, real3* updateO, uint* d_body_num, uint* counter, real3* fap) {
	int start = (index == 0) ? 0 : counter[index - 1], end = counter[index];
	int id = d_body_num[end - 1], j;
	//real3 auxd = active[id];

	if (active[id] == 0) {
		return;
	}

	real3 mUpdateV = R3(0);
	real3 mUpdateO = R3(0);

	for (j = 0; j < end - start; j++) {
		mUpdateV = mUpdateV + updateV[j + start];
		mUpdateO = mUpdateO + updateO[j + start];
	}

	//fap[id] += (mUpdateV / mass[id]) / step_size_const;
	vel[id] += (mUpdateV);
	omega[id] += (mUpdateO);
}

__global__ void device_Reduce_Speeds(bool* active, real* mass, real3* vel, real3* omega, real3* updateV, real3* updateO, uint* d_body_num, uint* counter, real3* fap) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_updates_const);
	function_Reduce_Speeds(index, active, mass, vel, omega, updateV, updateO, d_body_num, counter, fap);
}
void ChLcpSolverGPU::host_Reduce_Speeds(bool* active, real* mass, real3* vel, real3* omega, real3* updateV, real3* updateO, uint* d_body_num, uint* counter, real3* fap) {
#pragma omp parallel for schedule(guided)

	for (uint index = 0; index < number_of_updates; index++) {
		function_Reduce_Speeds(index, active, mass, vel, omega, updateV, updateO, d_body_num, counter, fap);
	}
}
__host__ __device__ void function_Integrate_Timestep(uint& index, real& step_size, bool* active, real3* acc, real4* rot, real3* vel, real3* omega, real3* pos, real3* lim) {
	real3 velocity = vel[index];

	if (active[index] == 0) {
		return;
	}

	// Do 1st order integration of quaternion position as q[t+dt] = qw_abs^(dt) * q[dt] = q[dt] * qw_local^(dt)
	// where qw^(dt) is the quaternion { cos(0.5|w|), wx/|w| sin(0.5|w|), wy/|w| sin(0.5|w|), wz/|w| sin(0.5|w|)}^dt
	// that is argument of sine and cosines are multiplied by dt.
	real3 omg = omega[index];
	real3 limits = lim[index];
	real wlen = length(omg);
//    if (limits.x == 1) {
//        real w = 2.0 * wlen;
//
//        if (w > limits.z) {
//            omg = omg * limits.z / w;
//            wlen = sqrtf(dot3(omg, omg));
//        }
//
//        real v = length(velocity);
//
//        if (v > limits.y) {
//            velocity = velocity * limits.y / v;
//        }
//
//        vel[index] = velocity;
//        omega[index] = omg;
//    }
	pos[index] = pos[index] + velocity * step_size; // Do 1st order integration of linear speeds
	real4 Rw = (fabs(wlen) > 10e-10) ? Q_from_AngAxis(step_size * wlen, omg / wlen) : R4(1, 0, 0, 0); // to avoid singularity for near zero angular speed
	Rw = normalize(Rw);
	real4 mq = mult(rot[index], Rw);
	mq = mq * rsqrtf(dot(mq, mq));
	rot[index] = mq;
	acc[index] = (velocity - acc[index]) / step_size;
}
__global__ void device_Integrate_Timestep(bool* active, real3* acc, real4* rot, real3* vel, real3* omega, real3* pos, real3* lim) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_Integrate_Timestep(index, step_size_const, active, acc, rot, vel, omega, pos, lim);
}
void ChLcpSolverGPU::host_Integrate_Timestep(bool* active, real3* acc, real4* rot, real3* vel, real3* omega, real3* pos, real3* lim) {
#pragma omp parallel for schedule(guided)

	for (uint index = 0; index < number_of_objects; index++) {
		function_Integrate_Timestep(index, step_size, active, acc, rot, vel, omega, pos, lim);
	}
}

__host__ __device__ void function_ComputeGyro(uint& index, real3* omega, real3* inertia, real3* gyro, real3* torque) {
	real3 body_inertia = inertia[index];
	body_inertia = R3(1.0 / body_inertia.x, 1.0 / body_inertia.y, 1.0 / body_inertia.z);
	real3 body_omega = omega[index];
	real3 gyr = cross(body_omega, body_inertia * body_omega);
	gyro[index] = gyr;
}

__global__ void device_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	function_ComputeGyro(index, omega, inertia, gyro, torque);
}

void ChLcpSolverGPU::host_ComputeGyro(real3* omega, real3* inertia, real3* gyro, real3* torque) {
#pragma omp parallel for schedule(guided)

	for (uint index = 0; index < number_of_objects; index++) {
		function_ComputeGyro(index, omega, inertia, gyro, torque);
	}
}

__global__ void device_Offsets(int2* ids, real4* bilaterals, uint* Body) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < number_of_contacts_const) {
		int2 temp_id = ids[index];
		Body[index] = temp_id.x;
		Body[index + number_of_contacts_const] = temp_id.y;
	}

	if (index < number_of_bilaterals_const) {
		Body[2 * number_of_contacts_const + index] = bilaterals[index].w;
		Body[2 * number_of_contacts_const + index + number_of_bilaterals_const] = bilaterals[index + number_of_bilaterals_const].w;
	}
}

void ChLcpSolverGPU::host_Offsets(int2* ids, real4* bilaterals, uint* Body) {
	for (uint index = 0; index < number_of_constraints; index++) {
		if (index < number_of_contacts) {
			int2 temp_id = ids[index];
			Body[index] = temp_id.x;
			Body[index + number_of_contacts] = temp_id.y;
		}

		if (index < number_of_bilaterals) {
			Body[2 * number_of_contacts + index] = bilaterals[index].w;
			Body[2 * number_of_contacts + index + number_of_bilaterals] = bilaterals[index + number_of_bilaterals].w;
		}
	}
}

void ChLcpSolverGPU::Preprocess(gpu_container& gpu_data) {
	gpu_data.number_of_updates = 0;
	gpu_data.device_gyr_data.resize(number_of_objects);
	custom_vector<uint> body_num;
	custom_vector<uint> update_number;
	gpu_data.device_gam_data.resize(number_of_constraints);
#ifdef SIM_ENABLE_GPU_MODE
	device_ComputeGyro CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(CASTR3(gpu_data.device_omg_data), CASTR3(gpu_data.device_inr_data), CASTR3(gpu_data.device_gyr_data),
			CASTR3(gpu_data.device_trq_data));
#else
	host_ComputeGyro(
			gpu_data.device_omg_data.data(),
			gpu_data.device_inr_data.data(),
			gpu_data.device_gyr_data.data(),
			gpu_data.device_trq_data.data());
#endif
#ifdef SIM_ENABLE_GPU_MODE
	device_addForces CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(CASTB1(gpu_data.device_active_data), CASTR1(gpu_data.device_mass_data), CASTR3(gpu_data.device_inr_data),
			CASTR3(gpu_data.device_frc_data), CASTR3(gpu_data.device_trq_data), CASTR3(gpu_data.device_vel_data), CASTR3(gpu_data.device_omg_data));
#else
	host_addForces(
			gpu_data.device_active_data.data(),
			gpu_data.device_mass_data.data(),
			gpu_data.device_inr_data.data(),
			gpu_data.device_frc_data.data(),
			gpu_data.device_trq_data.data(),
			gpu_data.device_vel_data.data(),
			gpu_data.device_omg_data.data());
#endif
	gpu_data.device_fap_data.resize(number_of_objects);
	Thrust_Fill(gpu_data.device_fap_data, R3(0));

	if (number_of_constraints > 0) {
		update_number.resize((number_of_constraints) * 2, 0);
		gpu_data.offset_counter.resize((number_of_constraints) * 2, 0);
		gpu_data.update_offset.resize((number_of_constraints) * 2, 0);
		body_num.resize((number_of_constraints) * 2, 0);
		gpu_data.device_dgm_data.resize((number_of_constraints));
		Thrust_Fill(gpu_data.device_dgm_data, 1);
		gpu_data.vel_update.resize((number_of_constraints) * 2);
		gpu_data.omg_update.resize((number_of_constraints) * 2);
#ifdef SIM_ENABLE_GPU_MODE
		device_Offsets CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(CASTI2(gpu_data.device_bids_data), CASTR4(gpu_data.device_bilateral_data), CASTU1(body_num));
#else
		host_Offsets(gpu_data.device_bids_data.data(), gpu_data.device_bilateral_data.data(), body_num.data());
#endif
		Thrust_Sequence(update_number);
		Thrust_Sequence(gpu_data.update_offset);
		Thrust_Fill(gpu_data.offset_counter, 0);
		Thrust_Sort_By_Key(body_num, update_number);
		Thrust_Sort_By_Key(update_number, gpu_data.update_offset);
		gpu_data.body_number = body_num;
		Thrust_Reduce_By_KeyB(gpu_data.number_of_updates, body_num, update_number, gpu_data.offset_counter);
//        host_vector<uint> body_num_t=body_num;
//        host_vector<uint> update_number_t=update_number;
//        host_vector<uint> offset_counter_t=gpu_data.offset_counter;
//        Thrust_Reduce_By_KeyB(gpu_data.number_of_updates, body_num_t, update_number_t, offset_counter_t);
//        body_num=body_num_t;
//        update_number=update_number_t;
//        gpu_data.offset_counter=offset_counter_t;
		Thrust_Inclusive_Scan(gpu_data.offset_counter);
	}
}

void ChLcpSolverGPU::Iterate(gpu_container& gpu_data) {
}
void ChLcpSolverGPU::Reduce(gpu_container& gpu_data) {
}
void ChLcpSolverGPU::Integrate(gpu_container& gpu_data) {
#ifdef SIM_ENABLE_GPU_MODE
	device_Integrate_Timestep CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(CASTB1(gpu_data.device_active_data), CASTR3(gpu_data.device_acc_data), CASTR4(gpu_data.device_rot_data),
			CASTR3(gpu_data.device_vel_data), CASTR3(gpu_data.device_omg_data), CASTR3(gpu_data.device_pos_data), CASTR3(gpu_data.device_lim_data));
#else
	host_Integrate_Timestep(
			gpu_data.device_active_data.data(),
			gpu_data.device_acc_data.data(),
			gpu_data.device_rot_data.data(),
			gpu_data.device_vel_data.data(),
			gpu_data.device_omg_data.data(),
			gpu_data.device_pos_data.data(),
			gpu_data.device_lim_data.data());
#endif
}
void ChLcpSolverGPU::RunTimeStep(real step, gpu_container& gpu_data) {
	lcp_omega_contact = omega;
	step_size = step;
	number_of_constraints = gpu_data.number_of_contacts + gpu_data.number_of_bilaterals;
	number_of_contacts = gpu_data.number_of_contacts;
	number_of_bilaterals = 0; ///gpu_data.number_of_bilaterals;
	number_of_objects = gpu_data.number_of_objects;
	//cout << number_of_constraints << " " << number_of_contacts << " " << number_of_bilaterals << " " << number_of_objects << endl;
#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(number_of_contacts);
	COPY_TO_CONST_MEM(number_of_bilaterals);
	COPY_TO_CONST_MEM(number_of_objects);
	COPY_TO_CONST_MEM(step_size);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_ComputeGyro), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_addForces), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Offsets), hipFuncCachePreferL1);
#else
#endif
	Preprocess(gpu_data);
	number_of_updates = gpu_data.number_of_updates;
#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(compliance);
	COPY_TO_CONST_MEM(complianceT);
	COPY_TO_CONST_MEM(alpha);
	COPY_TO_CONST_MEM(lcp_omega_bilateral);
	COPY_TO_CONST_MEM(lcp_omega_contact);
	COPY_TO_CONST_MEM(lcp_contact_factor);
	COPY_TO_CONST_MEM(number_of_updates);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_process_contacts), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Bilaterals), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Reduce_Speeds), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Integrate_Timestep), hipFuncCachePreferL1);
#else
#endif
	real old_gam = 1;
	//COMPUTE JACOBIANS

	gpu_data.device_QXYZ_data.resize(number_of_objects);
	gpu_data.device_QUVW_data.resize(number_of_objects);

	gpu_data.device_JXYZA_data.resize(number_of_contacts * 3 + number_of_constraints);
	gpu_data.device_JXYZB_data.resize(number_of_contacts * 3 + number_of_constraints);
	gpu_data.device_JUVWA_data.resize(number_of_contacts * 3 + number_of_constraints);
	gpu_data.device_JUVWB_data.resize(number_of_contacts * 3 + number_of_constraints);
#ifdef SIM_ENABLE_GPU_MODE
	device_ContactJacobians CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(CASTR3(gpu_data.device_norm_data), CASTR3(gpu_data.device_cpta_data), CASTR3(gpu_data.device_cptb_data),
			CASTI2(gpu_data.device_bids_data), CASTR4(gpu_data.device_rot_data), CASTR3(gpu_data.device_pos_data), CASTR3(gpu_data.device_JXYZA_data), CASTR3(gpu_data.device_JXYZB_data),
			CASTR3(gpu_data.device_JUVWA_data), CASTR3(gpu_data.device_JUVWB_data));

#else
	host_ContactJacobians(
			gpu_data.device_norm_data.data(),
			gpu_data.device_cpta_data.data(),
			gpu_data.device_cptb_data.data(),
			gpu_data.device_bids_data.data(),
			gpu_data.device_rot_data.data(),
			gpu_data.device_pos_data.data(),
			gpu_data.device_JXYZA_data.data(),
			gpu_data.device_JXYZB_data.data(),
			gpu_data.device_JUVWA_data.data(),
			gpu_data.device_JUVWB_data.data()
	);
#endif
cout<<max_iterations<<" "<<tolerance<<endl;
	if (number_of_constraints != 0) {
		for (iteration_number = 0; iteration_number < 500; iteration_number++) {
#ifdef SIM_ENABLE_GPU_MODE
			device_process_contacts CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(CASTR3(gpu_data.device_JXYZA_data), CASTR3(gpu_data.device_JXYZB_data), CASTR3(gpu_data.device_JUVWA_data),
					CASTR3(gpu_data.device_JUVWB_data), CASTR1(gpu_data.device_dpth_data), CASTI2(gpu_data.device_bids_data), CASTR3(gpu_data.device_gam_data), CASTR1(gpu_data.device_dgm_data),
					CASTR1(gpu_data.device_mass_data), CASTR1(gpu_data.device_fric_data), CASTR3(gpu_data.device_inr_data), CASTR4(gpu_data.device_rot_data), CASTR3(gpu_data.device_vel_data),
					CASTR3(gpu_data.device_omg_data), CASTR3(gpu_data.device_pos_data), CASTR3(gpu_data.vel_update), CASTR3(gpu_data.omg_update), CASTU1(gpu_data.update_offset));
#else
			host_process_contacts(
					gpu_data.device_JXYZA_data.data(),
					gpu_data.device_JXYZB_data.data(),
					gpu_data.device_JUVWA_data.data(),
					gpu_data.device_JUVWB_data.data(),
					gpu_data.device_dpth_data.data(),
					gpu_data.device_bids_data.data(),
					gpu_data.device_gam_data.data(),
					gpu_data.device_dgm_data.data(),
					gpu_data.device_mass_data.data(),
					gpu_data.device_fric_data.data(),
					gpu_data.device_inr_data.data(),
					gpu_data.device_rot_data.data(),
					gpu_data.device_vel_data.data(),
					gpu_data.device_omg_data.data(),
					gpu_data.device_pos_data.data(),
					gpu_data.vel_update.data(),
					gpu_data.omg_update.data(),
					gpu_data.update_offset.data());
#endif
#ifdef SIM_ENABLE_GPU_MODE
			device_Bilaterals CUDA_KERNEL_DIM(BLOCKS(number_of_bilaterals), THREADS)(CASTR4(gpu_data.device_bilateral_data), CASTR1(gpu_data.device_mass_data), CASTR3(gpu_data.device_inr_data),
					CASTR4(gpu_data.device_rot_data), CASTR3(gpu_data.device_vel_data), CASTR3(gpu_data.device_omg_data), CASTR3(gpu_data.device_pos_data), CASTR3(gpu_data.vel_update),
					CASTR3(gpu_data.omg_update), CASTU1(gpu_data.update_offset), CASTR1(gpu_data.device_dgm_data));
#else
			host_Bilaterals(
					gpu_data.device_bilateral_data.data(),
					gpu_data.device_mass_data.data(),
					gpu_data.device_inr_data.data(),
					gpu_data.device_rot_data.data(),
					gpu_data.device_vel_data.data(),
					gpu_data.device_omg_data.data(),
					gpu_data.device_pos_data.data(),
					gpu_data.vel_update.data(),
					gpu_data.omg_update.data(),
					gpu_data.update_offset.data(),
					gpu_data.device_dgm_data.data());
#endif
#ifdef SIM_ENABLE_GPU_MODE
			device_Reduce_Speeds CUDA_KERNEL_DIM(BLOCKS(number_of_updates), THREADS)(CASTB1(gpu_data.device_active_data), CASTR1(gpu_data.device_mass_data), CASTR3(gpu_data.device_vel_data),
					CASTR3(gpu_data.device_omg_data), CASTR3(gpu_data.vel_update), CASTR3(gpu_data.omg_update), CASTU1(gpu_data.body_number), CASTU1(gpu_data.offset_counter),
					CASTR3(gpu_data.device_fap_data));
#else
			host_Reduce_Speeds(
					gpu_data.device_active_data.data(),
					gpu_data.device_mass_data.data(),
					gpu_data.device_vel_data.data(),
					gpu_data.device_omg_data.data(),
					gpu_data.vel_update.data(),
					gpu_data.omg_update.data(),
					gpu_data.body_number.data(),
					gpu_data.offset_counter.data(),
					gpu_data.device_fap_data.data());
#endif

			if (tolerance != 0) {
				if (iteration_number > 20 && iteration_number % 20 == 0) {
					real gam = Max_DeltaGamma(gpu_data.device_dgm_data);

					if (fabsf(old_gam - gam) < 1e-8) {
						break;
					}

					old_gam = gam;
				}
			}
		}
	}
cout<<" "<<iteration_number<<endl;
	Integrate(gpu_data);
}
real ChLcpSolverGPU::Max_DeltaGamma(custom_vector<real>& device_dgm_data) {
	return Thrust_Max(device_dgm_data);
}
real ChLcpSolverGPU::Min_DeltaGamma(custom_vector<real>& device_dgm_data) {
	return Thrust_Min(device_dgm_data);
}
real ChLcpSolverGPU::Avg_DeltaGamma(uint number_of_constraints, custom_vector<real>& device_dgm_data) {
	real gamma = (Thrust_Total(device_dgm_data)) / real(number_of_constraints);
	//cout << gamma << endl;
		return gamma;
	}
__global__ void Compute_KE(real3* vel, real* mass, real* ke) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_objects_const);
	real3 velocity = vel[index];
	ke[index] = .5 / mass[index] * dot(velocity, velocity);
}
real ChLcpSolverGPU::Total_KineticEnergy(gpu_container& gpu_data) {
	custom_vector<real> device_ken_data;
	device_ken_data.resize(gpu_data.number_of_objects);
	Compute_KE CUDA_KERNEL_DIM(BLOCKS(gpu_data.number_of_objects), THREADS)(CASTR3(gpu_data.device_vel_data), CASTR1(gpu_data.device_mass_data), CASTR1(device_ken_data));
	return (Thrust_Total(device_ken_data));
}

