#include "hip/hip_runtime.h"
#include "ChLcpSolverGPU.h"

using namespace chrono;

//helper functions
template<class T, class U> // dot product of the first three elements of float3/float4 values
inline __host__ __device__ float dot3(const T & a, const U & b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__constant__ float lcp_omega_bilateral_const;
__constant__ float lcp_omega_contact_const;
__constant__ float lcp_contact_factor_const;
__constant__ unsigned int number_of_objects_const;
__constant__ unsigned int number_of_contacts_const;
__constant__ unsigned int number_of_bilaterals_const;
__constant__ unsigned int number_of_updates_const;
//__constant__ float force_factor_const; // usually, the step size
//__constant__ float negated_recovery_speed_const;
__constant__ float step_size_const;
__constant__ float compliance_const;
__constant__ float complianceT_const;
__constant__ float alpha_const; // [R]=alpha*[K]
////////////////////////////////////////////////////////////////////////////////////////////////////
// Creates a quaternion as a function of a vector of rotation and an angle (the vector is assumed already
// normalized, and angle is in radians).

__host__ __device__ inline float4 Quat_from_AngAxis(const float &angle, const float3 & v) {
	float sinhalf = sinf(angle * 0.5f);
	float4 quat;
	quat.x = cosf(angle * 0.5f);
	quat.y = v.x * sinhalf;
	quat.z = v.y * sinhalf;
	quat.w = v.z * sinhalf;
	return quat;
}

/// The quaternion becomes the quaternion product of the two quaternions A and B:
/// following the classic Hamilton rule:  this=AxB
/// This is the true, typical quaternion product. It is NOT commutative.

__host__ __device__ inline float4 Quaternion_Product(const float4 &qa, const float4 &qb) {
	float4 quat;
	quat.x = qa.x * qb.x - qa.y * qb.y - qa.z * qb.z - qa.w * qb.w;
	quat.y = qa.x * qb.y + qa.y * qb.x - qa.w * qb.z + qa.z * qb.w;
	quat.z = qa.x * qb.z + qa.z * qb.x + qa.w * qb.y - qa.y * qb.w;
	quat.w = qa.x * qb.w + qa.w * qb.x - qa.z * qb.y + qa.y * qb.z;
	return quat;
}
__device__ void inline Compute_Jacobian(float3 &T3, float3 &T4, float3 &T5, float4 Eb, const float3 &N, const float3 &U, const float3 &W, const float3 &pb) {

	float t1 = Eb.y * Eb.z;
	float t2 = Eb.x * Eb.w;
	float t3 = 2 * t1 - 2 * t2;
	float t4 = Eb.y * Eb.w;
	float t5 = Eb.x * Eb.z;
	float t6 = 2 * t4 + 2 * t5;
	float t7 = Eb.z * Eb.w;
	float t8 = Eb.x * Eb.y;
	float t9 = 2 * t7 - 2 * t8;
	float t10 = Eb.x * Eb.x;
	float t11 = Eb.y * Eb.y;
	float t12 = Eb.w * Eb.w;
	float t13 = Eb.z * Eb.z;
	float t14 = t10 - t11 - t13 + t12;
	float t15 = t6 * pb.x + t9 * pb.y + t14 * pb.z;
	float t16 = t10 - t11 + t13 - t12;
	t7 = 2 * t7 + 2 * t8;
	t8 = -t3 * pb.x - t16 * pb.y - t7 * pb.z;
	float t17 = t3 * t15 + t6 * t8;
	float t18 = t16 * t15 + t9 * t8;
	float t19 = t7 * t15 + t14 * t8;
	t10 = t10 + t11 - t13 - t12;
	t11 = -t15;
	t1 = 2 * t1 + 2 * t2;
	t2 = 2 * t4 - 2 * t5;
	t4 = t10 * pb.x + t1 * pb.y + t2 * pb.z;
	t5 = t10 * t11 + t6 * t4;
	t6 = t1 * t11 + t9 * t4;
	t9 = t2 * t11 + t14 * t4;
	t8 = -t8;
	t4 = -t4;
	t3 = t10 * t8 + t3 * t4;
	t1 = t1 * t8 + t16 * t4;
	t2 = t2 * t8 + t7 * t4;
	T3.x = N.x * t17 + N.y * t18 + N.z * t19;
	T3.y = N.x * t5 + N.y * t6 + N.z * t9;
	T3.z = N.x * t3 + N.y * t1 + N.z * t2;
	T4.x = U.x * t17 + U.y * t18 + U.z * t19;
	T4.y = U.x * t5 + U.y * t6 + U.z * t9;
	T4.z = U.x * t3 + U.y * t1 + U.z * t2;
	T5.x = W.x * t17 + W.y * t18 + W.z * t19;
	T5.y = W.x * t5 + W.y * t6 + W.z * t9;
	T5.z = W.x * t3 + W.y * t1 + W.z * t2;
}

// 	Kernel for a single iteration of the LCP over all contacts
//   	Version 2.0 - Tasora
//	Version 2.2- Hammad (optimized, cleaned etc)
__global__ void LCP_Iteration_Contacts(
		float3* norm,
		float3* ptA,
		float3* ptB,
		float* contactDepth,
		int2* ids,
		float3* G,
		float* dG,
		float3* aux,
		float3* inertia,
		float4* rot,
		float3* vel,
		float3* omega,
		float3* pos,
		float4* updateV,
		float4* updateO,
		uint* offset) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_contacts_const) {
		return;
	}

	float3 gamma, T3, T4, T5, T6, T7, T8;
	int2 temp_id = ids[i];
	float depth = -fabs(contactDepth[i]);

	int B1_i = temp_id.x;
	int B2_i = temp_id.y;

	float3 N = norm[i]; //assume: normalized, and if depth=0 norm=(1,0,0)

	float3 W = fabs(N); //Gramm Schmidt; work with the global axis that is "most perpendicular" to the contact normal vector;effectively this means looking for the smallest component (in abs) and using the corresponding direction to carry out cross product.
	float3 U = F3(0, N.z, -N.y); //it turns out that X axis is closest to being perpendicular to contact vector;
	{
		if (W.x > W.y) {
			U = F3(-N.z, 0, N.x);
		} //it turns out that Y axis is closest to being perpendicular to contact vector;
		if (W.y > W.z) {
			U = F3(N.y, -N.x, 0);
		} //it turns out that Z axis is closest to being perpendicular to contact vector;
		U = normalize(U); //normalize the local contact Y,Z axis
		W = cross(N, U); //carry out the last cross product to find out the contact local Z axis : multiply the contact normal by the local Y component
	}

	{
		float3 sbar = ptA[i] - pos[B1_i]; //Contact Point on A - Position of A
		float4 E1 = rot[B1_i]; //bring in the Euler parameters associated with body 1;
		Compute_Jacobian(T3, T4, T5, E1, N, U, W, sbar); //A_i,p'*A_A*(sbar~_i,A)
	}
	{
		float3 sbar = ptB[i] - pos[B2_i]; //Contact Point on B - Position of B
		float4 E2 = rot[B2_i]; //bring in the Euler parameters associated with body 2;
		Compute_Jacobian(T6, T7, T8, E2, N, U, W, sbar); //A_i,p'*A_B*(sbar~_i,B)
	}

	T6 = -T6;
	T7 = -T7;
	T8 = -T8;

//	float normV = dot(N, ((V2 - V1)));
//	normV = (normV > 0) ? 0 : normV;
	float cfm = 0, cfmT = 0;
//	if (compliance_const) {
//		float h = step_size_const;
//		float inv_hpa = 1.0 / (h + alpha_const); // 1/(h+a)
//		float inv_hhpa = 1.0 / (h * (h + alpha_const)); // 1/(h*(h+a))
//		bi = inv_hpa * depth;
//		cfm = inv_hhpa * compliance_const;
//		cfmT = inv_hhpa * complianceT_const;
//	} else {
	float bi = fmaxf((depth / (step_size_const)), -lcp_contact_factor_const);
//	}

	//c_i = [Cq_i]*q + b_i + cfm_i*l_i
	float3 W1 = omega[B1_i];
	float3 W2 = omega[B2_i];
	float3 V1 = vel[B1_i];
	float3 V2 = vel[B2_i];

	gamma.x = dot(T3, W1) - dot(N, V1) + dot(T6, W2) + dot(N, V2) + bi /*+ cfm * gamma_old.x*/; //+bi
	gamma.y = dot(T4, W1) - dot(U, V1) + dot(T7, W2) + dot(U, V2) /*+ cfmT * gamma_old.y*/;
	gamma.z = dot(T5, W1) - dot(W, V1) + dot(T8, W2) + dot(W, V2) /*+ cfmT * gamma_old.z*/;
	float3 In1 = inertia[B1_i]; // bring in the inertia attributes; to be used to compute \eta
	float3 In2 = inertia[B2_i]; // bring in the inertia attributes; to be used to compute \eta

	float3 aux1 = aux[B1_i];
	float3 aux2 = aux[B2_i];

	float eta = dot(T3 * T3, In1) + dot(T4 * T4, In1) + dot(T5 * T5, In1); // update expression of eta
	eta += dot(T6 * T6, In2) + dot(T7 * T7, In2) + dot(T8 * T8, In2);
	eta += (dot(N, N) + dot(U, U) + dot(W, W)) * (aux1.z + aux2.z); // multiply by inverse of mass matrix of B1 and B2, add contribution from mass and matrix A_c.
	eta = lcp_omega_contact_const / eta; // final value of eta
	float3 gamma_old = G[i];
	gamma = eta * gamma; // perform gamma *= omega*eta
	gamma = gamma_old - gamma; // perform gamma = gamma_old - gamma ;  in place.
	/// ---- perform projection of 'a8' onto friction cone  --------
	float f_tang = sqrtf(gamma.y * gamma.y + gamma.z * gamma.z);
	float mu = (aux1.y + aux2.y) * .5f;

	if (f_tang > (mu * gamma.x)) { // inside upper cone? keep untouched!
		if ((f_tang) < -(1.0 / mu) * gamma.x || (fabsf(gamma.x) < 0)) { // inside lower cone? reset  normal,u,v to zero!
			gamma = F3(0);
		} else { // remaining case: project orthogonally to generator segment of upper cone
			gamma.x = (f_tang * mu + gamma.x) / (mu * mu + 1);
			float tproj_div_t = (gamma.x * mu) / f_tang; //  reg = tproj_div_t
			gamma.y *= tproj_div_t;
			gamma.z *= tproj_div_t;
		}
	} else if (mu == 0) {
		gamma.y = gamma.z = 0;
	}

	G[i] = gamma; // store gamma_new
	gamma -= gamma_old; // compute delta_gamma = gamma_new - gamma_old   = delta_gamma.
	dG[i] = length(gamma);
	float3 vB = N * gamma.x + U * gamma.y + W * gamma.z;
	int offset1 = offset[i];
	int offset2 = offset[i + number_of_contacts_const];
	updateV[offset1] = F4(-vB * aux1.z); // compute and store dv1
	updateO[offset1] = F4((T3 * gamma.x + T4 * gamma.y + T5 * gamma.z) * In1); // compute dw1 =  Inert.1' * J1w^ * deltagamma  and store  dw1
	updateV[offset2] = F4(vB * aux2.z); // compute and store dv2
	updateO[offset2] = F4((T6 * gamma.x + T7 * gamma.y + T8 * gamma.z) * In2); // compute dw2 =  Inert.2' * J2w^ * deltagamma  and store  dw2

}
///////////////////////////////////////////////////////////////////////////////////
// Kernel for a single iteration of the LCP over all scalar bilateral contacts
// (a bit similar to the ChKernelLCPiteration above, but without projection etc.)
// Version 2.0 - Tasora
//
__global__ void LCP_Iteration_Bilaterals(
		CH_REALNUMBER4* bilaterals,
		float3* aux,
		float3* inertia,
		float4* rot,
		float3* vel,
		float3* omega,
		float3* pos,
		float4* updateV,
		float4* updateO,
		uint* offset,
		float * dG) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_bilaterals_const) {
		return;
	}
	float4 vA;
	float3 vB;
	float gamma_new = 0, gamma_old = 0;
	int B1_index = 0, B2_index = 0;
	B1_index = bilaterals[i].w;
	B2_index = bilaterals[i + number_of_bilaterals_const].w;
	float3 aux1 = aux[B1_index];
	float3 aux2 = aux[B2_index];
	// ---- perform   gamma_new = ([J1 J2] {v1 | v2}^ + b)
	{
		vA = bilaterals[i]; // line 0
		vB = vel[B1_index]; // v1
		gamma_new += dot3(vA, vB);

		vA = bilaterals[i + 2 * number_of_bilaterals_const]; // line 2
		vB = omega[B1_index]; // w1
		gamma_new += dot3(vA, vB);
	}
	{
		vA = bilaterals[i + number_of_bilaterals_const]; // line 1
		vB = vel[B2_index]; // v2
		gamma_new += dot3(vA, vB);

		vA = bilaterals[i + 3 * number_of_bilaterals_const]; // line 3
		vB = omega[B2_index]; // w2
		gamma_new += dot3(vA, vB);
	}
	vA = bilaterals[i + 4 * number_of_bilaterals_const]; // line 4   (eta, b, gamma, 0)
	gamma_new += vA.y; // add known term     + b
	gamma_old = vA.z; // old gamma
	/// ---- perform gamma_new *= omega/g_i
	gamma_new *= lcp_omega_bilateral_const; // lcp_omega_const is in constant memory
	gamma_new *= vA.x; // eta = 1/g_i;
	/// ---- perform gamma_new = gamma_old - gamma_new ; in place.
	gamma_new = gamma_old - gamma_new;
	/// ---- perform projection of 'a' (only if simple unilateral behavior C>0 is requested)
	if (vA.w && gamma_new < 0.) {
		gamma_new = 0.;
	}
	// ----- store gamma_new
	vA.z = gamma_new;
	bilaterals[i + 4 * number_of_bilaterals_const] = vA;
	/// ---- compute delta in multipliers: gamma_new = gamma_new - gamma_old   = delta_gamma    , in place.
	gamma_new -= gamma_old;
	//dG[number_of_contacts_const + i] = (gamma_new);
	/// ---- compute dv1 =  invInert.18 * J1^ * deltagamma
	vB = inertia[B1_index]; // iJ iJ iJ im
	vA = (bilaterals[i]) * aux1.z * gamma_new; // line 0: J1(x)
	int offset1 = offset[2 * number_of_contacts_const + i];
	int offset2 = offset[2 * number_of_contacts_const + i + number_of_bilaterals_const];
	updateV[offset1] = (vA); //  ---> store  v1 vel. in reduction buffer
	updateO[offset1] = F4(F3(bilaterals[i + 2 * number_of_bilaterals_const]) * vB * gamma_new); // line 2:  J1(w)// ---> store  w1 vel. in reduction buffer
	vB = inertia[B2_index]; // iJ iJ iJ im
	vA = (bilaterals[i + number_of_bilaterals_const]) * aux2.z * gamma_new; // line 1: J2(x)
	updateV[offset2] = (vA); //  ---> store  v2 vel. in reduction buffer
	updateO[offset2] = F4(F3(bilaterals[i + 3 * number_of_bilaterals_const]) * vB * gamma_new); // line 3:  J2(w)// ---> store  w2 vel. in reduction buffer
}

__device__ __host__ inline float4 computeRot_dt(float3 & omega, float4 &rot) {
	return mult(F4(0, omega.x, omega.y, omega.z), rot) * .5;
}
__device__ __host__ float3 RelPoint_AbsSpeed(float3 & vel, float3 & omega, float4 & rot, float3 &point) {
	float4 q = mult(computeRot_dt(omega, rot), mult(F4(0, point.x, point.y, point.z), inv(rot)));
	return vel + ((F3(q.y, q.z, q.w)) * 2);
}

__global__ void DEM_Contacts(
		float3* norm,
		float3* ptA,
		float3* ptB,
		float* contactDepth,
		int2* ids,
		float3* aux,
		float3* inertia,
		float4* rot,
		float3* vel,
		float3* omega,
		float3* pos,
		float3* updateV,
		float3* updateO,
		uint* offset) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_contacts_const) {
		return;
	}
	float mkn = 3924, mgn = 420, mgt = 420, mkt = 2.0f / 7.0f * 3924;
	//long long id=ids[i];
	int2 temp_id = ids[i];
	int B1_i = int(temp_id.x);
	int B2_i = int(temp_id.y);

	float3 vN = norm[i]; //normal
	float3 pA = ptA[i];
	float3 pB = ptB[i];
	float3 B1 = vel[B1_i]; //vel B1
	float3 B2 = vel[B2_i]; //vel B2
	float3 oA = omega[B1_i];
	float3 oB = omega[B2_i];
	float4 rotA = rot[B1_i];
	float4 rotB = rot[B2_i];
	float3 aux1 = aux[B1_i];
	float3 aux2 = aux[B2_i];
	float3 posA = pos[B1_i];
	float3 posB = pos[B2_i];
	float3 f_n = mkn * -fabs(contactDepth[i]) * vN;
	float3 local_pA = quatRotate(pA - posA, inv(rotA));
	float3 local_pB = quatRotate(pB - posB, inv(rotB));

	float3 v_BA = (RelPoint_AbsSpeed(B2, oB, rotB, local_pB)) - (RelPoint_AbsSpeed(B1, oA, rotA, local_pA));
	float3 v_n = normalize(dot(v_BA, vN) * vN);
	float m_eff = (1.0 / aux1.z) * (1.0 / aux2.z) / (1.0 / aux1.z + 1.0 / aux2.z);
	f_n += mgn * m_eff * v_n;

	float mu = (aux1.y + aux2.y) * .5;
	float3 v_t = v_BA - v_n;

	float3 f_t = (mgt * m_eff * v_t) + (mkt * (v_t * step_size_const));

	if (length(f_t) > mu * length(f_n)) {
		f_t *= mu * length(f_n) / length(f_t);
	}

	float3 f_r = f_n + f_t;

	int offset1 = offset[i];
	int offset2 = offset[i + number_of_contacts_const];

	float3 force1_loc = quatRotate(f_r, inv(rotA));
	float3 force2_loc = quatRotate(f_r, inv(rotB));

	float3 trq1 = cross(local_pA, force1_loc);
	float3 trq2 = cross(local_pB, -force2_loc);

	f_r *= step_size_const;

	updateV[offset1] = (f_r) * aux1.z;
	updateV[offset2] = (f_r) * -aux2.z;

	updateO[offset1] = (trq1 * step_size_const) * (inertia[B1_i]);
	updateO[offset2] = (trq2 * step_size_const) * (inertia[B2_i]);
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__global__ void ChKernelLCPaddForces(float3* aux, float3* inertia, float3* forces, float3* torques, float3* vel, float3* omega) {
	// Compute the i values used to access data inside the large
	// array using pointer arithmetic.
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < number_of_objects_const) {
		float3 temp_aux = aux[i];
		if (temp_aux.x != 0) {
			float3 mF, minvMasses = inertia[i];
			// v += m_inv * h * f
			mF = forces[i]; // vector with f (force)
			//mF *= 1;//step_size_const;
			mF *= temp_aux.z;
			vel[i] += mF;
			// w += J_inv * h * c
			mF = torques[i]; // vector with f (torque)
			//mF *= 1;//step_size_const;
			mF.x *= minvMasses.x;
			mF.y *= minvMasses.y;
			mF.z *= minvMasses.z;
			omega[i] += mF;
		}
	}
}
////////////////////////////////////////////////////////////////////////////////////////////////////
// Updates the speeds in the body buffer with values accumulated in the
// reduction buffer:   V_new = V_old + delta_speeds

__global__ void LCP_Reduce_Speeds(float3* aux, float3* vel, float3* omega, float4* updateV, float4* updateO, uint* d_body_num, uint* counter, float3* fap) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_updates_const) {
		return;
	}
	int start = (i == 0) ? 0 : counter[i - 1], end = counter[i];
	int id = d_body_num[end - 1], j;
	float3 auxd = aux[id];
	if (auxd.x == 0) {
		return;
	}
	float4 mUpdateV = F4(0);
	float4 mUpdateO = F4(0);
	for (j = 0; j < end - start; j++) {
		mUpdateV += updateV[j + start];
		mUpdateO += updateO[j + start];
	}
	//fap[id] += (mUpdateV / auxd.z) / step_size_const;
	vel[id] += F3(mUpdateV);
	omega[id] += F3(mUpdateO);

}
//  Kernel for performing the time step integration (with 1st o;rder Eulero)
//  on the body data stream.
//
//  number of registers: 12 (suggested 320 threads/block)

__global__ void LCP_Integrate_Timestep(float3* aux, float3* acc, float4* rot, float3* vel, float3* omega, float3* pos, float3* lim) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_objects_const) {
		return;
	}
	float3 velocity = vel[i];
	float3 aux1 = aux[i];
	if (aux1.x == 0) {
		return;
	}

	// Do 1st order integration of quaternion position as q[t+dt] = qw_abs^(dt) * q[dt] = q[dt] * qw_local^(dt)
	// where qw^(dt) is the quaternion { cos(0.5|w|), wx/|w| sin(0.5|w|), wy/|w| sin(0.5|w|), wz/|w| sin(0.5|w|)}^dt
	// that is argument of sine and cosines are multiplied by dt.
	float3 omg = omega[i];

	float3 limits = lim[i];
	float wlen = length(omg);

	if (limits.x == 1) {
		float w = 2.0 * wlen;
		if (w > limits.z) {
			omg *= limits.z / w;
			wlen = sqrtf(dot3(omg, omg));
		}

		float v = length(velocity);
		if (v > limits.y) {
			velocity *= limits.y / v;
		}
		vel[i] = velocity;
		omega[i] = omg;
	}
	pos[i] = pos[i] + velocity * step_size_const; // Do 1st order integration of linear speeds

	float4 Rw = (fabs(wlen) > 10e-10) ? Quat_from_AngAxis(step_size_const * wlen, omg / wlen) : F4(1., 0, 0, 0); // to avoid singularity for near zero angular speed

	float4 mq = mult(rot[i], Rw);
	mq *= rsqrtf(dot(mq, mq));
	rot[i] = mq;
	acc[i] = (velocity - acc[i]) / step_size_const;
}
__global__ void LCP_ComputeGyro(float3* omega, float3* inertia, float3* gyro, float3* torque) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_objects_const) {
		return;
	}
	float3 body_inertia = inertia[i];
	body_inertia = F3(1 / body_inertia.x, 1 / body_inertia.y, 1 / body_inertia.z);
	float3 body_omega = omega[i];
	float3 gyr = cross(body_omega, body_inertia * body_omega);
	gyro[i] = gyr;
}

__global__ void ChKernelOffsets(int2* ids, CH_REALNUMBER4* bilaterals, uint* Body) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < number_of_contacts_const) {
		int2 temp_id = ids[i];
		Body[i] = temp_id.x;
		Body[i + number_of_contacts_const] = temp_id.y;
	}
	if (i < number_of_bilaterals_const) {
		Body[2 * number_of_contacts_const + i] = bilaterals[i].w;
		Body[2 * number_of_contacts_const + i + number_of_bilaterals_const] = bilaterals[i + number_of_bilaterals_const].w;
	}
}

void ChLcpSolverGPU::WarmContact(const int & i) {

}

void ChLcpSolverGPU::Preprocess(gpu_container & gpu_data) {
	gpu_data.number_of_updates = 0;
	gpu_data.device_gyr_data.resize(number_of_objects);
	thrust::device_vector<uint> body_num;
	thrust::device_vector<uint> update_number;
	LCP_ComputeGyro CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
			CASTF3(gpu_data.device_omg_data),
			CASTF3(gpu_data.device_inr_data),
			CASTF3(gpu_data.device_gyr_data),
			CASTF3(gpu_data.device_trq_data));
	ChKernelLCPaddForces CUDA_KERNEL_DIM(BLOCKS(number_of_objects), THREADS)(
			CASTF3(gpu_data.device_aux_data),
			CASTF3(gpu_data.device_inr_data),
			CASTF3(gpu_data.device_frc_data),
			CASTF3(gpu_data.device_trq_data),
			CASTF3(gpu_data.device_vel_data),
			CASTF3(gpu_data.device_omg_data));
	gpu_data.device_fap_data.resize(number_of_objects);
	Thrust_Fill(gpu_data.device_fap_data, F3(0));

	if (number_of_constraints > 0) {

		update_number.resize((number_of_constraints) * 2, 0);
		gpu_data.offset_counter.resize((number_of_constraints) * 2, 0);
		gpu_data.update_offset.resize((number_of_constraints) * 2, 0);
		body_num.resize((number_of_constraints) * 2, 0);
		gpu_data.device_dgm_data.resize((number_of_constraints));

		Thrust_Fill(gpu_data.device_dgm_data, 1);
		gpu_data.vel_update.resize((number_of_constraints) * 2);
		gpu_data.omg_update.resize((number_of_constraints) * 2);

		ChKernelOffsets CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(CASTI2(gpu_data.device_bids_data), CASTF4(gpu_data.device_bilateral_data), CASTU1(body_num));

		Thrust_Sequence(update_number);
		Thrust_Sequence(gpu_data.update_offset);
		Thrust_Fill(gpu_data.offset_counter, 0);
		Thrust_Sort_By_Key(body_num, update_number);
		Thrust_Sort_By_Key(update_number, gpu_data.update_offset);
		gpu_data.body_number = body_num;
		Thrust_Reduce_By_KeyB(gpu_data.number_of_updates, body_num, update_number, gpu_data.offset_counter);
		Thrust_Inclusive_Scan(gpu_data.offset_counter);

	}
}

void ChLcpSolverGPU::Iterate(gpu_container & gpu_data) {

}
void ChLcpSolverGPU::Reduce(gpu_container & gpu_data) {

}
void ChLcpSolverGPU::Integrate(gpu_container & gpu_data) {
	LCP_Integrate_Timestep CUDA_KERNEL_DIM( BLOCKS(number_of_objects), THREADS)(
			CASTF3(gpu_data.device_aux_data),
			CASTF3(gpu_data.device_acc_data),
			CASTF4(gpu_data.device_rot_data),
			CASTF3(gpu_data.device_vel_data),
			CASTF3(gpu_data.device_omg_data),
			CASTF3(gpu_data.device_pos_data),
			CASTF3(gpu_data.device_lim_data));
}
void ChLcpSolverGPU::RunTimeStep(float step, gpu_container & gpu_data) {

	lcp_omega_contact = omega;
	step_size = step;
	number_of_constraints = gpu_data.number_of_contacts + gpu_data.number_of_bilaterals;
	number_of_contacts = gpu_data.number_of_contacts;
	number_of_bilaterals = gpu_data.number_of_bilaterals;
	number_of_objects = gpu_data.number_of_objects;

	COPY_TO_CONST_MEM(number_of_contacts);
	COPY_TO_CONST_MEM(number_of_bilaterals);
	COPY_TO_CONST_MEM(number_of_objects);
	COPY_TO_CONST_MEM(step_size);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_ComputeGyro), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ChKernelLCPaddForces), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ChKernelOffsets), hipFuncCachePreferL1);

	Preprocess(gpu_data);
	number_of_updates = gpu_data.number_of_updates;

	COPY_TO_CONST_MEM(compliance);
	COPY_TO_CONST_MEM(complianceT);
	COPY_TO_CONST_MEM(alpha);
	COPY_TO_CONST_MEM(lcp_omega_bilateral);
	COPY_TO_CONST_MEM(lcp_omega_contact);
	COPY_TO_CONST_MEM(lcp_contact_factor);
	COPY_TO_CONST_MEM(number_of_updates);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Iteration_Contacts), hipFuncCachePreferL1);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Iteration_Bilaterals), hipFuncCachePreferL1);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Reduce_Speeds), hipFuncCachePreferL1);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(LCP_Integrate_Timestep), hipFuncCachePreferL1);

	if (number_of_constraints != 0) {
		for (iteration_number = 0; iteration_number < max_iterations; iteration_number++) {
			LCP_Iteration_Contacts CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(
					CASTF3(gpu_data.device_norm_data),
					CASTF3(gpu_data.device_cpta_data),
					CASTF3(gpu_data.device_cptb_data),
					CASTF1(gpu_data.device_dpth_data),
					CASTI2(gpu_data.device_bids_data),
					CASTF3(gpu_data.device_gam_data),
					CASTF1(gpu_data.device_dgm_data),
					CASTF3(gpu_data.device_aux_data),
					CASTF3(gpu_data.device_inr_data),
					CASTF4(gpu_data.device_rot_data),
					CASTF3(gpu_data.device_vel_data),
					CASTF3(gpu_data.device_omg_data),
					CASTF3(gpu_data.device_pos_data),
					CASTF4(gpu_data.vel_update),
					CASTF4(gpu_data.omg_update),
					CASTU1(gpu_data.update_offset));

			LCP_Iteration_Bilaterals CUDA_KERNEL_DIM(BLOCKS(number_of_bilaterals), THREADS)(
					CASTF4(gpu_data.device_bilateral_data),
					CASTF3(gpu_data.device_aux_data),
					CASTF3(gpu_data.device_inr_data),
					CASTF4(gpu_data.device_rot_data),
					CASTF3(gpu_data.device_vel_data),
					CASTF3(gpu_data.device_omg_data),
					CASTF3(gpu_data.device_pos_data),
					CASTF4(gpu_data.vel_update),
					CASTF4(gpu_data.omg_update),
					CASTU1(gpu_data.update_offset),
					CASTF1(gpu_data.device_dgm_data));

			LCP_Reduce_Speeds CUDA_KERNEL_DIM(BLOCKS( number_of_updates), THREADS)(
					CASTF3(gpu_data.device_aux_data),
					CASTF3(gpu_data.device_vel_data),
					CASTF3(gpu_data.device_omg_data),
					CASTF4(gpu_data.vel_update),
					CASTF4(gpu_data.omg_update),
					CASTU1(gpu_data.body_number),
					CASTU1(gpu_data.offset_counter),
					CASTF3(gpu_data.device_fap_data));

			if (tolerance != 0) {
				if (iteration_number > 5 && iteration_number % 5 == 0) {
					if (Max_DeltaGamma(gpu_data.device_dgm_data) < tolerance) {
						break;
					}
				}
			}
		}
	}

	Integrate(gpu_data);
}
float ChLcpSolverGPU::Max_DeltaGamma(device_vector<float> &device_dgm_data) {
	return Thrust_Max(device_dgm_data);
}
float ChLcpSolverGPU::Min_DeltaGamma(device_vector<float> &device_dgm_data) {
	return Thrust_Min(device_dgm_data);
}
float ChLcpSolverGPU::Avg_DeltaGamma(uint number_of_constraints, device_vector<float> &device_dgm_data) {

	float gamma = (Thrust_Total(device_dgm_data)) / float(number_of_constraints);
	//cout << gamma << endl;
	return gamma;
}
__global__ void Compute_KE(float3* vel, float3* aux, float* ke) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_objects_const) {
		return;
	}
	float3 velocity = vel[i];
	ke[i] = .5 / aux[i].z * dot(velocity, velocity);
}
float ChLcpSolverGPU::Total_KineticEnergy(gpu_container & gpu_data) {

	thrust::device_vector<float> device_ken_data;
	device_ken_data.resize(gpu_data.number_of_objects);
	Compute_KE CUDA_KERNEL_DIM(BLOCKS(gpu_data.number_of_objects), THREADS)(CASTF3(gpu_data.device_vel_data), CASTF3(gpu_data.device_aux_data), CASTF1(device_ken_data));
	return (Thrust_Total(device_ken_data));
}

