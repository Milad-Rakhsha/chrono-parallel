#include "hip/hip_runtime.h"
#include "ChSolverGPU.cuh"
using namespace chrono;

__constant__ uint number_of_constraints_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ real step_size_const;
__constant__ real alpha_const;
__constant__ real compliance_const;
__constant__ real inv_hpa_const;
__constant__ real inv_hhpa_const;
__constant__ real contact_recovery_speed_const;

__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__host__ __device__ void function_Project(uint &index, uint number_of_contacts, int2 *ids, real *fric, real *gam) {
	int2 body_id = ids[index];
	real3 gamma;
	gamma.x = gam[index + number_of_contacts * 0];
	gamma.y = gam[index + number_of_contacts * 1];
	gamma.z = gam[index + number_of_contacts * 2];
	real f_tang = sqrt(gamma.y * gamma.y + gamma.z * gamma.z);
	real mu = (fric[body_id.x] == 0 || fric[body_id.y] == 0) ? 0 : (fric[body_id.x] + fric[body_id.y]) * .5;
	if (mu == 0) {
		gamma.x = gamma.x < 0 ? 0 : gamma.x;
		gamma.y = gamma.z = 0;

		gam[index + number_of_contacts * 0] = gamma.x;
		gam[index + number_of_contacts * 1] = gamma.y;
		gam[index + number_of_contacts * 2] = gamma.z;

		return;

	}
	// inside upper cone? keep untouched!
	if (f_tang < (mu * gamma.x)) {
		return;
	}

	// inside lower cone? reset  normal,u,v to zero!

	if ((f_tang) < -(1.0 / mu) * gamma.x || (fabs(gamma.x) < 10e-15)) {
		gamma = R3(0);
		gam[index + number_of_contacts * 0] = gamma.x;
		gam[index + number_of_contacts * 1] = gamma.y;
		gam[index + number_of_contacts * 2] = gamma.z;

		return;
	}

	// remaining case: project orthogonally to generator segment of upper cone

	gamma.x = (f_tang * mu + gamma.x) / (mu * mu + 1);
	real tproj_div_t = (gamma.x * mu) / f_tang;
	gamma.y *= tproj_div_t;
	gamma.z *= tproj_div_t;

	gam[index + number_of_contacts * 0] = gamma.x;
	gam[index + number_of_contacts * 1] = gamma.y;
	gam[index + number_of_contacts * 2] = gamma.z;
}

__global__ void device_Project(int2 *ids, real *friction, real *gamma) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
	function_Project(index, number_of_contacts_const, ids, friction, gamma);
}

void ChSolverGPU::host_Project(int2 *ids, real *friction, real *gamma) {
	for (uint index = 0; index < number_of_contacts; index++) {
		function_Project(index, number_of_contacts, ids, friction, gamma);
	}
}
void ChSolverGPU::Project(custom_vector<real> & gamma) {
	timer_project.start();

#ifdef SIM_ENABLE_GPU_MODE
		device_Project CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(
				CASTI2(gpu_data->device_bids_data),
				CASTR1(gpu_data->device_fric_data),
				CASTR1(gamma));
#else
		host_Project (
				gpu_data->device_bids_data.data(),
				gpu_data->device_fric_data.data(),
				gamma.data());
#endif
		timer_project.stop();
		time_project += timer_project();
	}

__host__ __device__ void function_Reduce_Shur(uint& index, bool* active, real3* QXYZ, real3* QUVW, real *inv_mass, real3 *inv_inertia, real3* updateQXYZ, real3* updateQUVW, uint* d_body_num,
		uint* counter) {
	int start = (index == 0) ? 0 : counter[index - 1], end = counter[index];
	int id = d_body_num[end - 1], j;

	if (active[id] == 0) {
		return;
	}
	real3 mUpdateV = R3(0);
	real3 mUpdateO = R3(0);

	for (j = 0; j < end - start; j++) {
		mUpdateV += updateQXYZ[j + start];
		mUpdateO += updateQUVW[j + start];
	}
	QXYZ[id] = mUpdateV * inv_mass[id];
	QUVW[id] = mUpdateO * inv_inertia[id];
}

void ChSolverGPU::host_Reduce_Shur(bool* active, real3* QXYZ, real3* QUVW, real *inv_mass, real3 *inv_inertia, real3* updateQXYZ, real3* updateQUVW, uint* d_body_num, uint* counter) {
#pragma omp parallel for
	for (uint index = 0; index < number_of_updates; index++) {
		function_Reduce_Shur(index, active, QXYZ, QUVW, inv_mass, inv_inertia, updateQXYZ, updateQUVW, d_body_num, counter);
	}
}

__global__ void device_shurA(int2 *ids, bool *active, real *inv_mass, real3 *inv_inertia, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real *gamma, real3 *QXYZ, real3 *QUVW) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_constraints_const);
}

void ChSolverGPU::host_shurA(int2 *ids, bool *active, real *inv_mass, real3 *inv_inertia, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real *gamma, real3 *updateV, real3 *updateO,
		real3* QXYZ, real3* QUVW, uint* offset) {
#pragma omp parallel for
	for (int index = 0; index < number_of_contacts; index++) {
		real3 gam;
		gam.x = gamma[index + number_of_contacts * 0];
		gam.y = gamma[index + number_of_contacts * 1];
		gam.z = gamma[index + number_of_contacts * 2];
		uint b1 = ids[index].x;

		int offset1 = offset[index];
		int offset2 = offset[index + number_of_contacts];

		if (active[b1] != 0) {
			updateV[offset1] = JXYZA[index + number_of_contacts * 0] * gam.x + JXYZA[index + number_of_contacts * 1] * gam.y + JXYZA[index + number_of_contacts * 2] * gam.z;
			updateO[offset1] = JUVWA[index + number_of_contacts * 0] * gam.x + JUVWA[index + number_of_contacts * 1] * gam.y + JUVWA[index + number_of_contacts * 2] * gam.z;
		}
		uint b2 = ids[index].y;
		if (active[b2] != 0) {
			updateV[offset2] = JXYZB[index + number_of_contacts * 0] * gam.x + JXYZB[index + number_of_contacts * 1] * gam.y + JXYZB[index + number_of_contacts * 2] * gam.z;
			updateO[offset2] = JUVWB[index + number_of_contacts * 0] * gam.x + JUVWB[index + number_of_contacts * 1] * gam.y + JUVWB[index + number_of_contacts * 2] * gam.z;
		}
	}
//#pragma omp parallel for
//	for (int index = 0; index < number_of_bilaterals; index++) {
//		real gam;
//		gam = gamma[index + number_of_contacts * 3];
//		uint b1 = ids[index].x;
//
//		int offset1 = offset[2 * number_of_contacts + index];
//		int offset2 = offset[2 * number_of_contacts + index + number_of_bilaterals];
//
//		if (active[b1] != 0) {
//			updateV[offset1] = JXYZA[index + number_of_contacts * 0] * gam.x + JXYZA[index + number_of_contacts * 1] * gam.y + JXYZA[index + number_of_contacts * 2] * gam.z;
//			updateO[offset1] = JUVWA[index + number_of_contacts * 0] * gam.x + JUVWA[index + number_of_contacts * 1] * gam.y + JUVWA[index + number_of_contacts * 2] * gam.z;
//		}
//		uint b2 = ids[index].y;
//		if (active[b2] != 0) {
//			updateV[offset2] = JXYZB[index + number_of_contacts * 0] * gam.x + JXYZB[index + number_of_contacts * 1] * gam.y + JXYZB[index + number_of_contacts * 2] * gam.z;
//			updateO[offset2] = JUVWB[index + number_of_contacts * 0] * gam.x + JUVWB[index + number_of_contacts * 1] * gam.y + JUVWB[index + number_of_contacts * 2] * gam.z;
//		}
//	}
}
void ChSolverGPU::shurA(custom_vector<real> &x) {

#ifdef SIM_ENABLE_GPU_MODE
		device_shurA CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(
				CASTI2(temp_bids),
				CASTB1(gpu_data->device_active_data),
				CASTR1(gpu_data->device_mass_data),
				CASTR3(gpu_data->device_inr_data),
				CASTR3(gpu_data->device_JXYZA_data),
				CASTR3(gpu_data->device_JXYZB_data),
				CASTR3(gpu_data->device_JUVWA_data),
				CASTR3(gpu_data->device_JUVWB_data),
				CASTR1(x),
				CASTR3(gpu_data->device_QXYZ_data),
				CASTR3(gpu_data->device_QUVW_data));
		hipDeviceSynchronize();
#else
		host_shurA(
				temp_bids.data(),
				gpu_data->device_active_data.data(),
				gpu_data->device_mass_data.data(),
				gpu_data->device_inr_data.data(),
				gpu_data->device_JXYZA_data.data(),
				gpu_data->device_JXYZB_data.data(),
				gpu_data->device_JUVWA_data.data(),
				gpu_data->device_JUVWB_data.data(),
				x.data(),
				gpu_data->vel_update.data(),
				gpu_data->omg_update.data(),
				gpu_data->device_QXYZ_data.data(),
				gpu_data->device_QUVW_data.data(),
				gpu_data->update_offset.data());
#endif

		host_Reduce_Shur(
				gpu_data->device_active_data.data(),
				gpu_data->device_QXYZ_data.data(),
				gpu_data->device_QUVW_data.data(),
				gpu_data->device_mass_data.data(),
				gpu_data->device_inr_data.data(),
				gpu_data->vel_update.data(),
				gpu_data->omg_update.data(),
				gpu_data->body_number.data(),
				gpu_data->offset_counter.data());

	}

__host__ __device__
void function_shurB(uint &index, int2 *ids, bool *active, real *inv_mass, real3 *inv_inertia, real * gamma, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real3 *QXYZ, real3 *QUVW,
		const real &alpha, const real &compliance, const real & inv_hhpa, real *AX) {
	real temp = 0;
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;
	real3 JXYZ = JXYZA[index];
	if (active[b1] != 0) {
		temp += dot(QXYZ[b1], JXYZ);
		temp += dot(QUVW[b1], JUVWA[index]);
	}
	if (active[b2] != 0) {
		temp += dot(QXYZ[b2], -JXYZ);
		temp += dot(QUVW[b2], JUVWB[index]);
	}
	AX[index] = temp; //+ gamma[index] * inv_hhpa * compliance;
}

__global__ void device_shurB(int2 *ids, bool *active, real *inv_mass, real3 *inv_inertia, real * gamma, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real3 *QXYZ, real3 *QUVW, real *AX) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_constraints_const);
	function_shurB(index, ids, active, inv_mass, inv_inertia, gamma, JXYZA, JXYZB, JUVWA, JUVWB, QXYZ, QUVW, alpha_const, compliance_const, inv_hhpa_const, AX);
}

void ChSolverGPU::host_shurB(int2 *ids, bool *active, real *inv_mass, real3 *inv_inertia, real * gamma, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real3 *QXYZ, real3 *QUVW, real *AX) {

//#pragma omp parallel for
//	for (uint index = 0; index < number_of_constraints; index++) {
//		function_shurB(index, ids, active, inv_mass, inv_inertia, gamma, JXYZA, JXYZB, JUVWA, JUVWB, QXYZ, QUVW, alpha, compliance, inv_hhpa, AX);
//	}

#pragma omp parallel for
	for (uint index = 0; index < number_of_contacts; index++) {
		real3 temp = R3(0);
		int2 id_ = ids[index];
		uint b1 = id_.x;
		uint b2 = id_.y;

		if (active[b1] != 0) {
			real3 XYZ = QXYZ[b1];
			real3 UVW = QUVW[b1];

			temp.x += dot(XYZ, JXYZA[index + number_of_contacts * 0]);
			temp.x += dot(UVW, JUVWA[index + number_of_contacts * 0]);

			temp.y += dot(XYZ, JXYZA[index + number_of_contacts * 1]);
			temp.y += dot(UVW, JUVWA[index + number_of_contacts * 1]);

			temp.z += dot(XYZ, JXYZA[index + number_of_contacts * 2]);
			temp.z += dot(UVW, JUVWA[index + number_of_contacts * 2]);

		}
		if (active[b2] != 0) {
			real3 XYZ = QXYZ[b2];
			real3 UVW = QUVW[b2];

			temp.x += dot(XYZ, JXYZB[index + number_of_contacts * 0]);
			temp.x += dot(UVW, JUVWB[index + number_of_contacts * 0]);

			temp.y += dot(XYZ, JXYZB[index + number_of_contacts * 1]);
			temp.y += dot(UVW, JUVWB[index + number_of_contacts * 1]);

			temp.z += dot(XYZ, JXYZB[index + number_of_contacts * 2]);
			temp.z += dot(UVW, JUVWB[index + number_of_contacts * 2]);
		}
		AX[index + number_of_contacts * 0] += temp.x + gamma[index + number_of_contacts * 0] * inv_hhpa * compliance;
		AX[index + number_of_contacts * 1] += temp.y + gamma[index + number_of_contacts * 1] * inv_hhpa * complianceT;
		AX[index + number_of_contacts * 2] += temp.z + gamma[index + number_of_contacts * 2] * inv_hhpa * complianceT;
	}

}

void ChSolverGPU::shurB(custom_vector<real> &x, custom_vector<real> &out) {
#ifdef SIM_ENABLE_GPU_MODE
		device_shurB CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(
				CASTI2(temp_bids),
				CASTB1(gpu_data->device_active_data),
				CASTR1(gpu_data->device_mass_data),
				CASTR3(gpu_data->device_inr_data),
				CASTR1(x),
				CASTR3(gpu_data->device_JXYZA_data),
				CASTR3(gpu_data->device_JXYZB_data),
				CASTR3(gpu_data->device_JUVWA_data),
				CASTR3(gpu_data->device_JUVWB_data),
				CASTR3(gpu_data->device_QXYZ_data),
				CASTR3(gpu_data->device_QUVW_data),
				CASTR1(out));
#else
		host_shurB(
				temp_bids.data(),
				gpu_data->device_active_data.data(),
				gpu_data->device_mass_data.data(),
				gpu_data->device_inr_data.data(),
				x.data(),
				gpu_data->device_JXYZA_data.data(),
				gpu_data->device_JXYZB_data.data(),
				gpu_data->device_JUVWA_data.data(),
				gpu_data->device_JUVWB_data.data(),
				gpu_data->device_QXYZ_data.data(),
				gpu_data->device_QUVW_data.data(),
				out.data());
#endif
	}

__host__ __device__
void function_bi(uint &index, uint & num_contacts, real &step_size, real *correction, real & recovery_speed, const real & alpha, real *bi) {
	//std::cout<<real(1.0)/step_size* correction[index]<<std::endl;
	bi[index + num_contacts * 0] = fmax(real(1.0) / step_size * correction[index], -recovery_speed);
	bi[index + num_contacts * 1] = 0;
	bi[index + num_contacts * 2] = 0;
}
__global__ void device_bi(real *correction, real* bi) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
	function_bi(index, number_of_contacts_const, step_size_const, correction, contact_recovery_speed_const, alpha_const, bi);
}

void ChSolverGPU::host_bi(real *correction, real* bi) {
#pragma omp parallel for
	for (uint index = 0; index < number_of_contacts; index++) {

		if (compliance) {
			bi[index + number_of_contacts * 0] = inv_hpa * correction[index];
			bi[index + number_of_contacts * 1] = 0;
			bi[index + number_of_contacts * 2] = 0;
		} else {
			bi[index + number_of_contacts * 0] = fmax(real(1.0) / step_size * correction[index], -contact_recovery_speed);
			bi[index + number_of_contacts * 1] = 0;
			bi[index + number_of_contacts * 2] = 0;
		}

		//function_bi(index, number_of_contacts, step_size, correction, contact_recovery_speed, alpha, bi);
	}
}

__host__ __device__ void function_RHS(uint &index, real &step_size, int2 *ids, real *bi, real & recovery_speed, bool* active, real3 *vel, real3 *omega, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA,
		real3 *JUVWB, const real & alpha, real *rhs) {
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;
	real temp = 0;
	if (active[b1]) {
		temp += dot(JXYZA[index], vel[b1]);
		temp += dot(JUVWA[index], omega[b1]);
	}
	if (active[b2]) {
		temp += dot(JXYZB[index], vel[b2]);
		temp += dot(JUVWB[index], omega[b2]);
	}

//	temp -= JXYZA[index].x * vel[b1].x;
//	temp -= JXYZA[index].y * vel[b1].y;
//	temp -= JXYZA[index].z * vel[b1].z;
//	temp -= JUVWA[index].x * omega[b1].x;
//	temp -= JUVWA[index].y * omega[b1].y;
//	temp -= JUVWA[index].z * omega[b1].z;
//	temp -= JXYZB[index].x * vel[b2].x;
//	temp -= JXYZB[index].y * vel[b2].y;
//	temp -= JXYZB[index].z * vel[b2].z;
//	temp -= JUVWB[index].x * omega[b2].x;
//	temp -= JUVWB[index].y * omega[b2].y;
//	temp -= JUVWB[index].z * omega[b2].z;
	//rhs[index] = -((-temp) + fmaxf(inv_hpa * -fabs(correction[index]), 0));
	rhs[index] = -temp - bi[index]; //(temp + fmax(inv_hpa * correction[index], real(-recovery_speed)));

}

__global__ void device_RHS(int2 *ids, real *bi, bool * active, real3 *vel, real3 *omega, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real *rhs) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_constraints_const);
	function_RHS(index, step_size_const, ids, bi, contact_recovery_speed_const, active, vel, omega, JXYZA, JXYZB, JUVWA, JUVWB, alpha_const, rhs);
}

void ChSolverGPU::host_RHS(int2 *ids, real *bi, bool * active, real3 *vel, real3 *omega, real3 *JXYZA, real3 *JXYZB, real3 *JUVWA, real3 *JUVWB, real *rhs) {
#pragma omp parallel for
	for (uint index = 0; index < number_of_constraints; index++) {
		function_RHS(index, step_size, ids, bi, contact_recovery_speed, active, vel, omega, JXYZA, JXYZB, JUVWA, JUVWB, alpha, rhs);
	}
}

void ChSolverGPU::ComputeRHS() {
	timer_rhs.start();
	//Thrust_Fill(gpu_data->device_QXYZ_data, R3(0));
	//Thrust_Fill(gpu_data->device_QUVW_data, R3(0));

#ifdef SIM_ENABLE_GPU_MODE
	device_bi CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(CASTR1(correction), CASTR1(bi));
	hipDeviceSynchronize();
#else
	host_bi(correction.data(), bi.data());
#endif

#ifdef SIM_ENABLE_GPU_MODE
	device_RHS CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(CASTI2(temp_bids), CASTR1(bi), CASTB1(gpu_data->device_active_data), CASTR3(gpu_data->device_vel_data),
			CASTR3(gpu_data->device_omg_data), CASTR3(gpu_data->device_JXYZA_data), CASTR3(gpu_data->device_JXYZB_data), CASTR3(gpu_data->device_JUVWA_data), CASTR3(gpu_data->device_JUVWB_data),
			CASTR1(rhs));
	hipDeviceSynchronize();
#else
	host_RHS(temp_bids.data(), bi.data(), gpu_data->device_active_data.data(), gpu_data->device_vel_data.data(), gpu_data->device_omg_data.data(), gpu_data->device_JXYZA_data.data(),
			gpu_data->device_JXYZB_data.data(), gpu_data->device_JUVWA_data.data(), gpu_data->device_JUVWB_data.data(), rhs.data());
#endif

	timer_rhs.stop();
	time_rhs = timer_rhs();
}

__global__ void device_Offsets(int2* ids, real4* bilaterals, uint* Body) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < number_of_contacts_const) {
		int2 temp_id = ids[index];
		Body[index] = temp_id.x;
		Body[index + number_of_contacts_const] = temp_id.y;
	}

	if (index < number_of_bilaterals_const) {
		Body[2 * number_of_contacts_const + index] = bilaterals[index].w;
		Body[2 * number_of_contacts_const + index + number_of_bilaterals_const] = bilaterals[index + number_of_bilaterals_const].w;
	}
}

void ChSolverGPU::host_Offsets(int2* ids, real4* bilaterals, uint* Body) {
	for (uint index = 0; index < number_of_contacts + number_of_bilaterals; index++) {
		if (index < number_of_contacts) {
			int2 temp_id = ids[index];
			Body[index] = temp_id.x;
			Body[index + number_of_contacts] = temp_id.y;
		}

		if (index < number_of_bilaterals) {
			Body[2 * number_of_contacts + index] = bilaterals[index].w;
			Body[2 * number_of_contacts + index + number_of_bilaterals] = bilaterals[index + number_of_bilaterals].w;
		}
	}
}

void ChSolverGPU::Setup() {
	time_rhs = time_shurcompliment = time_project = time_integrate = 0;
	///
	maxd_hist.clear();
	maxdeltalambda_hist.clear();
	iter_hist.clear();
	///
	number_of_constraints = gpu_data->number_of_contacts * 3 + gpu_data->number_of_bilaterals;
	number_of_contacts = gpu_data->number_of_contacts;
	number_of_bilaterals = gpu_data->number_of_bilaterals;
	number_of_objects = gpu_data->number_of_objects;
	temp_bids.resize(number_of_constraints);
	correction.resize(number_of_constraints);
	rhs.resize(number_of_constraints);
	bi.resize(number_of_constraints);
	gpu_data->device_gam_data.resize((number_of_constraints));

	gpu_data->device_QXYZ_data.resize(number_of_objects);
	gpu_data->device_QUVW_data.resize(number_of_objects);
	Thrust_Fill(gpu_data->device_QXYZ_data, R3(0));
	Thrust_Fill(gpu_data->device_QUVW_data, R3(0));
	///
	Thrust_Fill(gpu_data->device_gam_data, 0);
	Thrust_Fill(correction, 0);
	///
	thrust::copy_n(gpu_data->device_dpth_data.begin(), gpu_data->number_of_contacts, correction.begin() + gpu_data->number_of_contacts * 0);
	///
	thrust::copy_n(gpu_data->device_bids_data.begin(), gpu_data->number_of_contacts, temp_bids.begin() + gpu_data->number_of_contacts * 0);
	thrust::copy_n(gpu_data->device_bids_data.begin(), gpu_data->number_of_contacts, temp_bids.begin() + gpu_data->number_of_contacts * 1);
	thrust::copy_n(gpu_data->device_bids_data.begin(), gpu_data->number_of_contacts, temp_bids.begin() + gpu_data->number_of_contacts * 2);

	inv_hpa = 1.0 / (step_size + alpha);
	inv_hhpa = 1.0 / (step_size * (step_size + alpha));

#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(number_of_constraints);
	COPY_TO_CONST_MEM(number_of_contacts);
	COPY_TO_CONST_MEM(number_of_bilaterals);
	COPY_TO_CONST_MEM(step_size);
	COPY_TO_CONST_MEM(alpha);
	COPY_TO_CONST_MEM(compliance);
	COPY_TO_CONST_MEM(inv_hpa);
	COPY_TO_CONST_MEM(inv_hhpa);
	COPY_TO_CONST_MEM(contact_recovery_speed);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Project), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_shurA), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_shurB), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_RHS), hipFuncCachePreferL1);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Offsets), hipFuncCachePreferL1);

#else
#endif
	uint number_of_cont_bilat = number_of_contacts + number_of_bilaterals;

	if (number_of_cont_bilat > 0) {
		update_number.resize((number_of_cont_bilat) * 2, 0);
		gpu_data->offset_counter.resize((number_of_cont_bilat) * 2, 0);
		gpu_data->update_offset.resize((number_of_cont_bilat) * 2, 0);
		body_num.resize((number_of_cont_bilat) * 2, 0);
		gpu_data->device_dgm_data.resize((number_of_constraints));
		gpu_data->device_gam_data.resize((number_of_constraints));
		Thrust_Fill(gpu_data->device_dgm_data, 1);
		gpu_data->vel_update.resize((number_of_cont_bilat) * 2);
		gpu_data->omg_update.resize((number_of_cont_bilat) * 2);
#ifdef SIM_ENABLE_GPU_MODE
		device_Offsets CUDA_KERNEL_DIM(BLOCKS(number_of_cont_bilat), THREADS)(CASTI2(gpu_data->device_bids_data), CASTR4(gpu_data->device_bilateral_data), CASTU1(body_num));
#else
		host_Offsets(gpu_data->device_bids_data.data(), gpu_data->device_bilateral_data.data(), body_num.data());
#endif
		Thrust_Sequence(update_number);
		Thrust_Sequence(gpu_data->update_offset);
		Thrust_Fill(gpu_data->offset_counter, 0);
		Thrust_Sort_By_Key(body_num, update_number);
		Thrust_Sort_By_Key(update_number, gpu_data->update_offset);
		gpu_data->body_number = body_num;
		Thrust_Reduce_By_KeyB(gpu_data->number_of_updates, body_num, update_number, gpu_data->offset_counter);
		Thrust_Inclusive_Scan(gpu_data->offset_counter);
	}
	number_of_updates = gpu_data->number_of_updates;
}

void ChSolverGPU::ShurProduct(custom_vector<real> &x, custom_vector<real> & output) {
	timer_shurcompliment.start();
	Thrust_Fill(output, 0);
	shurA(x);
	shurB(x,output);
	timer_shurcompliment.stop();
	time_shurcompliment +=timer_shurcompliment();
}

void ChSolverGPU::ComputeImpulses() {
	shurA(gpu_data->device_gam_data);
	gpu_data->device_vel_data += gpu_data->device_QXYZ_data;
	gpu_data->device_omg_data += gpu_data->device_QUVW_data;
}

void ChSolverGPU::Solve(GPUSOLVERTYPE solver_type, real step, gpu_container &gpu_data_) {
	timer_solver.start();
	gpu_data = &gpu_data_;
	step_size = step;
	Setup();
	total_iteration = 0;
	if (number_of_constraints > 0) {
		ComputeRHS();

		if (solver_type == STEEPEST_DESCENT) {
			total_iteration += SolveSD(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == GRADIENT_DESCENT) {
			total_iteration += SolveGD(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == CONJUGATE_GRADIENT) {
			total_iteration += SolveCG(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == CONJUGATE_GRADIENT_SQUARED) {
			total_iteration += SolveCGS(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == BICONJUGATE_GRADIENT) {
			total_iteration += SolveBiCG(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == BICONJUGATE_GRADIENT_STAB) {
			total_iteration += SolveBiCGStab(gpu_data->device_gam_data, rhs, max_iteration);
		} else if (solver_type == MINIMUM_RESIDUAL) {
			total_iteration += SolveMinRes(gpu_data->device_gam_data, rhs, max_iteration);
		}
		//else if(solver_type==QUASAI_MINIMUM_RESIDUAL){SolveQMR(gpu_data->device_gam_data, rhs, max_iteration);}
		else if (solver_type == ACCELERATED_PROJECTED_GRADIENT_DESCENT) {
			total_iteration += SolveAPGD(gpu_data->device_gam_data, rhs, max_iteration);
		}
		current_iteration = total_iteration;
		ComputeImpulses();
		timer_solver.stop();
		time_solver = timer_solver();
	}
}

//
//__host__ __device__ void function_InvNDiag(uint index, int b1, int b2, int2* ids, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real * inv_mass, real3 * inv_inertia, real & inv_n_diag) {
//
//  real Jx1 = JXYZA[index].x;
//  real Jy1 = JXYZA[index].y;
//  real Jz1 = JXYZA[index].z;
//  real Ju1 = JUVWA[index].x;
//  real Jv1 = JUVWA[index].y;
//  real Jw1 = JUVWA[index].z;
//  //
//  real Jx2 = JXYZB[index].x;
//  real Jy2 = JXYZB[index].y;
//  real Jz2 = JXYZB[index].z;
//  real Ju2 = JUVWB[index].x;
//  real Jv2 = JUVWB[index].y;
//  real Jw2 = JUVWB[index].z;
//  //
//  real m1 = inv_mass[b1];
//  real m2 = inv_mass[b2];
//  //
//  real3 i1 = inv_inertia[b1];
//  real3 i2 = inv_inertia[b2];
//
//  real part1 = dot(JXYZA[index], JXYZA[index]) * m1;
//  real part2 = dot(JUVWA[index] * JUVWA[index], i1);
//  real part3 = dot(JXYZB[index], JXYZB[index]) * m2;
//  real part4 = dot(JUVWB[index] * JUVWB[index], i2);
//  inv_n_diag = (part1 + part2 + part3 + part4);
//}
