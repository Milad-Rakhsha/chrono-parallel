#include "hip/hip_runtime.h"
#include "ChSolverGPU.cuh"
using namespace chrono;

__constant__ uint number_of_constraints_const;
__constant__ uint number_of_contacts_const;
__constant__ uint number_of_bilaterals_const;
__constant__ real step_size_const;
__constant__ real alpha_const;
__constant__ real compliance_const;
__constant__ real inv_hpa_const;
__constant__ real inv_hhpa_const;
__constant__ real contact_recovery_speed_const;

__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*) address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

//=================================================================================================================================
__host__ __device__ void function_Project(uint &index, uint number_of_contacts, int2 *ids, real *fric, real* cohesion, real *gam) {
	int2 body_id = ids[index];
	real3 gamma;
	gamma.x = gam[index + number_of_contacts * 0];
	gamma.y = gam[index + number_of_contacts * 1];
	gamma.z = gam[index + number_of_contacts * 2];

	real coh = (cohesion[body_id.x] + cohesion[body_id.y]) * .5;
	if (coh < 0) {
		coh = 0;
	}
	gamma.x += coh;
	real f_tang = sqrt(gamma.y * gamma.y + gamma.z * gamma.z);
	real mu = (fric[body_id.x] == 0 || fric[body_id.y] == 0) ? 0 : (fric[body_id.x] + fric[body_id.y]) * .5;
	if (mu == 0) {
		gamma.x = gamma.x < 0 ? 0 : gamma.x;
		gamma.y = gamma.z = 0;

		gam[index + number_of_contacts * 0] = gamma.x;
		gam[index + number_of_contacts * 1] = gamma.y;
		gam[index + number_of_contacts * 2] = gamma.z;

		return;

	}
	// inside upper cone? keep untouched!
	if (f_tang < (mu * gamma.x)) {
		return;
	}

	// inside lower cone? reset  normal,u,v to zero!

	if ((f_tang) < -(1.0 / mu) * gamma.x || (fabs(gamma.x) < 10e-15)) {
		gamma = R3(0);
		gam[index + number_of_contacts * 0] = gamma.x;
		gam[index + number_of_contacts * 1] = gamma.y;
		gam[index + number_of_contacts * 2] = gamma.z;

		return;
	}

	// remaining case: project orthogonally to generator segment of upper cone

	gamma.x = (f_tang * mu + gamma.x) / (mu * mu + 1);
	real tproj_div_t = (gamma.x * mu) / f_tang;
	gamma.y *= tproj_div_t;
	gamma.z *= tproj_div_t;

	gam[index + number_of_contacts * 0] = gamma.x - coh;
	gam[index + number_of_contacts * 1] = gamma.y;
	gam[index + number_of_contacts * 2] = gamma.z;
}

__global__ void device_Project(int2 *ids, real *friction, real* cohesion, real *gamma) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_contacts_const);
	function_Project(index, number_of_contacts_const, ids, friction, cohesion, gamma);
}

void ChSolverGPU::host_Project(int2 *ids, real *friction, real* cohesion, real *gamma) {
#pragma omp parallel for
	for (uint index = 0; index < number_of_contacts; index++) {
		function_Project(index, number_of_contacts, ids, friction, cohesion, gamma);
	}
}
void ChSolverGPU::Project(custom_vector<real> & gamma) {
	timer_project.start();

#ifdef SIM_ENABLE_GPU_MODE
		device_Project CUDA_KERNEL_DIM(BLOCKS(number_of_contacts), THREADS)(
				CASTI2(data_container->device_data.device_bids_data),
				CASTR1(data_container->device_data.device_fric_data),
				CASTR1(data_container->device_data.device_cohesion_data),
				CASTR1(gamma));
#else
		host_Project (
				data_container->host_data.bids_data.data(),
				data_container->host_data.fric_data.data(),
				data_container->host_data.cohesion_data.data(),
				gamma.data());
#endif
		timer_project.stop();
		time_project += timer_project();
	}
//=================================================================================================================================

__host__ __device__ void function_Reduce_Shur(
		uint& index,
		bool* active,
		real3* QXYZ,
		real3* QUVW,
		real *inv_mass,
		real3 *inv_inertia,
		real3* updateQXYZ,
		real3* updateQUVW,
		uint* d_body_num,
		uint* counter) {
	int start = (index == 0) ? 0 : counter[index - 1], end = counter[index];
	int id = d_body_num[end - 1], j;

	if (active[id] == 0) {
		return;
	}
	real3 mUpdateV = R3(0);
	real3 mUpdateO = R3(0);

	for (j = 0; j < end - start; j++) {
		mUpdateV += updateQXYZ[j + start];
		mUpdateO += updateQUVW[j + start];
	}
	QXYZ[id] = mUpdateV * inv_mass[id];
	QUVW[id] = mUpdateO * inv_inertia[id];
}

void ChSolverGPU::host_Reduce_Shur(
		bool* active,
		real3* QXYZ,
		real3* QUVW,
		real *inv_mass,
		real3 *inv_inertia,
		real3* updateQXYZ,
		real3* updateQUVW,
		uint* d_body_num,
		uint* counter) {
#pragma omp parallel for
	for (uint index = 0; index < number_of_updates; index++) {
		function_Reduce_Shur(index, active, QXYZ, QUVW, inv_mass, inv_inertia, updateQXYZ, updateQUVW, d_body_num, counter);
	}
}
//=================================================================================================================================
__global__ void device_shurA(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real *gamma,
		real3 *QXYZ,
		real3 *QUVW) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_constraints_const);
}

void ChSolverGPU::host_shurA_contacts(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real *gamma,
		real3 *updateV,
		real3 *updateO,
		real3* QXYZ,
		real3* QUVW,
		uint* offset) {
#pragma omp parallel for
	for (int index = 0; index < number_of_contacts; index++) {
		real3 gam;
		gam.x = gamma[index + number_of_contacts * 0];
		gam.y = gamma[index + number_of_contacts * 1];
		gam.z = gamma[index + number_of_contacts * 2];
		uint b1 = ids[index].x;

		int offset1 = offset[index];
		int offset2 = offset[index + number_of_contacts];

		if (active[b1] != 0) {
			updateV[offset1] = JXYZA[index + number_of_contacts * 0] * gam.x + JXYZA[index + number_of_contacts * 1] * gam.y
					+ JXYZA[index + number_of_contacts * 2] * gam.z;
			updateO[offset1] = JUVWA[index + number_of_contacts * 0] * gam.x + JUVWA[index + number_of_contacts * 1] * gam.y
					+ JUVWA[index + number_of_contacts * 2] * gam.z;
		}
		uint b2 = ids[index].y;
		if (active[b2] != 0) {
			updateV[offset2] = JXYZB[index + number_of_contacts * 0] * gam.x + JXYZB[index + number_of_contacts * 1] * gam.y
					+ JXYZB[index + number_of_contacts * 2] * gam.z;
			updateO[offset2] = JUVWB[index + number_of_contacts * 0] * gam.x + JUVWB[index + number_of_contacts * 1] * gam.y
					+ JUVWB[index + number_of_contacts * 2] * gam.z;
		}
	}
}

void ChSolverGPU::host_shurA_bilaterals(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real *gamma,
		real3 *updateV,
		real3 *updateO,
		real3* QXYZ,
		real3* QUVW,
		uint* offset) {

#pragma omp parallel for
	for (int index = 0; index < number_of_bilaterals; index++) {
		real gam;
		gam = gamma[index + number_of_contacts * 3];
		uint b1 = ids[index + number_of_contacts * 3].x;
		int offset1 = offset[2 * number_of_contacts + index];
		int offset2 = offset[2 * number_of_contacts + index + number_of_bilaterals];

		if (active[b1] != 0) {
			updateV[offset1] = JXYZA[index + number_of_contacts * 3] * gam;
			updateO[offset1] = JUVWA[index + number_of_contacts * 3] * gam;
		}

		uint b2 = ids[index + number_of_contacts * 3].y;
		if (active[b2] != 0) {
			updateV[offset2] = JXYZB[index + number_of_contacts * 3] * gam;
			updateO[offset2] = JUVWB[index + number_of_contacts * 3] * gam;
		}
	}
}
void ChSolverGPU::shurA(custom_vector<real> &x) {

#ifdef SIM_ENABLE_GPU_MODE
		device_shurA CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(
				CASTI2(data_container->device_data.device_bidlist_data),
				CASTB1(data_container->device_data.device_active_data),
				CASTR1(data_container->device_data.device_mass_data),
				CASTR3(data_container->device_data.device_inr_data),
				CASTR3(data_container->device_data.device_JXYZA_data),
				CASTR3(data_container->device_data.device_JXYZB_data),
				CASTR3(data_container->device_data.device_JUVWA_data),
				CASTR3(data_container->device_data.device_JUVWB_data),
				CASTR1(x),
				CASTR3(data_container->device_data.device_QXYZ_data),
				CASTR3(data_container->device_data.device_QUVW_data));
		hipDeviceSynchronize();
#else
		host_shurA_contacts(
				data_container->host_data.bidlist_data.data(),
				data_container->host_data.active_data.data(),
				data_container->host_data.mass_data.data(),
				data_container->host_data.inr_data.data(),
				data_container->host_data.JXYZA_data.data(),
				data_container->host_data.JXYZB_data.data(),
				data_container->host_data.JUVWA_data.data(),
				data_container->host_data.JUVWB_data.data(),
				x.data(),
				data_container->host_data.vel_update.data(),
				data_container->host_data.omg_update.data(),
				data_container->host_data.QXYZ_data.data(),
				data_container->host_data.QUVW_data.data(),
				data_container->host_data.update_offset.data());
		host_shurA_bilaterals(
				data_container->host_data.bidlist_data.data(),
				data_container->host_data.active_data.data(),
				data_container->host_data.mass_data.data(),
				data_container->host_data.inr_data.data(),
				data_container->host_data.JXYZA_data.data(),
				data_container->host_data.JXYZB_data.data(),
				data_container->host_data.JUVWA_data.data(),
				data_container->host_data.JUVWB_data.data(),
				x.data(),
				data_container->host_data.vel_update.data(),
				data_container->host_data.omg_update.data(),
				data_container->host_data.QXYZ_data.data(),
				data_container->host_data.QUVW_data.data(),
				data_container->host_data.update_offset.data());

		host_Reduce_Shur(
				data_container->host_data.active_data.data(),
				data_container->host_data.QXYZ_data.data(),
				data_container->host_data.QUVW_data.data(),
				data_container->host_data.mass_data.data(),
				data_container->host_data.inr_data.data(),
				data_container->host_data.vel_update.data(),
				data_container->host_data.omg_update.data(),
				data_container->host_data.body_number.data(),
				data_container->host_data.offset_counter.data());

#endif

	}
//=================================================================================================================================
__host__ __device__
void function_shurB(
		uint &index,
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real * compliance,
		real * gamma,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real3 *QXYZ,
		real3 *QUVW,
		const real &alpha,
		const real & inv_hhpa,
		real *AX) {
	real temp = 0;
	uint b1 = ids[index].x;
	uint b2 = ids[index].y;

	real3 JXYZ = JXYZA[index];
	if (active[b1] != 0) {
		temp += dot(QXYZ[b1], JXYZ);
		temp += dot(QUVW[b1], JUVWA[index]);
	}
	if (active[b2] != 0) {
		temp += dot(QXYZ[b2], -JXYZ);
		temp += dot(QUVW[b2], JUVWB[index]);
	}
	AX[index] = temp + gamma[index] * inv_hhpa * compliance[index];
}

__global__ void device_shurB(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real * compliance,
		real * gamma,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real3 *QXYZ,
		real3 *QUVW,
		real *AX) {
	INIT_CHECK_THREAD_BOUNDED(INDEX1D, number_of_constraints_const);
	function_shurB(
			index,
			ids,
			active,
			inv_mass,
			inv_inertia,
			compliance,
			gamma,
			JXYZA,
			JXYZB,
			JUVWA,
			JUVWB,
			QXYZ,
			QUVW,
			alpha_const,
			inv_hhpa_const,
			AX);
}

void ChSolverGPU::host_shurB_contacts(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real * compliance,
		real * gamma,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real3 *QXYZ,
		real3 *QUVW,
		real *AX) {

#pragma omp parallel for
	for (uint index = 0; index < number_of_contacts; index++) {
		real3 temp = R3(0);
		int2 id_ = ids[index];
		uint b1 = id_.x;
		uint b2 = id_.y;

		if (active[b1] != 0) {
			real3 XYZ = QXYZ[b1];
			real3 UVW = QUVW[b1];

			temp.x += dot(XYZ, JXYZA[index + number_of_contacts * 0]);
			temp.x += dot(UVW, JUVWA[index + number_of_contacts * 0]);

			temp.y += dot(XYZ, JXYZA[index + number_of_contacts * 1]);
			temp.y += dot(UVW, JUVWA[index + number_of_contacts * 1]);

			temp.z += dot(XYZ, JXYZA[index + number_of_contacts * 2]);
			temp.z += dot(UVW, JUVWA[index + number_of_contacts * 2]);

		}
		if (active[b2] != 0) {
			real3 XYZ = QXYZ[b2];
			real3 UVW = QUVW[b2];

			temp.x += dot(XYZ, JXYZB[index + number_of_contacts * 0]);
			temp.x += dot(UVW, JUVWB[index + number_of_contacts * 0]);

			temp.y += dot(XYZ, JXYZB[index + number_of_contacts * 1]);
			temp.y += dot(UVW, JUVWB[index + number_of_contacts * 1]);

			temp.z += dot(XYZ, JXYZB[index + number_of_contacts * 2]);
			temp.z += dot(UVW, JUVWB[index + number_of_contacts * 2]);
		}
		AX[index + number_of_contacts * 0] += temp.x + gamma[index + number_of_contacts * 0] * inv_hhpa * compliance[index];
		AX[index + number_of_contacts * 1] += temp.y + gamma[index + number_of_contacts * 1] * inv_hhpa * compliance[index];
		AX[index + number_of_contacts * 2] += temp.z + gamma[index + number_of_contacts * 2] * inv_hhpa * compliance[index];
	}

}
void ChSolverGPU::host_shurB_bilaterals(
		int2 *ids,
		bool *active,
		real *inv_mass,
		real3 *inv_inertia,
		real * gamma,
		real3 *JXYZA,
		real3 *JXYZB,
		real3 *JUVWA,
		real3 *JUVWB,
		real3 *QXYZ,
		real3 *QUVW,
		real *AX) {

#pragma omp parallel for
	for (uint index = 0; index < number_of_bilaterals; index++) {

		real temp = 0;
		uint b1 = ids[index + number_of_contacts * 3].x;
		if (active[b1] != 0) {
			real3 XYZ = QXYZ[b1];
			real3 UVW = QUVW[b1];
			temp += dot(XYZ, JXYZA[index + number_of_contacts * 3]);
			temp += dot(UVW, JUVWA[index + number_of_contacts * 3]);

		}
		uint b2 = ids[index + number_of_contacts * 3].y;
		if (active[b2] != 0) {

			real3 XYZ = QXYZ[b2];
			real3 UVW = QUVW[b2];
			temp += dot(XYZ, JXYZB[index + number_of_contacts * 3]);
			temp += dot(UVW, JUVWB[index + number_of_contacts * 3]);

		}

		AX[index + number_of_contacts * 3] = temp;
	}
}
void ChSolverGPU::shurB(custom_vector<real> &x, custom_vector<real> &out) {
#ifdef SIM_ENABLE_GPU_MODE
		device_shurB CUDA_KERNEL_DIM(BLOCKS(number_of_constraints), THREADS)(
				CASTI2(data_container->device_data.device_bidlist_data),
				CASTB1(data_container->device_data.device_active_data),
				CASTR1(data_container->device_data.device_mass_data),
				CASTR3(data_container->device_data.device_inr_data),
				CASTR1(data_container->device_data.device_compliance_data),
				CASTR1(data_container->device_data.device_complianceT_data),
				CASTR1(x),
				CASTR3(data_container->device_data.device_JXYZA_data),
				CASTR3(data_container->device_data.device_JXYZB_data),
				CASTR3(data_container->device_data.device_JUVWA_data),
				CASTR3(data_container->device_data.device_JUVWB_data),
				CASTR3(data_container->device_data.device_QXYZ_data),
				CASTR3(data_container->device_data.device_QUVW_data),
				CASTR1(out));
#else
		host_shurB_contacts(
				data_container->host_data.bidlist_data.data(),
				data_container->host_data.active_data.data(),
				data_container->host_data.mass_data.data(),
				data_container->host_data.inr_data.data(),
				data_container->host_data.comp_data.data(),
				x.data(),
				data_container->host_data.JXYZA_data.data(),
				data_container->host_data.JXYZB_data.data(),
				data_container->host_data.JUVWA_data.data(),
				data_container->host_data.JUVWB_data.data(),
				data_container->host_data.QXYZ_data.data(),
				data_container->host_data.QUVW_data.data(),
				out.data());
		host_shurB_bilaterals(
				data_container->host_data.bidlist_data.data(),
				data_container->host_data.active_data.data(),
				data_container->host_data.mass_data.data(),
				data_container->host_data.inr_data.data(),
				x.data(),
				data_container->host_data.JXYZA_data.data(),
				data_container->host_data.JXYZB_data.data(),
				data_container->host_data.JUVWA_data.data(),
				data_container->host_data.JUVWB_data.data(),
				data_container->host_data.QXYZ_data.data(),
				data_container->host_data.QUVW_data.data(),
				out.data());
#endif
	}
void ChSolverGPU::ShurProduct(custom_vector<real> &x, custom_vector<real> & output) {
	timer_shurcompliment.start();
	Thrust_Fill(output, 0);
#ifdef SIM_ENABLE_GPU_MODE
		Thrust_Fill(data_container->device_data.device_QXYZ_data, R3(0));
		Thrust_Fill(data_container->device_data.device_QUVW_data, R3(0));
#else
#pragma omp parallel for
		for (int i = 0; i < number_of_objects; i++) {
			data_container->host_data.QXYZ_data[i] = R3(0);
			data_container->host_data.QUVW_data[i] = R3(0);
		}
#endif
		shurA(x);
		shurB(x,output);
		timer_shurcompliment.stop();
		time_shurcompliment +=timer_shurcompliment();
	}
#define JXYZA_bilateral data_container->host_data.JXYZA_bilateral
#define JXYZB_bilateral data_container->host_data.JXYZB_bilateral
#define JUVWA_bilateral data_container->host_data.JUVWA_bilateral
#define JUVWB_bilateral data_container->host_data.JUVWB_bilateral
//=================================================================================================================================
void ChSolverGPU::ShurBilaterals(custom_vector<real> &x_t, custom_vector<real> & AX) {

	for (int i = 0; i < number_of_objects; i++) {
		data_container->host_data.QXYZ_data[i] = R3(0);
		data_container->host_data.QUVW_data[i] = R3(0);
	}

	for (int index = 0; index < number_of_bilaterals; index++) {
		real gam;
		gam = x_t[index];
		uint b1 = data_container->host_data.bidlist_data[index + number_of_contacts * 3].x;

		if (data_container->host_data.active_data[b1] != 0) {
			data_container->host_data.QXYZ_data[b1] += JXYZA_bilateral[index] * gam;
			data_container->host_data.QUVW_data[b1] += JUVWA_bilateral[index] * gam;
		}

		uint b2 = data_container->host_data.bidlist_data[index + number_of_contacts * 3].y;
		if (data_container->host_data.active_data[b2] != 0) {
			data_container->host_data.QXYZ_data[b2] += JXYZB_bilateral[index ] * gam;
			data_container->host_data.QUVW_data[b2] += JUVWB_bilateral[index ] * gam;
		}
	}

#pragma omp parallel for
		for (uint index = 0; index < number_of_bilaterals; index++) {

			real temp = 0;
			uint b1 = data_container->host_data.bidlist_data[index + number_of_contacts * 3].x;
			if (data_container->host_data.active_data[b1] != 0) {
				real3 XYZ = data_container->host_data.QXYZ_data[b1]* data_container->host_data.mass_data[b1];
				real3 UVW = data_container->host_data.QUVW_data[b1]* data_container->host_data.inr_data[b1];
				temp += dot(XYZ, JXYZA_bilateral[index ]);
				temp += dot(UVW, JUVWA_bilateral[index ]);

			}
			uint b2 = data_container->host_data.bidlist_data[index + number_of_contacts * 3].y;
			if (data_container->host_data.active_data[b2] != 0) {

				real3 XYZ = data_container->host_data.QXYZ_data[b2]* data_container->host_data.mass_data[b2];;
				real3 UVW = data_container->host_data.QUVW_data[b2]* data_container->host_data.inr_data[b2];
				temp += dot(XYZ, JXYZB_bilateral[index ]);
				temp += dot(UVW, JUVWB_bilateral[index ]);

			}
			AX[index ] = temp;
		}

	}
//=================================================================================================================================
__global__ void device_Offsets(int2* ids, real4* bilaterals, uint* Body) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < number_of_contacts_const) {
		int2 temp_id = ids[index];
		Body[index] = temp_id.x;
		Body[index + number_of_contacts_const] = temp_id.y;
	}

	if (index < number_of_bilaterals_const) {
		Body[2 * number_of_contacts_const + index] = bilaterals[index].w;
		Body[2 * number_of_contacts_const + index + number_of_bilaterals_const] = bilaterals[index + number_of_bilaterals_const].w;
	}
}

void ChSolverGPU::host_Offsets(int2* ids_contacts, int2* ids_bilaterals, uint* Body) {
	for (uint index = 0; index < number_of_contacts + number_of_bilaterals; index++) {
		if (index < number_of_contacts) {
			int2 temp_id = ids_contacts[index];
			Body[index] = temp_id.x;
			Body[index + number_of_contacts] = temp_id.y;
		}

		if (index < number_of_bilaterals) {
			int2 temp_id = ids_bilaterals[index];
			Body[2 * number_of_contacts + index] = temp_id.x;
			Body[2 * number_of_contacts + index + number_of_bilaterals] = temp_id.y;
		}
	}
}
//=================================================================================================================================
void ChSolverGPU::Setup() {
	time_shurcompliment = time_project = time_integrate = 0;
	///////////////////////////////////////
	maxd_hist.clear();
	maxdeltalambda_hist.clear();
	iter_hist.clear();
	///////////////////////////////////////
	Initialize();

	data_container->host_data.gam_data.resize((number_of_constraints));
	data_container->host_data.QXYZ_data.resize(number_of_objects);
	data_container->host_data.QUVW_data.resize(number_of_objects);

	///////////////////////////////////////
#ifdef SIM_ENABLE_GPU_MODE
	Thrust_Fill(data_container->device_data.device_QXYZ_data, R3(0));
	Thrust_Fill(data_container->device_data.device_QUVW_data, R3(0));
	Thrust_Fill(data_container->device_data.device_gam_data, 0);
	Thrust_Fill(correction, 0);

#else
#pragma omp parallel for
	for (int i = 0; i < number_of_objects; i++) {
		data_container->host_data.QXYZ_data[i] = R3(0);
		data_container->host_data.QUVW_data[i] = R3(0);
	}
#pragma omp parallel for
	for (int i = 0; i < number_of_constraints; i++) {
		data_container->host_data.gam_data[i] = 0;
	}
#endif

	///////////////////////////////////////

	///////////////////////////////////////
	thrust::copy_n(
			data_container->host_data.gamma_bilateral.begin(),
			data_container->number_of_bilaterals,
			data_container->host_data.gam_data.begin() + data_container->number_of_contacts * 3);

	///////////////////////////////////////

#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(number_of_constraints);
	COPY_TO_CONST_MEM(number_of_contacts);
	COPY_TO_CONST_MEM(number_of_bilaterals);
	COPY_TO_CONST_MEM(step_size);
	COPY_TO_CONST_MEM(alpha);
	COPY_TO_CONST_MEM(inv_hpa);
	COPY_TO_CONST_MEM(inv_hhpa);
	COPY_TO_CONST_MEM(contact_recovery_speed);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Project), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_shurA), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_shurB), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_RHS), hipFuncCachePreferL1);
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Offsets), hipFuncCachePreferL1);

#else
#endif
	uint number_of_cont_bilat = number_of_contacts + number_of_bilaterals;

	if (number_of_cont_bilat > 0) {
		update_number.resize((number_of_cont_bilat) * 2, 0);
		data_container->host_data.offset_counter.resize((number_of_cont_bilat) * 2, 0);
		data_container->host_data.update_offset.resize((number_of_cont_bilat) * 2, 0);
		body_num.resize((number_of_cont_bilat) * 2, 0);
		data_container->host_data.vel_update.resize((number_of_cont_bilat) * 2);
		data_container->host_data.omg_update.resize((number_of_cont_bilat) * 2);
#ifdef SIM_ENABLE_GPU_MODE
		device_Offsets CUDA_KERNEL_DIM(BLOCKS(number_of_cont_bilat), THREADS)(CASTI2(data_container->device_data.device_bids_data), CASTR4(data_container->device_data.device_bilateral_data), CASTU1(body_num));
#else
		host_Offsets(data_container->host_data.bids_data.data(), data_container->host_data.bids_bilateral.data(), body_num.data());
#endif
		Thrust_Sequence(update_number);
		Thrust_Sequence(data_container->host_data.update_offset);
		Thrust_Fill(data_container->host_data.offset_counter, 0);
		Thrust_Sort_By_Key(body_num, update_number);
		Thrust_Sort_By_Key(update_number, data_container->host_data.update_offset);
		data_container->host_data.body_number = body_num;
		Thrust_Reduce_By_KeyB(data_container->number_of_updates, body_num, update_number, data_container->host_data.offset_counter);
		Thrust_Inclusive_Scan(data_container->host_data.offset_counter);
	}
	number_of_updates = data_container->number_of_updates;

}

void ChSolverGPU::ComputeImpulses() {
	shurA(data_container->host_data.gam_data);
	data_container->host_data.vel_data += data_container->host_data.QXYZ_data;
	data_container->host_data.omg_data += data_container->host_data.QUVW_data;
}

void ChSolverGPU::Solve(GPUSOLVERTYPE solver_type, real step, ChGPUDataManager *data_container_) {
	timer_solver.start();
	data_container = data_container_;
	step_size = step;

	Setup();
	total_iteration = 0;
	if (number_of_constraints > 0) {

		if (solver_type == STEEPEST_DESCENT) {
			total_iteration += SolveSD(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == GRADIENT_DESCENT) {
			total_iteration += SolveGD(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == CONJUGATE_GRADIENT) {
			total_iteration += SolveCG(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == CONJUGATE_GRADIENT_SQUARED) {
			total_iteration += SolveCGS(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == BICONJUGATE_GRADIENT) {
			total_iteration += SolveBiCG(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == BICONJUGATE_GRADIENT_STAB) {
			total_iteration += SolveBiCGStab(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		} else if (solver_type == MINIMUM_RESIDUAL) {
			total_iteration += SolveMinRes(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
		}
		//else if(solver_type==QUASAI_MINIMUM_RESIDUAL){SolveQMR(data_container->gpu_data.device_gam_data, rhs, max_iteration);}
		else if (solver_type == ACCELERATED_PROJECTED_GRADIENT_DESCENT) {
			if (do_stab) {
				custom_vector<real> rhs_bilateral(data_container->number_of_bilaterals);
				thrust::copy_n(data_container->host_data.rhs_data.begin() + data_container->number_of_contacts * 3, data_container->number_of_bilaterals, rhs_bilateral.begin());

				for (int i = 0; i < max_iteration; i += 4) {
					total_iteration += SolveAPGD(data_container->host_data.gam_data, data_container->host_data.rhs_data, 4);
					thrust::copy_n(
							data_container->host_data.gam_data.begin() + data_container->number_of_contacts * 3,
							data_container->number_of_bilaterals,
							data_container->host_data.gamma_bilateral.begin());

					SolveStab(data_container->host_data.gamma_bilateral, rhs_bilateral, 50);

					thrust::copy_n(
							data_container->host_data.gamma_bilateral.begin(),
							data_container->number_of_bilaterals,
							data_container->host_data.gam_data.begin() + data_container->number_of_contacts * 3);

					total_iteration += SolveAPGD(data_container->host_data.gam_data, data_container->host_data.rhs_data, 4);
				}
			} else {
				total_iteration += SolveAPGD(data_container->host_data.gam_data, data_container->host_data.rhs_data, max_iteration);
			}
		} else if (solver_type == BLOCK_JACOBI) {
			SolveJacobi();
		}
		thrust::copy_n(
				data_container->host_data.gam_data.begin() + data_container->number_of_contacts * 3,
				data_container->number_of_bilaterals,
				data_container->host_data.gamma_bilateral.begin());

		current_iteration = total_iteration;
		ComputeImpulses();
		timer_solver.stop();
		time_solver = timer_solver();
	}
}

void ChSolverGPU::VelocityStabilization(ChGPUDataManager *data_container_) {

}

uint ChSolverGPU::SolveStab(custom_vector<real> &x, const custom_vector<real> &b, const uint max_iter) {
	uint N = b.size();
	custom_vector<real> v(N, 0), v_hat(x.size()), w(N, 0), w_old, xMR, v_old, Av(x.size()), w_oold;
	real beta, c = 1, eta, norm_rMR, norm_r0, c_old = 1, s_old = 0, s = 0, alpha, beta_old, c_oold, s_oold, r1_hat, r1, r2, r3;
	ShurBilaterals(x,v_hat);
	v_hat = b - v_hat;
	beta = Norm(v_hat);
	w_old = w;
	eta = beta;
	xMR = x;
	norm_rMR = beta;
	norm_r0 = beta;

	if (beta == 0 || norm_rMR / norm_r0 < tolerance) {return 0;}

	for (current_iteration = 0; current_iteration < max_iter; current_iteration++) {
		//// Lanczos
		v_old = v;
		v = 1.0 / beta * v_hat;
		ShurBilaterals(v,Av);
		alpha = Dot(v, Av);
		v_hat = Av - alpha * v - beta * v_old;
		beta_old = beta;
		beta = Norm(v_hat);
		//// QR factorization
		c_oold = c_old;
		c_old = c;
		s_oold = s_old;
		s_old = s;
		r1_hat = c_old * alpha - c_oold * s_old * beta_old;
		r1 = 1 / sqrt(r1_hat * r1_hat + beta * beta);
		r2 = s_old * alpha + c_oold * c_old * beta_old;
		r3 = s_oold * beta_old;
		//// Givens Rotation
		c = r1_hat * r1;
		s = beta * r1;
		//// update
		w_oold = w_old;
		w_old = w;
		w = r1 * (v - r3 * w_oold - r2 * w_old);
		x = x + c * eta * w;
		norm_rMR = norm_rMR * abs(s);
		eta = -s * eta;
		residual = norm_rMR / norm_r0;

		if (residual < tolerance) {break;}
	}

	return current_iteration;

}

//
//__host__ __device__ void function_InvNDiag(uint index, int b1, int b2, int2* ids, real3* JXYZA, real3* JXYZB, real3* JUVWA, real3* JUVWB, real * inv_mass, real3 * inv_inertia, real & inv_n_diag) {
//
//  real Jx1 = JXYZA[index].x;
//  real Jy1 = JXYZA[index].y;
//  real Jz1 = JXYZA[index].z;
//  real Ju1 = JUVWA[index].x;
//  real Jv1 = JUVWA[index].y;
//  real Jw1 = JUVWA[index].z;
//  //
//  real Jx2 = JXYZB[index].x;
//  real Jy2 = JXYZB[index].y;
//  real Jz2 = JXYZB[index].z;
//  real Ju2 = JUVWB[index].x;
//  real Jv2 = JUVWB[index].y;
//  real Jw2 = JUVWB[index].z;
//  //
//  real m1 = inv_mass[b1];
//  real m2 = inv_mass[b2];
//  //
//  real3 i1 = inv_inertia[b1];
//  real3 i2 = inv_inertia[b2];
//
//  real part1 = dot(JXYZA[index], JXYZA[index]) * m1;
//  real part2 = dot(JUVWA[index] * JUVWA[index], i1);
//  real part3 = dot(JXYZB[index], JXYZB[index]) * m2;
//  real part4 = dot(JUVWB[index] * JUVWB[index], i2);
//  inv_n_diag = (part1 + part2 + part3 + part4);
//}
