#include "hip/hip_runtime.h"
#include "ChSolverGPU.h"
using namespace chrono;

uint ChSolverGPU::SolveAPGD(custom_vector<real> &x, const custom_vector<real> &b, const uint max_iter) {
    real gdiff = 0.000001;
    custom_vector<real> zvec(x.size());
    Thrust_Fill(zvec,0.0);
    
    real lastgoodres=10e30;
    real theta_k=1.0;
    real theta_k1=theta_k;
    real beta_k1=0.0;
    custom_vector<real> ms, mg_tmp2, mb_tmp(x.size()), d01(x.size());
    custom_vector<real> mg_tmp(x.size()), mg_tmp1(x.size());
    custom_vector<real> mg = ShurProduct(x) - b;

    Thrust_Fill(d01, -1.0);
    real L_k = Norm(ShurProduct(d01)) / Norm(d01);
    real t_k = 1 / L_k;
    cout << "L_k:" << L_k << " t_k:" << t_k << "\n";
    custom_vector<real>  my = x;
    custom_vector<real>  mx = x;
    custom_vector<real>  ml = x;
    custom_vector<real>  ml_candidate = x;
    
    real obj1=0.0;
    real obj2=0.0;
    
    for (current_iteration = 0; current_iteration < max_iter; current_iteration++) {
    	mg_tmp1 = ShurProduct(my);
        mg = mg_tmp1-b;
        SEAXPY(-t_k, mg, my, mx);   // mx = my + mg*(-t_k);
        Project(mx);
        
        mg_tmp = ShurProduct(mx);
        mg_tmp2 = mg_tmp-b;
        //mg_tmp = 0.5*mg_tmp-b;
        obj1 = Dot(mx, 0.5*mg_tmp-b);
        obj2 = Dot(my, 0.5*mg_tmp1-b);
        ms = mx - my;
        while (obj1 > obj2 + Dot(mg, ms) + 0.5 * L_k * pow(Norm(ms), 2.0)) {
            L_k = 2 * L_k;
            t_k = 1 / L_k;
            SEAXPY(-t_k, mg, my, mx); // mx = my + mg*(-t_k);
            Project(mx);
            
            mg_tmp = ShurProduct(mx);
        	mg_tmp2 = mg_tmp-b;
            obj1 = Dot(mx, 0.5*mg_tmp-b);
            ms = mx - my;

            cout << "APGD halving stepsize at it " << current_iteration << ", now t_k=" << t_k << "\n";
        }
        theta_k1 = (-pow(theta_k, 2) + theta_k * sqrt(pow(theta_k, 2) + 4)) / 2.0;
        beta_k1 = theta_k * (1.0 - theta_k) / (pow(theta_k, 2) + theta_k1);
        ms = mx - ml;
        my = mx + beta_k1 * ms;
        if (Dot(mg, ms) > 0) {
            my = mx;
            theta_k1 = 1.0;
            cout << "Restarting APGD at it " << current_iteration  << "\n";
        }
        L_k = 0.9 * L_k;
        t_k = 1 / L_k;
        
        ml = mx;
        theta_k = theta_k1;
        
       	//this is res1
       	real g_proj_norm=CompRes(mg_tmp2,number_of_contacts);
        
        //this is res4
        //SEAXPY(-gdiff, mg_tmp2, x, mb_tmp); //mb_tmp=x+mg_tmp2*(-gdiff)
        //Project(mb_tmp);
        //d01=mb_tmp-x;
        //mb_tmp = (-1.0/gdiff)*d01;
        //real g_proj_norm = Norm(mb_tmp);
        
        if(g_proj_norm < lastgoodres) {
        	lastgoodres = g_proj_norm;
        	ml_candidate = ml;
        }
        
        residual=lastgoodres;
        if (residual < tolerance) {
            break;
        }
    }
    x=ml_candidate;
    return current_iteration;
}
