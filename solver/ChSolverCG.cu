#include "hip/hip_runtime.h"
#include "ChSolverGPU.h"
using namespace chrono;


uint ChSolverGPU::SolveCG(custom_vector<real> &x, const custom_vector<real> &b, const uint max_iter) {
    custom_vector<real> r(x.size()), p, Ap;
    real rsold, alpha, rsnew = 0, normb = Norm(b);
    if (normb == 0.0) {
        normb = 1;
    }
    p = r = b - ShurProduct(x);
    rsold = Dot(r, r);
    normb = 1.0 / normb;
    if (sqrt(rsold) * normb <= tolerance) {
        return 0;
    }
    for (current_iteration = 0; current_iteration < max_iter; current_iteration++) {
        Ap = ShurProduct(p);
        alpha = rsold / Dot(p, Ap);
        rsnew = 0;
#ifdef SIM_ENABLE_GPU_MODE
        SEAXPY(alpha, p, x, x);
        SEAXPY(-alpha, Ap, r, r);
        rsnew=Dot(r,r);
#else
        #pragma omp parallel for reduction(+:rsnew)
        for (int i = 0; i < x.size(); i++) {
            x[i] = x[i] + alpha * p[i];
            r[i] = r[i] - alpha * Ap[i];
            rsnew += r[i] * r[i];
        }
#endif
        residual = sqrtf(rsnew) * normb;
        if (residual < tolerance) {
            break;
        }
        SEAXPY(rsnew / rsold, p, r, p); //p = r + rsnew / rsold * p;
        rsold = rsnew;
    }
    Project(x);
    return current_iteration;
}
