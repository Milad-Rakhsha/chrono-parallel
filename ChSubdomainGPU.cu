#include "hip/hip_runtime.h"
#include "ChSubdomainGPU.h"
#include <thrust/copy.h>
using namespace chrono;

void ChSubdomainGPU::Collide(ChGPUDataManager *data_manager) {
	gpu_data_manager = data_manager;
	mod_list.resize(gpu_data_manager->number_of_models, 0);
	bod_list.resize(gpu_data_manager->number_of_objects, 0);
	for (int i = 0; i < gpu_data_manager->number_of_models; i++) {
		float3 minP = gpu_data_manager->host_aabb_data[i];
		float3 maxP = gpu_data_manager->host_aabb_data[i + gpu_data_manager->number_of_models];

		if ((minP.x <= max_bounding_point.x && min_bounding_point.x <= maxP.x) && (minP.y <= max_bounding_point.y && min_bounding_point.y <= maxP.y)
				&& (minP.z <= max_bounding_point.z && min_bounding_point.z <= maxP.z)) {
			mod_list[i] = true;
			bod_list[gpu_data_manager->host_typ_data[i].z] = true;
		}

	}
	number_of_models = Thrust_Total(mod_list);
	number_of_objects = Thrust_Total(bod_list);
}
struct pred {
		__host__ __device__
		bool operator()(int x) {
			return bool(x);
		}
};
void ChSubdomainGPU::Copy(int i) {
	subdiv_id = i;

	host_ObA_data.resize(number_of_models);
	host_ObB_data.resize(number_of_models);
	host_ObC_data.resize(number_of_models);
	host_ObR_data.resize(number_of_models);
	host_fam_data.resize(number_of_models);
	host_typ_data.resize(number_of_models);

	thrust::copy_if(gpu_data_manager->host_ObA_data.begin(), gpu_data_manager->host_ObA_data.end(), mod_list.begin(), host_ObA_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_ObB_data.begin(), gpu_data_manager->host_ObB_data.end(), mod_list.begin(), host_ObB_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_ObC_data.begin(), gpu_data_manager->host_ObC_data.end(), mod_list.begin(), host_ObC_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_ObR_data.begin(), gpu_data_manager->host_ObR_data.end(), mod_list.begin(), host_ObR_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_fam_data.begin(), gpu_data_manager->host_fam_data.end(), mod_list.begin(), host_fam_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_typ_data.begin(), gpu_data_manager->host_typ_data.end(), mod_list.begin(), host_typ_data.begin(), pred());

	gpu_data.device_ObA_data = host_ObA_data;
	gpu_data.device_ObB_data = host_ObB_data;
	gpu_data.device_ObC_data = host_ObC_data;
	gpu_data.device_ObR_data = host_ObR_data;
	gpu_data.device_fam_data = host_fam_data;
	gpu_data.device_typ_data = host_typ_data;

	host_vel_data.resize(number_of_objects);
	host_omg_data.resize(number_of_objects);
	host_pos_data.resize(number_of_objects);
	host_rot_data.resize(number_of_objects);
	host_inr_data.resize(number_of_objects);
	host_frc_data.resize(number_of_objects);
	host_trq_data.resize(number_of_objects);
	host_acc_data.resize(number_of_objects);
	host_aux_data.resize(number_of_objects);
	host_lim_data.resize(number_of_objects);

	thrust::copy_if(gpu_data_manager->host_vel_data.begin(), gpu_data_manager->host_vel_data.end(), bod_list.begin(), host_vel_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_omg_data.begin(), gpu_data_manager->host_omg_data.end(), bod_list.begin(), host_omg_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_pos_data.begin(), gpu_data_manager->host_pos_data.end(), bod_list.begin(), host_pos_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_rot_data.begin(), gpu_data_manager->host_rot_data.end(), bod_list.begin(), host_rot_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_inr_data.begin(), gpu_data_manager->host_inr_data.end(), bod_list.begin(), host_inr_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_frc_data.begin(), gpu_data_manager->host_frc_data.end(), bod_list.begin(), host_frc_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_trq_data.begin(), gpu_data_manager->host_trq_data.end(), bod_list.begin(), host_trq_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_acc_data.begin(), gpu_data_manager->host_acc_data.end(), bod_list.begin(), host_acc_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_aux_data.begin(), gpu_data_manager->host_aux_data.end(), bod_list.begin(), host_aux_data.begin(), pred());
	thrust::copy_if(gpu_data_manager->host_lim_data.begin(), gpu_data_manager->host_lim_data.end(), bod_list.begin(), host_lim_data.begin(), pred());

	gpu_data.device_vel_data = host_vel_data;
	gpu_data.device_omg_data = host_omg_data;
	gpu_data.device_pos_data = host_pos_data;
	gpu_data.device_rot_data = host_rot_data;
	gpu_data.device_inr_data = host_inr_data;
	gpu_data.device_frc_data = host_frc_data;
	gpu_data.device_trq_data = host_trq_data;
	gpu_data.device_acc_data = host_vel_data;
	gpu_data.device_aux_data = host_aux_data;
	gpu_data.device_lim_data = host_lim_data;

	//gpu_data_manager->gpu_data[i].device_bilateral_data = gpu_data_manager->host_bilateral_data;

}

void ChSubdomainGPU::Run(ChLcpIterativeSolverGPUsimple* LCP_solver_speed) {

	float3 bin_size_vec;
	float max_dimension;
	float collision_envelope = 0;
	number_of_bilaterals = 0;

	ChCCollisionGPU::ComputeAABB(gpu_data_manager->gpu_data);
	ChCCollisionGPU::ComputeBounds(gpu_data_manager->gpu_data);
	ChCCollisionGPU::UpdateAABB(bin_size_vec, max_dimension, collision_envelope, gpu_data_manager->gpu_data);
	ChCCollisionGPU::Broadphase(bin_size_vec, gpu_data_manager->gpu_data);
	ChCCollisionGPU::Narrowphase(gpu_data_manager->gpu_data);

	LCP_solver_speed->SolveSys(gpu_data);
}
