#include "hip/hip_runtime.h"
#include "ChLcpIterativeSolverGPU.h"

using namespace chrono;

//helper functions
template<class T, class U> // dot product of the first three elements of float3/float4 values
inline __host__ __device__ float dot3(const T & a, const U & b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__constant__ float lcp_omega_bilateral_const;
__constant__ float lcp_omega_contact_const;
__constant__ unsigned int number_of_bodies_const;
__constant__ unsigned int number_of_contacts_const;
__constant__ unsigned int number_of_bilaterals_const;
__constant__ unsigned int number_of_updates_const;
__constant__ float force_factor_const; // usually, the step size
__constant__ float negated_recovery_speed_const;
__constant__ float c_factor_const; // usually 1/dt
__constant__ float step_size_const;

////////////////////////////////////////////////////////////////////////////////////////////////////
// Creates a quaternion as a function of a vector of rotation and an angle (the vector is assumed already
// normalized, and angle is in radians).

inline __host__                        __device__ float4 Quat_from_AngAxis(const float &angle, const float3 & v) {
	float sinhalf = sinf(angle * 0.5f);
	float4 quat;
	quat.x = cosf(angle * 0.5f);
	quat.y = v.x * sinhalf;
	quat.z = v.y * sinhalf;
	quat.w = v.z * sinhalf;
	return quat;
}

/// The quaternion becomes the quaternion product of the two quaternions A and B:
/// following the classic Hamilton rule:  this=AxB
/// This is the true, typical quaternion product. It is NOT commutative.

inline __host__             __device__ float4 Quaternion_Product(const float4 &qa, const float4 &qb) {
	float4 quat;
	quat.x = qa.x * qb.x - qa.y * qb.y - qa.z * qb.z - qa.w * qb.w;
	quat.y = qa.x * qb.y + qa.y * qb.x - qa.w * qb.z + qa.z * qb.w;
	quat.z = qa.x * qb.z + qa.z * qb.x + qa.w * qb.y - qa.y * qb.w;
	quat.w = qa.x * qb.w + qa.w * qb.x - qa.z * qb.y + qa.y * qb.z;
	return quat;
}

__host__ __device__ void inline Compute_Mat(float3 &A, float3 &B, float3 &C, float4 TT, const float3 &n, const float3 &u, const float3 &w, const float3 &pos) {
	float t00 = pos.z * n.y - pos.y * n.z;
	float t01 = TT.x * TT.x;
	float t02 = TT.y * TT.y;
	float t03 = TT.z * TT.z;
	float t04 = TT.w * TT.w;
	float t05 = t01 + t02 - t03 - t04;
	float t06 = -pos.z * n.x + pos.x * n.z;
	float t07 = TT.y * TT.z;
	float t08 = TT.x * TT.w;
	float t09 = t07 + t08;
	float t10 = pos.y * n.x - pos.x * n.y;
	float t11 = TT.y * TT.w;
	float t12 = TT.x * TT.z;
	float t13 = t11 - t12;
	float t14 = t07 - t08;
	float t15 = t01 - t02 + t03 - t04;
	float t16 = TT.z * TT.w;
	float t17 = TT.x * TT.y;
	float t18 = t16 + t17;
	float t19 = t11 + t12;
	float t20 = t16 - t17;
	float t21 = t01 - t02 - t03 + t04;
	float t22 = pos.z * u.y - pos.y * u.z;
	float t23 = -pos.z * u.x + pos.x * u.z;
	float t24 = pos.y * u.x - pos.x * u.y;
	float t25 = pos.z * w.y - pos.y * w.z;
	float t26 = -pos.z * w.x + pos.x * w.z;
	float t27 = pos.y * w.x - pos.x * w.y;
	A.x = t00 * t05 + 2 * t06 * t09 + 2 * t10 * t13;
	A.y = 2 * t00 * t14 + t06 * t15 + 2 * t10 * t18;
	A.z = 2 * t00 * t19 + 2 * t06 * t20 + t10 * t21;
	B.x = t22 * t05 + 2 * t23 * t09 + 2 * t24 * t13;
	B.y = 2 * t22 * t14 + t23 * t15 + 2 * t24 * t18;
	B.z = 2 * t22 * t19 + 2 * t23 * t20 + t24 * t21;
	C.x = t25 * t05 + 2 * t26 * t09 + 2 * t27 * t13;
	C.y = 2 * t25 * t14 + t26 * t15 + 2 * t27 * t18;
	C.z = 2 * t25 * t19 + 2 * t26 * t20 + t27 * t21;
}
// 	Kernel for a single iteration of the LCP over all contacts
//   	Version 2.0 - Tasora
//	Version 2.2- Hammad (optimized, cleaned etc)
__global__ void LCP_Iteration_Contacts(float3* norm, float3* ptA, float3* ptB, float* contactDepth, int2* ids, float3* G, float* dG, float3* aux, float3* inertia, float4* rot, float3* vel,
		float3* omega, float3* pos, float3* updateV, float3* updateO, uint* offset) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_contacts_const) {
		return;
	}
	//if(dG[i]<1e-8){dG[i]=0;return;}
	float4 E1, E2;
	float3 vB, gamma, N, U, W, T3, T4, T5, T6, T7, T8, gamma_old, sbar, B1, B2, W1, W2, aux1, aux2;
	float reg, mu, eta;
	//long long id=ids[i];
	int2 temp_id = ids[i];
	reg = -fabs(contactDepth[i]);//c_factor_const;							//Scale contact distance, cfactor is usually 1
	int B1_i = temp_id.x;
	int B2_i = temp_id.y;
	B1 = vel[B1_i];
	B2 = vel[B2_i];
	aux1 = aux[B1_i];
	aux2 = aux[B2_i];
	N = norm[i]; //assume: normalized, and if depth=0 norm=(1,0,0)
	W = fabs(N); //Gramm Schmidt; work with the global axis that is "most perpendicular" to the contact normal vector;effectively this means looking for the smallest component (in abs) and using the corresponding direction to carry out cross product.
	U = F3(0.0, N.z, -N.y); //it turns out that X axis is closest to being perpendicular to contact vector;
	if (W.x > W.y) {
		U = F3(-N.z, 0.0, N.x);
	} //it turns out that Y axis is closest to being perpendicular to contact vector;
	if (W.y > W.z) {
		U = F3(N.y, -N.x, 0.0);
	} //it turns out that Z axis is closest to being perpendicular to contact vector;
	U = normalize(U); //normalize the local contact Y,Z axis
	W = cross(N, U); //carry out the last cross product to find out the contact local Z axis : multiply the contact normal by the local Y component
	float normm = dot(N, ((B2 - B1)));
	//reg=(reg>normm)? reg:normm;

	normm = (normm > 0) ? 0 : normm;
	//reg=min(0.0,max(reg,-1.));
	reg = min((reg + normm), (-step_size_const));
	//if(reg>0){printf("REG: %f\n",reg);}
	sbar = ptA[i] - pos[B1_i]; //Contact Point on A - Position of A
	E1 = rot[B1_i]; //bring in the Euler parameters associated with body 1;
	Compute_Mat(T3, T4, T5, E1, N, U, W, sbar); //A_i,p'*A_A*(sbar~_i,A)

	sbar = ptB[i] - pos[B2_i]; //Contact Point on B - Position of B
	E2 = rot[B2_i]; //bring in the Euler parameters associated with body 2;
	Compute_Mat(T6, T7, T8, E2, N, U, W, sbar); //A_i,p'*A_B*(sbar~_i,B)
	T6 = -T6;
	T7 = -T7;
	T8 = -T8;
	//if(i==0){printf("%f %f %f | %f %f %f | %f %f %f\n",T3.x,T3.y,T3.z,T4.x,T4.y,T4.z,T5.x,T5.y,T5.z);}

	W1 = omega[B1_i];
	W2 = omega[B2_i];

	mu = (aux1.y + aux2.y) * .5;
	gamma.x = dot3(T3, W1) - dot3(N, B1) + dot3(T6, W2) + dot3(N, B2) + reg; //+bi
	gamma.y = dot3(T4, W1) - dot3(U, B1) + dot3(T7, W2) + dot3(U, B2);
	gamma.z = dot3(T5, W1) - dot3(W, B1) + dot3(T8, W2) + dot3(W, B2);
	B1 = inertia[B1_i]; // bring in the inertia attributes; to be used to compute \eta
	B2 = inertia[B2_i]; // bring in the inertia attributes; to be used to compute \eta
	eta = dot3(T3 * T3, B1) + dot3(T4 * T4, B1) + dot3(T5 * T5, B1); // update expression of eta
	eta += dot3(T6 * T6, B2) + dot3(T7 * T7, B2) + dot3(T8 * T8, B2);
	eta += (dot(N, N) + dot(U, U) + dot(W, W)) * (aux1.z + aux2.z); // multiply by inverse of mass matrix of B1 and B2, add contribution from mass and matrix A_c.
	eta = 1.0 / eta; // final value of eta

	gamma *= lcp_omega_contact_const * eta; // perform gamma *= omega*eta
	gamma_old = G[i];
	gamma = gamma_old - gamma; // perform gamma = gamma_old - gamma ;  in place.
	/// ---- perform projection of 'a8' onto friction cone  --------
	reg = sqrtf(gamma.y * gamma.y + gamma.z * gamma.z); // reg = f_tang


	if (reg > (mu * gamma.x)) { // inside upper cone? keep untouched!
		if ((mu * reg) < -gamma.x) { // inside lower cone? reset  normal,u,v to zero!
			gamma = F3(0.f, 0.f, 0.f);
		} else { // remaining case: project orthogonally to generator segment of upper cone
			gamma.x = (reg * mu + gamma.x) / (mu * mu + 1.f);
			reg = (gamma.x * mu) / reg; //  reg = tproj_div_t
			gamma.y *= reg;
			gamma.z *= reg;
		}
	}
	G[i] = gamma; // store gamma_new
	gamma -= gamma_old; // compute delta_gamma = gamma_new - gamma_old   = delta_gamma.
	dG[i] = length(gamma);
	//if(i==0){printf("%f %f %f\n",gamma.x,gamma.y,gamma.z);}
	vB = N * gamma.x + U * gamma.y + W * gamma.z;
	int offset1 = offset[i];
	int offset2 = offset[i + number_of_contacts_const];
	if (aux1.x == 1) {
		updateV[offset1] = -vB * aux1.z; // compute and store dv1
		updateO[offset1] = (T3 * gamma.x + T4 * gamma.y + T5 * gamma.z) * B1; // compute dw1 =  Inert.1' * J1w^ * deltagamma  and store  dw1
	}
	if (aux2.x == 1) {
		updateV[offset2] = vB * aux2.z; // compute and store dv2
		updateO[offset2] = (T6 * gamma.x + T7 * gamma.y + T8 * gamma.z) * B2; // compute dw2 =  Inert.2' * J2w^ * deltagamma  and store  dw2
	}
}
///////////////////////////////////////////////////////////////////////////////////
// Kernel for a single iteration of the LCP over all scalar bilateral contacts
// (a bit similar to the ChKernelLCPiteration above, but without projection etc.)
// Version 2.0 - Tasora
//
__global__ void LCP_Iteration_Bilaterals(CH_REALNUMBER4* bilaterals, float3* aux, float3* inertia, float4* rot, float3* vel, float3* omega, float3* pos, float3* updateV, float3* updateO,
		uint* offset, float * dG) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_bilaterals_const) {
		return;
	}
	float4 vA;
	float3 vB;
	float gamma_new = 0, gamma_old;
	int B1_index = 0, B2_index = 0;
	B1_index = bilaterals[i].w;
	B2_index = bilaterals[i + number_of_bilaterals_const].w;
	float3 aux1 = aux[B1_index];
	float3 aux2 = aux[B2_index];
	// ---- perform   gamma_new = ([J1 J2] {v1 | v2}^ + b) 
	vA = bilaterals[i]; // line 0
	vB = vel[B1_index]; // v1
	gamma_new += dot3(vA, vB);

	vA = bilaterals[i + 2 * number_of_bilaterals_const];// line 2
	vB = omega[B1_index]; // w1
	gamma_new += dot3(vA, vB);

	vA = bilaterals[i + number_of_bilaterals_const]; // line 1
	vB = vel[B2_index]; // v2
	gamma_new += dot3(vA, vB);

	vA = bilaterals[i + 3 * number_of_bilaterals_const];// line 3
	vB = omega[B2_index]; // w2
	gamma_new += dot3(vA, vB);

	vA = bilaterals[i + 4 * number_of_bilaterals_const]; // line 4   (eta, b, gamma, 0)
	gamma_new += vA.y; // add known term     + b
	gamma_old = vA.z; // old gamma
	/// ---- perform gamma_new *= omega/g_i
	gamma_new *= lcp_omega_bilateral_const; // lcp_omega_const is in constant memory
	gamma_new *= vA.x; // eta = 1/g_i;
	/// ---- perform gamma_new = gamma_old - gamma_new ; in place.
	gamma_new = gamma_old - gamma_new;
	/// ---- perform projection of 'a' (only if simple unilateral behavior C>0 is requested)
	if (vA.w && gamma_new < 0.) {
		gamma_new = 0.;
	}
	// ----- store gamma_new
	vA.z = gamma_new;
	bilaterals[i + 4 * number_of_bilaterals_const] = vA;
	/// ---- compute delta in multipliers: gamma_new = gamma_new - gamma_old   = delta_gamma    , in place.
	gamma_new -= gamma_old;
	dG[number_of_contacts_const + i] = (gamma_new);
	/// ---- compute dv1 =  invInert.18 * J1^ * deltagamma
	vB = inertia[B1_index]; // iJ iJ iJ im
	vA = (bilaterals[i]) * aux1.z * gamma_new; // line 0: J1(x)
	int offset1 = offset[2 * number_of_contacts_const + i];
	int offset2 = offset[2 * number_of_contacts_const + i + number_of_bilaterals_const];
	updateV[offset1] = F3(vA);//  ---> store  v1 vel. in reduction buffer
	updateO[offset1] = F3(bilaterals[i + 2 * number_of_bilaterals_const]) * vB * gamma_new;// line 2:  J1(w)// ---> store  w1 vel. in reduction buffer
	vB = inertia[B2_index]; // iJ iJ iJ im
	vA = (bilaterals[i + number_of_bilaterals_const]) * aux2.z * gamma_new; // line 1: J2(x)
	updateV[offset2] = F3(vA);//  ---> store  v2 vel. in reduction buffer
	updateO[offset2] = F3(bilaterals[i + 3 * number_of_bilaterals_const]) * vB * gamma_new;// line 3:  J2(w)// ---> store  w2 vel. in reduction buffer
}

__device__ __host__ inline float4 computeRot_dt(float3 & omega, float4 &rot) {
	return mult(F4(0, omega.x, omega.y, omega.z), rot) * .5;
}
__device__ __host__ float3 RelPoint_AbsSpeed(float3 & vel, float3 & omega, float4 & rot, float3 &point) {
	float4 q = mult(computeRot_dt(omega, rot), mult(F4(0, point.x, point.y, point.z), inv(rot)));
	return vel + ((F3(q.y, q.z, q.w)) * 2);
}
__global__ void Warm_Contacts(float3* norm, float3* ptA, float3* ptB, float* contactDepth, int2* ids, float3* aux, float3* inertia, float4* rot, float3* vel, float3* omega, float3* pos, float3 *gamma) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_contacts_const) {
		return;
	}
	if (length(gamma[i]) != 0) {
		return;
	}
	float mkn = 1000, mgn = 200, mgt = 200, mkt = 100;
	//long long id=ids[i];
	int2 temp_id = ids[i];
	int B1_i = int(temp_id.x);
	int B2_i = int(temp_id.y);

	float3 vN = norm[i]; //normal
	float3 pA = ptA[i];
	float3 pB = ptB[i];
	float3 B1 = vel[B1_i]; //vel B1
	float3 B2 = vel[B2_i]; //vel B2
	float3 oA = omega[B1_i];
	float3 oB = omega[B2_i];
	float4 rotA = rot[B1_i];
	float4 rotB = rot[B2_i];
	float3 aux1 = aux[B1_i];
	float3 aux2 = aux[B2_i];
	float3 posA = pos[B1_i];
	float3 posB = pos[B2_i];
	float3 f_n = mkn * -fabs(contactDepth[i]) * vN;
	//float3 local_pA = quatRotate(pA - posA, inv(rotA));
	//float3 local_pB = quatRotate(pB - posB, inv(rotB));

	//float3 v_BA = (B1-B2);//(RelPoint_AbsSpeed(B2, oB, rotB, local_pB)) - (RelPoint_AbsSpeed(B1, oA, rotA, local_pA));
	//float3 v_n = normalize(dot(v_BA, vN) * vN);
	//float m_eff = (1.0 / aux1.z) * (1.0 / aux2.z) / (1.0 / aux1.z + 1.0 / aux2.z);
	//f_n += mgn * m_eff * v_n;

	//float mu = (aux1.y + aux2.y) * .5;
	//float3 v_t = v_BA - v_n;

	//float3 f_t = (mgt * m_eff * v_t) + (mkt * (v_t * step_size_const));

	//if (length(f_t) > mu * length(f_n)) {
	//	f_t *= mu * length(f_n) / length(f_t);
	//}

	float3 f_r = f_n ;//+ f_t;
	f_r *= step_size_const;

	gamma[i] = f_r;

}
__global__ void DEM_Contacts(float3* norm, float3* ptA, float3* ptB, float* contactDepth, int2* ids, float3* aux, float3* inertia, float4* rot, float3* vel, float3* omega, float3* pos,
		float3* updateV, float3* updateO, uint* offset) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_contacts_const) {
		return;
	}
	float mkn = 3924, mgn = 420, mgt = 420, mkt = 2.0f / 7.0f * 3924;
	//long long id=ids[i];
	int2 temp_id = ids[i];
	int B1_i = int(temp_id.x);
	int B2_i = int(temp_id.y);

	float3 vN = norm[i]; //normal
	float3 pA = ptA[i];
	float3 pB = ptB[i];
	float3 B1 = vel[B1_i]; //vel B1
	float3 B2 = vel[B2_i]; //vel B2
	float3 oA = omega[B1_i];
	float3 oB = omega[B2_i];
	float4 rotA = rot[B1_i];
	float4 rotB = rot[B2_i];
	float3 aux1 = aux[B1_i];
	float3 aux2 = aux[B2_i];
	float3 posA = pos[B1_i];
	float3 posB = pos[B2_i];
	float3 f_n = mkn * -fabs(contactDepth[i]) * vN;
	float3 local_pA = quatRotate(pA - posA, inv(rotA));
	float3 local_pB = quatRotate(pB - posB, inv(rotB));

	float3 v_BA = (RelPoint_AbsSpeed(B2, oB, rotB, local_pB)) - (RelPoint_AbsSpeed(B1, oA, rotA, local_pA));
	float3 v_n = normalize(dot(v_BA, vN) * vN);
	float m_eff = (1.0 / aux1.z) * (1.0 / aux2.z) / (1.0 / aux1.z + 1.0 / aux2.z);
	f_n += mgn * m_eff * v_n;

	float mu = (aux1.y + aux2.y) * .5;
	float3 v_t = v_BA - v_n;

	float3 f_t = (mgt * m_eff * v_t) + (mkt * (v_t * step_size_const));

	if (length(f_t) > mu * length(f_n)) {
		f_t *= mu * length(f_n) / length(f_t);
	}

	float3 f_r = f_n + f_t;

	int offset1 = offset[i];
	int offset2 = offset[i + number_of_contacts_const];

	float3 force1_loc = quatRotate(f_r, inv(rotA));
	float3 force2_loc = quatRotate(f_r, inv(rotB));

	float3 trq1 = cross(local_pA, force1_loc);
	float3 trq2 = cross(local_pB, -force2_loc);

	f_r *= step_size_const;

	updateV[offset1] = (f_r) * aux1.z;
	updateV[offset2] = (f_r) * -aux2.z;

	updateO[offset1] = (trq1 * step_size_const) * (inertia[B1_i]);
	updateO[offset2] = (trq2 * step_size_const) * (inertia[B2_i]);
}

////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for adding invmass*force*step_size_const to body speed vector.
// This kernel must be applied to the stream of the body buffer.

__global__ void ChKernelLCPaddForces(float3* aux, float3* inertia, float3* forces, float3* torques, float3* vel, float3* omega) {
	// Compute the i values used to access data inside the large 
	// array using pointer arithmetic.
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < number_of_bodies_const) {
		float3 temp_aux = aux[i];
		if (temp_aux.x != 0) {
			float3 mF, minvMasses = inertia[i];
			// v += m_inv * h * f
			mF = forces[i]; // vector with f (force)
			mF *= step_size_const;
			mF *= temp_aux.z;
			vel[i] += mF;
			// w += J_inv * h * c
			mF = torques[i]; // vector with f (torque)
			mF *= step_size_const;
			mF.x *= minvMasses.x;
			mF.y *= minvMasses.y;
			mF.z *= minvMasses.z;
			omega[i] += mF;
		}
	}
}
////////////////////////////////////////////////////////////////////////////////////////////////////
// Updates the speeds in the body buffer with values accumulated in the
// reduction buffer:   V_new = V_old + delta_speeds
__device__ __host__ uint nearest_pow(uint num) {
	uint n = num > 0 ? num - 1 : 0;

	n |= n >> 1;
	n |= n >> 2;
	n |= n >> 4;
	n |= n >> 8;
	n |= n >> 16;
	n++;

	return n;
}
__global__ void LCP_Reduce_Speeds(float3* aux, float3* vel, float3* omega, float3* updateV, float3* updateO, uint* d_body_num, uint* counter) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_updates_const) {
		return;
	}
	int start = (i == 0) ? 0 : counter[i - 1], end = counter[i];
	int id = d_body_num[end - 1], j;
	if (aux[id].x == 1) {
		float3 mUpdateV = F3(0);
		float3 mUpdateO = F3(0);
		for (j = 0; j < end - start; j++) {
			mUpdateV += updateV[j + start];
			mUpdateO += updateO[j + start];
		}
		vel[id] += mUpdateV;
		omega[id] += mUpdateO;
	}
}
//  Kernel for performing the time step integration (with 1st o;rder Eulero)
//  on the body data stream.
//
//  number of registers: 12 (suggested 320 threads/block)

__global__ void LCP_Integrate_Timestep(float3* aux, float3* acc, float4* rot, float3* vel, float3* omega, float3* pos, float3* lim) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_bodies_const) {
		return;
	}
	float3 velocity = vel[i];
	float3 aux1 = aux[i];
	if (aux1.x == 0) {
		return;
	}

	pos[i] = pos[i] + velocity * step_size_const; // Do 1st order integration of linear speeds

	//bodies[i] = velocity;
	// Do 1st order integration of quaternion position as q[t+dt] = qw_abs^(dt) * q[dt] = q[dt] * qw_local^(dt)
	// where qw^(dt) is the quaternion { cos(0.5|w|), wx/|w| sin(0.5|w|), wy/|w| sin(0.5|w|), wz/|w| sin(0.5|w|)}^dt
	// that is argument of sine and cosines are multiplied by dt.
	float3 omg = omega[i];
	float3 limits = lim[i];
	float wlen = sqrtf(dot3(omg, omg));
	//if (limits.x == 1) {
			float w = 2.0 * wlen;
			if (w > limits.z) {
				omg *= limits.z / w;
			}

			float v = length(velocity);
			if (v > limits.y) {
				velocity *= limits.y / v;
			}
			vel[i] = velocity;
			omega[i] = omg;
	//}



	float4 Rw = (fabs(wlen) > 10e-10) ? Quat_from_AngAxis(step_size_const * wlen, omg / wlen) : F4(1., 0, 0, 0);// to avoid singularity for near zero angular speed

	float4 mq = Quaternion_Product(rot[i], Rw);
	mq *= rsqrtf(dot(mq, mq));
	rot[i] = mq;
	acc[i] = (velocity - acc[i]) / step_size_const;
}
__global__ void LCP_ComputeGyro(float3* omega, float3* inertia, float3* gyro, float3* torque) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_bodies_const) {
		return;
	}
	float3 body_inertia = inertia[i];
	body_inertia = F3(1 / body_inertia.x, 1 / body_inertia.y, 1 / body_inertia.z);
	float3 body_omega = omega[i];
	float3 gyr = cross(body_omega, body_inertia * body_omega);
	gyro[i] = gyr;
	torque[i] -= gyr;

}

__global__ void ChKernelOffsets(int2* ids, CH_REALNUMBER4* bilaterals, uint* Body) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < number_of_contacts_const) {
		int2 temp_id = ids[i];
		Body[i] = temp_id.x;
		Body[i + number_of_contacts_const] = temp_id.y;
	}
	if (i < number_of_bilaterals_const) {
		Body[2 * number_of_contacts_const + i] = bilaterals[i].w;
		Body[2 * number_of_contacts_const + i + number_of_bilaterals_const] = bilaterals[i + number_of_bilaterals_const].w;
	}
}
ChLcpIterativeSolverGPU::~ChLcpIterativeSolverGPU() {
	body_number.clear();
	update_number.clear();
	offset_counter.clear();
	update_offset.clear();
	device_bilateral_data.clear();
}
void ChLcpIterativeSolverGPU::WarmContact() {

	if (use_DEM == false) {
Warm_Contacts	<<< BLOCKS(number_of_contacts), THREADS >>>(
			CASTF3(data_container->device_norm_data),
			CASTF3(data_container->device_cpta_data),
			CASTF3(data_container->device_cptb_data),
			CASTF1(data_container->device_dpth_data),
			CASTI2(data_container->device_bids_data),
			CASTF3(data_container->device_aux_data),
			CASTF3(data_container->device_inr_data),
			CASTF4(data_container->device_rot_data),
			CASTF3(data_container->device_vel_data),
			CASTF3(data_container->device_omg_data),
			CASTF3(data_container->device_pos_data),
			CASTF3(data_container->device_gam_data));
}
}
void ChLcpIterativeSolverGPU::RunTimeStep() {
	number_of_bodies = data_container->number_of_objects;
	number_of_contacts = data_container->number_of_contacts;
	number_of_constraints = number_of_contacts + number_of_bilaterals;

	data_container->device_gyr_data.resize(number_of_bodies);

	COPY_TO_CONST_MEM(c_factor);
	COPY_TO_CONST_MEM(negated_recovery_speed);
	COPY_TO_CONST_MEM(lcp_omega_bilateral);
	COPY_TO_CONST_MEM(lcp_omega_contact);
	COPY_TO_CONST_MEM(step_size);
	COPY_TO_CONST_MEM(number_of_contacts);
	COPY_TO_CONST_MEM(number_of_bilaterals);
	COPY_TO_CONST_MEM(number_of_bodies);

	LCP_ComputeGyro<<< BLOCKS(number_of_bodies),THREADS>>>(
			CASTF3(data_container->device_omg_data),
			CASTF3(data_container->device_inr_data),
			CASTF3(data_container->device_gyr_data),
			CASTF3(data_container->device_trq_data));

	ChKernelLCPaddForces<<< BLOCKS(number_of_bodies), THREADS >>>(
			CASTF3(data_container->device_aux_data),
			CASTF3(data_container->device_inr_data),
			CASTF3(data_container->device_frc_data),
			CASTF3(data_container->device_trq_data),
			CASTF3(data_container->device_vel_data),
			CASTF3(data_container->device_omg_data));

	if (number_of_constraints > 0) {

		device_bilateral_data = host_bilateral_data;

		update_number.resize((number_of_constraints) * 2, 0);
		offset_counter.resize((number_of_constraints) * 2, 0);
		update_offset.resize((number_of_constraints) * 2, 0);
		body_number.resize((number_of_constraints) * 2, 0);
		device_dgm_data.resize((number_of_constraints), (1));
		Thrust_Fill(device_dgm_data,1);
		vel_update.resize((number_of_constraints) * 2);
		omg_update.resize((number_of_constraints) * 2);

		ChKernelOffsets<<< BLOCKS(number_of_constraints),THREADS>>>(
				CASTI2(data_container->device_bids_data),
				CASTF4(device_bilateral_data),
				CASTU1(body_number));

		Thrust_Sequence(update_number);
		Thrust_Sequence(update_offset);
		Thrust_Fill(offset_counter,0);
		Thrust_Sort_By_Key(body_number, update_number);
		Thrust_Sort_By_Key(update_number,update_offset);
		thrust::device_vector<uint> body_number2 = body_number;
		Thrust_Reduce_By_KeyB(number_of_updates,body_number,update_number,offset_counter);
		Thrust_Inclusive_Scan(offset_counter);

		COPY_TO_CONST_MEM(number_of_updates);
		//WarmContact();
		for (iteration_number = 0; iteration_number < maximum_iterations; iteration_number++) {
			if (use_DEM == false) {

LCP_Iteration_Contacts			<<< BLOCKS(number_of_contacts), THREADS >>>(
					CASTF3(data_container->device_norm_data),
					CASTF3(data_container->device_cpta_data),
					CASTF3(data_container->device_cptb_data),
					CASTF1(data_container->device_dpth_data),
					CASTI2(data_container->device_bids_data),
					CASTF3(data_container->device_gam_data),
					CASTF1(device_dgm_data),
					CASTF3(data_container->device_aux_data),
					CASTF3(data_container->device_inr_data),
					CASTF4(data_container->device_rot_data),
					CASTF3(data_container->device_vel_data),
					CASTF3(data_container->device_omg_data),
					CASTF3(data_container->device_pos_data),
					CASTF3(vel_update),
					CASTF3(omg_update),
					CASTU1(update_offset));
		} else {
			DEM_Contacts<<< BLOCKS(number_of_contacts), THREADS >>>(
					CASTF3(data_container->device_norm_data),
					CASTF3(data_container->device_cpta_data),
					CASTF3(data_container->device_cptb_data),
					CASTF1(data_container->device_dpth_data),
					CASTI2(data_container->device_bids_data),
					CASTF3(data_container->device_aux_data),
					CASTF3(data_container->device_inr_data),
					CASTF4(data_container->device_rot_data),
					CASTF3(data_container->device_vel_data),
					CASTF3(data_container->device_omg_data),
					CASTF3(data_container->device_pos_data),
					CASTF3(vel_update),
					CASTF3(omg_update),
					CASTU1(update_offset));
		}
		LCP_Iteration_Bilaterals<<< BLOCKS(number_of_bilaterals), THREADS >>>(
				CASTF4(device_bilateral_data),
				CASTF3(data_container->device_aux_data),
				CASTF3(data_container->device_inr_data),
				CASTF4(data_container->device_rot_data),
				CASTF3(data_container->device_vel_data),
				CASTF3(data_container->device_omg_data),
				CASTF3(data_container->device_pos_data),
				CASTF3(vel_update),
				CASTF3(omg_update),
				CASTU1(update_offset),
				CASTF1(device_dgm_data));

		LCP_Reduce_Speeds<<< BLOCKS(number_of_updates), THREADS >>>(
				CASTF3(data_container->device_aux_data),
				CASTF3(data_container->device_vel_data),
				CASTF3(data_container->device_omg_data),
				CASTF3(vel_update),
				CASTF3(omg_update),
				CASTU1(body_number2),
				CASTU1(offset_counter));
		if (use_DEM == true) {break;}
		if(Avg_DeltaGamma()<tolerance) {break;}
	}
}
LCP_Integrate_Timestep<<< BLOCKS(number_of_bodies),THREADS>>>(
		CASTF3(data_container->device_aux_data),
		CASTF3(data_container->device_acc_data),
		CASTF4(data_container->device_rot_data),
		CASTF3(data_container->device_vel_data),
		CASTF3(data_container->device_omg_data),
		CASTF3(data_container->device_pos_data),
		CASTF3(data_container->device_lim_data));
}
float ChLcpIterativeSolverGPU::Max_DeltaGamma() {
	return Thrust_Max(device_dgm_data);
}
float ChLcpIterativeSolverGPU::Min_DeltaGamma() {
	return Thrust_Min(device_dgm_data);
}
float ChLcpIterativeSolverGPU::Avg_DeltaGamma() {
	return (Thrust_Total(device_dgm_data)) / float(number_of_constraints);
}
__global__ void Compute_KE(float3* vel, float3* aux, float* ke) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= number_of_bodies_const) {
		return;
	}
	float3 velocity = vel[i];
	ke[i] = .5 / aux[i].z * dot(velocity, velocity);

}
float ChLcpIterativeSolverGPU::Total_KineticEnergy() {
	device_ken_data.resize(number_of_bodies);
	Compute_KE<<< BLOCKS(number_of_bodies),THREADS>>>(
			CASTF3(data_container->device_vel_data),
			CASTF3(data_container->device_aux_data),
			CASTF1(device_ken_data));
	return (Thrust_Total(device_ken_data));
}
