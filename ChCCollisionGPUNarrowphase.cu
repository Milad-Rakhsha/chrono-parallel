#include "hip/hip_runtime.h"
#include "ChCCollisionGPU.h"
#include "ChCCollisionGPU.cuh"

__device__ inline void AddContact(contactGPU* CData,uint offset,  int A,int B, float3 onA, float3 onB,float3 n,float dist){
	CData[offset].I=F3(dist,A,B);
	CData[offset].N=n;
	CData[offset].Pa=onA;
	CData[offset].Pb=onB;
	CData[offset].G=F3(0,0,0);
}

__device__ bool pointInTriangle(const float3 &p1,const float3 &p2,const float3 &p3, const float3 &normal, const float3 &S ){
	float3 edge1=( p2 - p1 );
	float3 edge2=( p3 - p2 );
	float3 edge3=( p1 - p3 );

	float3 p1_to_p=( S - p1 );
	float3 p2_to_p=( S - p2 );
	float3 p3_to_p=( S - p3 );

	float3 edge1_normal=cross(edge1,normal);
	float3 edge2_normal=cross(edge2,normal);
	float3 edge3_normal=cross(edge3,normal);

	float r1 = dot(edge1_normal, p1_to_p );
	float r2 = dot(edge2_normal, p2_to_p );
	float r3 = dot(edge3_normal, p3_to_p );
	if ( ( r1 > 0 && r2 > 0 && r3 > 0 ) ||( r1 <= 0 && r2 <= 0 && r3 <= 0 ) ){return true;}
	return false;
}

__device__ float SegmentSqrDistance(const float3& from, const float3& to,const float3 &p, float3 &nearest) {
	float3  diff = p - from; 
	float3	v = to - from;
	float t = dot(v,diff);
	if (t > 0) {
		float dotVV = dot(v,v);
		if (t < dotVV) {
			t /= dotVV;
			diff -= t*v;
		}else {
			t = 1;
			diff -= v;
		}
	}else{t = 0;}
	nearest = from + t*v;
	return dot(diff,diff);	
}

__global__ void Sphere_Sphere(object * object_data,int3 * Pair , uint* Contact_Number,contactGPU* CData,uint totalPossibleConts){
	uint Index = blockIdx.x* blockDim.x + threadIdx.x;
	if(Index>=totalPossibleConts){return;}
	int3 pair=Pair[Index];
	if(pair.z==0){
		object A=object_data[pair.x];
		object B=object_data[pair.y];
		float3 N=F3(B.A)-F3(A.A);
		float centerDist =dot(N,N);
		float rAB =B.A.w + A.A.w;
		if (centerDist <= (rAB)*(rAB)){
			float dist=sqrtf(centerDist);
			N=N/dist;
			AddContact(CData,Index,A.B.x,B.B.x,F3(A.A)+A.A.w*N,F3(B.A)-B.A.w*N,N,-dist);
			Contact_Number[Index]=Index;
		}
	}
}

__global__ void Sphere_Triangle(object * object_data,int3 * Pair ,uint* Contact_Number,contactGPU* CData,uint totalPossibleConts){
	uint Index = blockIdx.x* blockDim.x + threadIdx.x;
	if(Index>=totalPossibleConts){return;}
	if(Contact_Number[Index]!=0xFFFFFFFF){return;}
	int3 pair=Pair[Index];
	if(pair.z==1||pair.z==2){
		bool hasContact = false;
		if(pair.z==2){
			uint temp=pair.x;
			pair.x=pair.y;
			pair.y=temp;
		}
		object sphere=object_data[pair.x];
		object triangle=object_data[pair.y];
		float3 
			A=F3(triangle.A),
			B=F3(triangle.B),
			C=F3(triangle.C),
			S=F3(sphere.A),
			contactPoint;

		float3 N=cross((B-A),(C-A));
		N/=sqrtf(dot(N,N));
		float distanceFromPlane = dot(S-A,N);

		if (distanceFromPlane < 0.0f){
			distanceFromPlane *= -1.0f;	
			N *= -1.0f;
		}
		if(distanceFromPlane > sphere.A.w){return;}
		else if(distanceFromPlane < sphere.A.w+collision_envelope_const) {
			if (pointInTriangle(A,B,C,N,S)) {
				hasContact = true;
				contactPoint = S - N*distanceFromPlane;
			}else{
				float RadSqr = (sphere.A.w+collision_envelope_const)*(sphere.A.w+collision_envelope_const);
				if (SegmentSqrDistance(A,B,S, contactPoint) < RadSqr) {hasContact = true;}
				if (SegmentSqrDistance(B,C,S, contactPoint) < RadSqr) {hasContact = true;}
				if (SegmentSqrDistance(C,A,S, contactPoint) < RadSqr) {hasContact = true;}
			}
		}
		if (hasContact) {
			N = S - contactPoint;
			float distance = dot(N,N);
			if (distance < (sphere.A.w - 0.)*(sphere.A.w - 0.)) {
				distance = sqrtf(distance);
				N = N/distance;
				CData[Index].I=F3(-(sphere.A.w-distance),triangle.A.w,sphere.B.x);
				CData[Index].N=N;
				CData[Index].Pa=contactPoint;
				CData[Index].Pb=S-N*sphere.A.w;
				//CData[Index].G=F4(0,0,0,0);
				Contact_Number[Index]=Index;
				return;
			}
		}
	}
}

__device__ __host__ uint getID(const object &A){
	if(A.B.w==0)	{return A.B.x;}
	else if(A.B.w==1){return A.A.w;}
	else if(A.B.w==2){return A.A.w;}
	else if(A.B.w==3){return A.A.w;}
	else return 100;
}

__device__ __host__ inline float3 GetSupportPoint_Sphere	(const object& p,const  float3 &n){		
	return (p.A.w) * n;
}
__device__ __host__ inline float3 GetSupportPoint_Triangle	(const object& p,const float3 &n){
	float3 Pa=make_float3(p.A);
	float3 Pb=make_float3(p.B);
	float3 Pc=make_float3(p.C);
	float dist = dot(Pa,n);
	float3 point=Pa;
	if(dot(Pb,n)>dist){dist=dot(Pb,n); point=Pb;}
	if(dot(Pc,n)>dist){dist=dot(Pc,n); point=Pc;}
	return point;
}
__device__ __host__ inline float3 GetSupportPoint_Box		(const object& p,const float3 &n){
	float3 result;
	result= make_float3(p.B);
	if (n.x < 0.) result.x = -result.x;
	if (n.y < 0.) result.y = -result.y;
	if (n.z < 0.) result.z = -result.z;
	//result.x=dot(n,F3(1,0,0))*(p.B.x);
	//result.y=dot(n,F3(0,1,0))*(p.B.y);
	//result.z=dot(n,F3(0,0,1))*(p.B.z);
	return result;
}
__device__ __host__ inline float3 GetSupportPoint_Ellipsoid	(const object& p,const float3 &n){
	return normalize(n* F3(p.B))* F3(p.B);
}
__device__ __host__ inline float3 GetSupportPoint_Cylinder	(const object& p,const float3 &n){
	return make_float3(0,0,0);
}
__device__ __host__ inline float3 GetSupportPoint_Plane		(const object& p,const float3 &n){
	float3 result = make_float3(p.B);
	if (n.x < 0) result.x = -result.x;
	if (n.y < 0) result.y = -result.y;
	return result;
}
__device__ __host__ inline float3 GetSupportPoint_Cone		(const object& p,const float3 &n){
	return make_float3(0,0,0);
}
__device__ __host__ inline float3 GetCenter_Sphere			(const object& p){
	return make_float3(0,0,0);
}
__device__ __host__ inline float3 GetCenter_Triangle		(const object& p){
	return make_float3((p.A.x+p.B.x+p.C.x)/3.0f,(p.A.y+p.B.y+p.C.y)/3.0f,(p.A.z+p.B.z+p.C.z)/3.0f);
}
__device__ __host__ inline float3 GetCenter_Box				(const object& p){return Zero_Vector;}
__device__ __host__ inline float3 GetCenter_Ellipsoid		(const object& p){return Zero_Vector;}
__device__ __host__ inline float3 GetCenter_Cylinder		(const object& p){return Zero_Vector;}
__device__ __host__ inline float3 GetCenter_Plane			(const object& p){return Zero_Vector;}
__device__ __host__ inline float3 GetCenter_Cone			(const object& p){return Zero_Vector;}
__device__ __host__ bool IsZero3(const float3 &v){
	return (	v.x < Vector_ZERO_EPSILON && v.x > -Vector_ZERO_EPSILON &&
				v.y < Vector_ZERO_EPSILON && v.y > -Vector_ZERO_EPSILON &&
				v.z < Vector_ZERO_EPSILON && v.z > -Vector_ZERO_EPSILON );
}
__device__ __host__ float3 GetCenter(const object& p){
	if(p.B.w==1){return GetCenter_Triangle(p);}	//triangle
	else{return make_float3(0,0,0)+ F3(p.A);}	//All other shapes assumed to be locally centered
}
__device__ __host__ float3 TransformSupportVert(const object& p,const float3& b){
	float3 localSupport;
	float3 n=normalize(b);
	if(p.B.w==1){//triangle
		return GetSupportPoint_Triangle(p,n);	
	}
	else if(p.B.w==0){//sphere
		localSupport = GetSupportPoint_Sphere(p,quatRotate(n,(~p.C)));	
	}
	else if(p.B.w==2){//box
		localSupport = GetSupportPoint_Box(p,quatRotate(n,(~p.C)));
	}
	else if(p.B.w==3){//ellipsoid
		localSupport = GetSupportPoint_Ellipsoid(p,quatRotate(n,(~p.C)));
	}
	else if(p.B.w==4){//cylinder
		localSupport = GetSupportPoint_Cylinder(p,quatRotate(n,(~p.C)));
	}
	else if(p.B.w==5){//plane
		localSupport = GetSupportPoint_Plane(p,quatRotate(n,(~p.C)));	
	}
	else if(p.B.w==6){//cone
		localSupport = GetSupportPoint_Cone(p,quatRotate(n,(~p.C)));
	}
	return quatRotate(localSupport,p.C) + F3(p.A); //globalSupport
}



//Code for Convex-Convex Collision detection, adopted from xeno-collide
__device__ __host__ bool CollideAndFindPoint(const object& p1, const object& p2, float3& returnNormal, float3& point1, float3& point2)
{
	float3 v01, v02, v0, n,v11, v12, v1, v21, v22, v2;
	// v0 = center of Minkowski sum
	v01 = GetCenter(p1);
	v02 = GetCenter(p2);
	v0 = v02 - v01;

	// Avoid case where centers overlap -- any direction is fine in this case
	if (IsZero3(v0)) v0 = make_float3(1, 0, 0);

	// v1 = support in direction of origin
	n = normalize(-v0);
	v11 = TransformSupportVert(p1, -n);
	v12 = TransformSupportVert(p2, n);
	v1 = v12 - v11;
	if (dot(v1 , n) <= 0){return false;}

	// v2 - support perpendicular to v1,v0
	n = cross(v1 , v0);
	if (IsZero3(n)){
		n = v1 - v0;
		n=normalize(n);
		returnNormal = n;
		point1=v11;
		point2=v12;
		return true;
	}
	v21 = TransformSupportVert(p1, -n);
	v22 = TransformSupportVert(p2, n);
	v2 = v22 - v21;
	if (dot(v2 , n) <= 0){return false;}

	// Determine whether origin is on + or - side of plane (v1,v0,v2)
	n = cross((v1 - v0) , (v2 - v0));
	// If the origin is on the - side of the plane, reverse the direction of the plane
	if (dot(n , v0) > 0){
		Swap(v1, v2);
		Swap(v11, v21);
		Swap(v12, v22);
		n = -n;
	}
	// Phase One: Identify a portal
	float3 v31, v32, v3;
	while (1){
		// Obtain the support point in a direction perpendicular to the existing plane
		// Note: This point is guaranteed to lie off the plane
		v31 = TransformSupportVert(p1, -n);
		v32 = TransformSupportVert(p2, n); 
		v3  = v32 - v31;
		if (dot(v3 , n) <= 0){return false;}
		// If origin is outside (v1,v0,v3), then eliminate v2 and loop
		else if (dot(cross(v1 , v3) , v0) < 0){
			v2 = v3;
			v21 = v31;
			v22 = v32;
			n = cross((v1 - v0) , (v3 - v0));
			continue;
		}
		// If origin is outside (v3,v0,v2), then eliminate v1 and loop
		else if (dot(cross(v3 , v2) , v0) < 0){
			v1 = v3;
			v11 = v31;
			v12 = v32;
			n = cross((v3 - v0) , (v2 - v0));
			continue;
		}
		break;
	}
	bool hit = false;

	// Phase Two: Refine the portal
	// We are now inside of a wedge...
	int phase2=0;
	while(1){
		phase2++;
		// Compute normal of the wedge face
		n = cross((v2 - v1) , (v3 - v1));
		n=n/sqrtf(dot(n,n));
		// Compute distance from origin to wedge face
		// If the origin is inside the wedge, we have a hit
		if (dot(n , v1) >= 0. && !hit){
			// Compute the barycentric coordinates of the origin
			float b0 = dot(cross(v1 , v2) , v3);
			float b1 = dot(cross(v3 , v2) , v0);
			float b2 = dot(cross(v0 , v1) , v3);
			float b3 = dot(cross(v2 , v1) , v0);

			float sum = b0 + b1 + b2 + b3;

			if (sum <= 0.){
				b0 = 0;
				b1 = dot(cross(v2 , v3) , n);
				b2 = dot(cross(v3 , v1) , n);
				b3 = dot(cross(v1 , v2) , n);

				sum = b1 + b2 + b3;
			}
			float inv = 1.0f / sum;
			point1 = (b0 * v01 + b1 * v11 + b2 * v21 + b3 * v31) * inv;
			point2 = (b0 * v02 + b1 * v12 + b2 * v22 + b3 * v32) * inv;
			hit = true;// HIT!!!
		}
		// Find the support point in the direction of the wedge face
		float3 v41 = TransformSupportVert(p1, -n);
		float3 v42 = TransformSupportVert(p2, n); 
		float3 v4 = v42 - v41;

		float delta = dot((v4 - v3) , n);
		float separation = -dot(v4 , n);

		// If the boundary is thin enough or the origin is outside the support plane for the newly discovered vertex, then we can terminate
		if ( delta <= kCollideEpsilon || separation >= 0. || phase2 > 400 ){
			returnNormal = n;
			return hit;
		}
		if (dot(cross(v4 , v1) , v0) < 0.){			// Compute the tetrahedron dividing face (v4,v0,v1)
			if (dot(cross(v4 , v2) , v0) < 0){		// Compute the tetrahedron dividing face (v4,v0,v2)
				v1 = v4;	v11 = v41;	v12 = v42;	// Inside d1 & inside d2 ==> eliminate v1
			}else{
				v3 = v4;	v31 = v41;	v32 = v42;	// Inside d1 & outside d2 ==> eliminate v3
			}
		}else{
			if (dot(cross(v4 , v3) , v0) < 0.){		// Compute the tetrahedron dividing face (v4,v0,v3)
				v2 = v4;	v21 = v41;	v22 = v42;	// Outside d1 & inside d3 ==> eliminate v2
			}else{
				v1 = v4;	v11 = v41;	v12 = v42;	// Outside d1 & outside d3 ==> eliminate v1
			}
		}
	}
}
__global__ void MPR_GPU_Store(
							  object * object_data,
							  int3 * Pair ,
							  uint* Contact_Number,
							  contactGPU* CData,
							  uint totalPossibleConts)
{
	uint Index = blockIdx.x* blockDim.x + threadIdx.x;
	if(Index>=totalPossibleConts){return;}
	if(Contact_Number[Index]!=0xFFFFFFFF){return;}
	int3 pair=Pair[Index];
	if(pair.z<3){return;}
	object A=object_data[pair.x];
	object B=object_data[pair.y];
	float3 N,p1,p2;
	if(!CollideAndFindPoint(A,B,N,p1, p2)){return;};
	p1=(TransformSupportVert(A,-N)-p1)*N*N+p1;
	p2=(TransformSupportVert(B,N)-p2)*N*N+p2;
	float depth=sqrtf(dot((p2-p1),(p2-p1)));
	AddContact(CData,Index,  getID(A),getID(B), p1, p2,-N,-depth);
	Contact_Number[Index]=Index;
}

void ChCCollisionGPU::Narrowphase(){       						//NarrowPhase Contact CD 
	generic_counter.resize(number_of_contacts);
	Thrust_Fill(generic_counter,0xFFFFFFFF);
	contact_data_gpu->resize(number_of_contacts);
	Sphere_Sphere<<<BLOCKS(number_of_contacts),THREADS>>>(		//Compute Sphere-Sphere Contacts
		OBJCAST(object_data),
		CASTI3(contact_pair),									//Indices of bodies that make up AABB contact
		CASTU1(generic_counter),								//Contact Index, store the thread index
		CONTCAST((*contact_data_gpu)),							//Contact Data GPU
		number_of_contacts);									//Number of potential contacts
	
	Sphere_Triangle<<<BLOCKS(number_of_contacts),THREADS>>>(	//Compute Sphere-Sphere Contacts
		OBJCAST(object_data),                         			//Object Data
		CASTI3(contact_pair),                     				//Indices of bodies that make up AABB contact
		CASTU1(generic_counter),								//Contact Index, store the thread index
		CONTCAST((*contact_data_gpu)),                          //Contact Data GPU
		number_of_contacts);									//Number of potential contacts

	MPR_GPU_Store<<<BLOCKS(number_of_contacts),THREADS>>>(		//Compute convex-covnex Contacts
		OBJCAST(object_data),                         			//Object Data
		CASTI3(contact_pair),                     				//Indices of bodies that make up AABB contact
		CASTU1(generic_counter),								//Contact Index, store the thread index
		CONTCAST((*contact_data_gpu)),                          //Contact Data GPU
		number_of_contacts);									//Number of potential contacts

	Thrust_Sort_By_Key(generic_counter,(*contact_data_gpu));
	number_of_contacts=number_of_contacts-Thrust_Count(generic_counter,0xFFFFFFFF);
	contact_data_gpu->resize(number_of_contacts);
}
