#include "hip/hip_runtime.h"
#include "ChCCollisionGPU.h"
#include "ChCCollisionGPU.cuh"
#include <thrust/remove.h>

//__global__ void Sphere_Sphere(object * object_data, int3 * Pair,
//		uint* Contact_Number, contactGPU* CData, uint totalPossibleConts) {
//	uint index = blockIdx.x * blockDim.x + threadIdx.x;
//	if (index >= totalPossibleConts) {
//		return;
//	}
//	int3 pair = Pair[index];
//	if (pair.z == 0) {
//		object A = object_data[pair.x];
//		object B = object_data[pair.y];
//		float3 N = F3(B.A) - F3(A.A);
//		float centerDist = dot(N, N);
//		float rAB = B.B.x + A.B.x;
//		if (centerDist <= (rAB) * (rAB)) {
//			float dist = sqrtf(centerDist);
//			N = N / dist;
//			AddContact(CData, index, A.A.w, B.A.w, F3(A.A) + A.B.x * N, F3(B.A)
//					- B.B.x * N, N, -dist);
//			Contact_Number[index] = index;
//		}
//	}
//}


__device__ __host__ inline float3 GetSupportPoint_Sphere(const float3 &B, const float3 &n) {
	return (B.x) * n;
}
__device__ __host__ inline float3 GetSupportPoint_Triangle(const float3 &A, const float3 &B, const float3 &C, const float3 &n) {
	float dist = dot(A, n);
	float3 point = A;
	if (dot(B, n) > dist) {
		dist = dot(B, n);
		point = B;
	}
	if (dot(C, n) > dist) {
		dist = dot(C, n);
		point = C;
	}
	return point;
}
__device__ __host__ inline float3 GetSupportPoint_Box(const float3 &B, const float3 &n) {
	float3 result = F3(0, 0, 0);
	result.x = n.x >= 0 ? B.x : -B.x;
	result.y = n.y >= 0 ? B.y : -B.y;
	result.z = n.z >= 0 ? B.z : -B.z;
	return result;
}
__device__ __host__ inline float3 GetSupportPoint_Ellipsoid(const float3 &B, const float3 &n) {
	return B * B * n / length(n * B);
}
__device__ __host__ float sign(float x) {
	if (x < 0) {
		return -1;
	} else {
		return 1;
	}
}

__device__ __host__ inline float3 GetSupportPoint_Cylinder(const float3 &B, const float3 &n) {
	//return make_float3(0,0,0);
	float3 u = F3(0, 1, 0);
	float3 w = n - (dot(u, n)) * u;
	float3 result;
	if (length(w) != 0) {
		result = sign(dot(u, n)) * B.y * u + B.x * normalize(w);
	} else {
		result = sign(dot(u, n)) * B.y * u;
	}
	return result;
}
__device__ __host__ inline float3 GetSupportPoint_Plane(const float3 &B, const float3 &n) {
	float3 result = B;
	if (n.x < 0) result.x = -result.x;
	if (n.y < 0) result.y = -result.y;
	return result;
}
__device__ __host__ inline float3 GetSupportPoint_Cone(const float3 &B, const float3 &n) {
	return make_float3(0, 0, 0);
}
__device__ __host__ inline float3 GetCenter_Sphere() {
	return Zero_Vector;
}
__device__ __host__ inline float3 GetCenter_Triangle(const float3 &A, const float3 &B, const float3 &C) {
	return make_float3((A.x + B.x + C.x) / 3.0f, (A.y + B.y + C.y) / 3.0f, (A.z + B.z + C.z) / 3.0f);
}
__device__ __host__ inline float3 GetCenter_Box() {
	return Zero_Vector;
}
__device__ __host__ inline float3 GetCenter_Ellipsoid() {
	return Zero_Vector;
}
__device__ __host__ inline float3 GetCenter_Cylinder() {
	return Zero_Vector;
}
__device__ __host__ inline float3 GetCenter_Plane() {
	return Zero_Vector;
}
__device__ __host__ inline float3 GetCenter_Cone() {
	return Zero_Vector;
}
__device__ __host__ bool IsZero3(const float3 &v) {
	return (v.x < Vector_ZERO_EPSILON && v.x > -Vector_ZERO_EPSILON && v.y < Vector_ZERO_EPSILON && v.y > -Vector_ZERO_EPSILON && v.z
			< Vector_ZERO_EPSILON && v.z > -Vector_ZERO_EPSILON);
}
__device__ __host__ bool IsZero(const float &val) {
	return fabs(val) < 1E-10;
}
__device__ __host__ bool isEqual(const float& _a, const float& _b) {
	float ab;

	ab = fabs(_a - _b);
	if (fabs(ab) < 1E-10) return 1;

	float a, b;
	a = fabs(_a);
	b = fabs(_b);
	if (b > a) {
		return ab < 1E-10 * b;
	} else {
		return ab < 1E-10 * a;
	}
}

__device__ __host__ float3 GetCenter(const int &type, const float3 &A, const float3 &B, const float3 &C) {
	if (type == 5) {
		return GetCenter_Triangle(A, B, C);
	} //triangle
	else {
		return make_float3(0, 0, 0) + A;
	} //All other shapes assumed to be locally centered
}
__device__ __host__ float3 TransformSupportVert(const int &type, const float3 &A, const float3 &B, const float3 &C, const float4 &R, const float3& b) {
	float3 localSupport;
	float3 n = normalize(b);
	if (type == 5) {//triangle
		return GetSupportPoint_Triangle(A, B, C, n);
	} else if (type == 0) {//sphere
		localSupport = GetSupportPoint_Sphere(B, quatRotate(n, inv(R)));
	} else if (type == 1) {//ellipsoid
		localSupport = GetSupportPoint_Ellipsoid(B, quatRotate(n, inv(R)));
	} else if (type == 2) {//box
		localSupport = GetSupportPoint_Box(B, quatRotate(n, inv(R)));
	} else if (type == 3) {//cylinder
		localSupport = GetSupportPoint_Cylinder(B, quatRotate(n, inv(R)));
	} else if (type == 6) {//plane
		localSupport = GetSupportPoint_Plane(B, quatRotate(n, inv(R)));
	} else if (type == 7) {//cone
		localSupport = GetSupportPoint_Cone(B, quatRotate(n, inv(R)));
	}
	return quatRotate(localSupport, R) + A; //globalSupport
}

__device__ __host__ float dist_line(float3 & P, float3 &x0, float3 &b, float3& witness) {
	float dist, t;
	float3 d, a;

	d = b - x0; // direction of segment
	a = x0 - P; // precompute vector from P to x0
	t = -(1.f) * dot(a, d);
	t /= dot(d, d);

	if (t < 0.0f || IsZero(t)) {
		dist = dot(x0 - P, x0 - P);
		witness = x0;
	} else if (t > 1.0f || isEqual(t, 1.0f)) {
		dist = dot(b - P, b - P);
		witness = b;
	} else {
		witness = d;
		witness *= t;
		witness += x0;
		dist = dot(witness - P, witness - P);
	}

	return dist;
}
__device__ __host__ float find_dist(float3 & P, float3 &x0, float3 &B, float3 &C, float3& witness) {
	float3 d1, d2, a;
	float u, v, w, p, q, r;
	float s, t, dist, dist2;
	float3 witness2;

	d1 = B - x0;
	d2 = C - x0;
	a = x0 - P;

	u = dot(a, a);
	v = dot(d1, d1);
	w = dot(d2, d2);
	p = dot(a, d1);
	q = dot(a, d2);
	r = dot(d1, d2);

	s = (q * r - w * p) / (w * v - r * r);
	t = (-s * r - q) / w;

	if ((IsZero(s) || s > 0.0f) && (isEqual(s, 1.0f) || s < 1.0f) && (IsZero(t) || t > 0.0f) && (isEqual(t, 1.0f) || t < 1.0f) && (isEqual(
																																			t + s,
																																			1.0f)
			|| t + s < 1.0f)) {
		d1 *= s;
		d2 *= t;
		witness = x0;
		witness += d1;
		witness += d2;
		dist = dot(witness - P, witness - P);
	} else {
		dist = dist_line(P, x0, B, witness);

		dist2 = dist_line(P, x0, C, witness2);
		if (dist2 < dist) {
			dist = dist2;
			(witness = witness2);
		}

		dist2 = dist_line(P, B, C, witness2);
		if (dist2 < dist) {
			dist = dist2;
			(witness = witness2);
		}
	}
	return dist;
}

//Code for Convex-Convex Collision detection, adopted from xeno-collide
__device__ __host__ bool CollideAndFindPoint(
												int typeA,
												float3 A_X,
												float3 A_Y,
												float3 A_Z,
												float4 A_R,
												int typeB,
												float3 B_X,
												float3 B_Y,
												float3 B_Z,
												float4 B_R,
												float3& returnNormal,
												float3& point1,
												float3& point2,
												float& depth) {
	float3 v01, v02, v0, n, v11, v12, v1, v21, v22, v2;
	// v0 = center of Minkowski sum
	v01 = GetCenter(typeA, A_X, A_Y, A_Z);
	v02 = GetCenter(typeB, B_X, B_Y, B_Z);
	v0 = v02 - v01;

	// Avoid case where centers overlap -- any direction is fine in this case
	if (IsZero3(v0)) v0 = make_float3(1, 0, 0);

	// v1 = support in direction of origin
	n = normalize(-v0);
	v11 = TransformSupportVert(typeA, A_X, A_Y, A_Z, A_R, -n);
	v12 = TransformSupportVert(typeB, B_X, B_Y, B_Z, B_R, n);
	v1 = v12 - v11;
	if (dot(v1, n) <= 0) {
		return false;
	}

	// v2 - support perpendicular to v1,v0
	n = cross(v1, v0);
	if (IsZero3(n)) {
		n = v1 - v0;
		n = normalize(n);
		returnNormal = n;
		point1 = v11;
		point2 = v12;
		return true;
	}
	v21 = TransformSupportVert(typeA, A_X, A_Y, A_Z, A_R, -n);
	v22 = TransformSupportVert(typeB, B_X, B_Y, B_Z, B_R, n);
	v2 = v22 - v21;
	if (dot(v2, n) <= 0) {
		return false;
	}

	// Determine whether origin is on + or - side of plane (v1,v0,v2)
	n = normalize(cross((v1 - v0), (v2 - v0)));
	// If the origin is on the - side of the plane, reverse the direction of the plane
	if (dot(n, v0) > 0) {
		Swap(v1, v2);
		Swap(v11, v21);
		Swap(v12, v22);
		n = -n;
	}
	// Phase One: Identify a portal
	float3 v31, v32, v3;
	while (1) {
		// Obtain the support point in a direction perpendicular to the existing plane
		// Note: This point is guaranteed to lie off the plane
		v31 = TransformSupportVert(typeA, A_X, A_Y, A_Z, A_R, -n);
		v32 = TransformSupportVert(typeB, B_X, B_Y, B_Z, B_R, n);
		v3 = v32 - v31;
		if (dot(v3, n) <= 0) {
			return false;
		}
		// If origin is outside (v1,v0,v3), then eliminate v2 and loop
		else if (dot(cross(v1, v3), v0) < 0) {
			v2 = v3;
			v21 = v31;
			v22 = v32;
			n = cross((v1 - v0), (v3 - v0));
			continue;
		}
		// If origin is outside (v3,v0,v2), then eliminate v1 and loop
		else if (dot(cross(v3, v2), v0) < 0) {
			v1 = v3;
			v11 = v31;
			v12 = v32;
			n = cross((v3 - v0), (v2 - v0));
			continue;
		}
		break;
	}
	bool hit = false;

	// Phase Two: Refine the portal
	// We are now inside of a wedge...
	int phase2 = 0;
	while (1) {
		phase2++;
		// Compute normal of the wedge face
		n = cross((v2 - v1), (v3 - v1));
		n = normalize(n);

		// Compute distance from origin to wedge face
		// If the origin is inside the wedge, we have a hit
		if (dot(n, v1) >= 0. && !hit) {
			// Compute the barycentric coordinates of the origin
			float b0 = dot(cross(v1, v2), v3);
			float b1 = dot(cross(v3, v2), v0);
			float b2 = dot(cross(v0, v1), v3);
			float b3 = dot(cross(v2, v1), v0);

			float sum = b0 + b1 + b2 + b3;

			if (sum <= 0.) {
				b0 = 0;
				b1 = dot(cross(v2, v3), n);
				b2 = dot(cross(v3, v1), n);
				b3 = dot(cross(v1, v2), n);

				sum = b1 + b2 + b3;
			}
			float inv = 1.0f / sum;
			point1 = (b0 * v01 + b1 * v11 + b2 * v21 + b3 * v31) * inv;
			point2 = (b0 * v02 + b1 * v12 + b2 * v22 + b3 * v32) * inv;
			//point1+=point2;
			//point1*=.5;

			hit = true;// HIT!!!
		}
		// Find the support point in the direction of the wedge face
		float3 v41 = TransformSupportVert(typeA, A_X, A_Y, A_Z, A_R, -n);
		float3 v42 = TransformSupportVert(typeB, B_X, B_Y, B_Z, B_R, n);
		float3 v4 = v42 - v41;

		float delta = dot((v4 - v3), n);
		float separation = -dot(v4, n);

		// If the boundary is thin enough or the origin is outside the support plane for the newly discovered vertex, then we can terminate
		if (delta <= kCollideEpsilon || separation >= 0. || phase2 > 100) {
			float3 O = F3(0, 0, 0);
			//depth=find_dist(O,v1,v2,v3,n);
			returnNormal = normalize(n);
			return hit;
		}
		if (dot(cross(v4, v1), v0) < 0.) { // Compute the tetrahedron dividing face (v4,v0,v1)
			if (dot(cross(v4, v2), v0) < 0) { // Compute the tetrahedron dividing face (v4,v0,v2)
				v1 = v4;
				v11 = v41;
				v12 = v42; // Inside d1 & inside d2 ==> eliminate v1
			} else {
				v3 = v4;
				v31 = v41;
				v32 = v42; // Inside d1 & outside d2 ==> eliminate v3
			}
		} else {
			if (dot(cross(v4, v3), v0) < 0.) { // Compute the tetrahedron dividing face (v4,v0,v3)
				v2 = v4;
				v21 = v41;
				v22 = v42; // Outside d1 & inside d3 ==> eliminate v2
			} else {
				v1 = v4;
				v11 = v41;
				v12 = v42; // Outside d1 & outside d3 ==> eliminate v1
			}
		}
	}
}
__global__ void MPR_GPU_Store(
								float3* pos,
								float4* rot,
								float3* obA,
								float3* obB,
								float3* obC,
								float4* obR,
								int3* typ,
								long long * Pair,
								uint* Contact_Number,
								float3* norm,
								float3* ptA,
								float3* ptB,
								float* contactDepth,
								int2* ids,

								uint totalPossibleConts) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= totalPossibleConts) {
		return;
	}
	if (Contact_Number[index] != 0xFFFFFFFF) {
		return;
	}
	long long p = Pair[index];
	int2 pair = I2(int(p >> 32), int(p & 0xffffffff));
	//if(pair.z<4){return;}

	int3 A_T = typ[pair.x], B_T = typ[pair.y];

	float3 posA = pos[A_T.z], posB = pos[B_T.z];
	float4 rotA = rot[A_T.z], rotB = rot[B_T.z];

	float3 A_X = obA[A_T.y], B_X = obA[B_T.y];
	float3 A_Y = obB[A_T.y], B_Y = obB[B_T.y];
	float3 A_Z = obC[A_T.y], B_Z = obC[B_T.y];

	float4 A_R = normalize(rotA + obR[A_T.y]);
	float4 B_R = normalize(rotB + obR[B_T.y]);

	if (A_T.x == 0 || A_T.x == 1 || A_T.x == 2 || A_T.x == 3) {
		A_X = A_X + posA;
	} else if (A_T.x == 5) {
		A_X = quatRotate(A_X + posA, A_R);
		A_Y = quatRotate(A_Y + posA, A_R);
		A_Z = quatRotate(A_Z + posA, A_R);
	}

	if (B_T.x == 0 || B_T.x == 1 || B_T.x == 2 || B_T.x == 3) {
		B_X = B_X + posB;
	} else if (B_T.x == 5) {
		B_X = quatRotate(B_X + posB, B_R);
		B_Y = quatRotate(B_Y + posB, B_R);
		B_Z = quatRotate(B_Z + posB, B_R);
	}

	float3 N, p1, p2;
	float depth = 0;
	if (!CollideAndFindPoint(A_T.x, A_X, A_Y, A_Z, A_R, B_T.x, B_X, B_Y, B_Z, B_R, N, p1, p2, depth)) {
		return;
	};

	p1 = (TransformSupportVert(A_T.x, A_X, A_Y, A_Z, A_R, -N) - p1) * N * N + p1;
	p2 = (TransformSupportVert(B_T.x, B_X, B_Y, B_Z, B_R, N) - p2) * N * N + p2;
	depth = sqrtf(dot((p2 - p1), (p2 - p1)));
	norm[index] = -N;
	ptA[index] = p1;
	ptB[index] = p2;
	contactDepth[index] = -depth;
	ids[index] = I2(A_T.z, B_T.z);
	Contact_Number[index] = 0;
}
__global__ void CopyGamma(int* to, float3* oldG, float3* newG, int contacts) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= contacts) {
		return;
	}
	newG[to[i]] = oldG[i];
}
void ChCCollisionGPU::Narrowphase_HOST(ChGPUDataManager * data_container) {
	data_container->number_of_contacts = 0;
	data_container->host_norm_data.clear();
	data_container->host_cpta_data.clear();
	data_container->host_cptb_data.clear();
	data_container->host_dpth_data.clear();
	data_container->host_bids_data.clear();
	for (int index = 0; index < data_container->number_of_contacts_possible; index++) {

		long long p = data_container->host_pair_data[index];
		int2 pair = I2(int(p >> 32), int(p & 0xffffffff));

		int3 A_T = data_container->host_typ_data[pair.x], B_T = data_container->host_typ_data[pair.y];

		float3 posA = data_container->host_pos_data[A_T.z], posB = data_container->host_pos_data[B_T.z];
		float4 rotA = data_container->host_rot_data[A_T.z], rotB = data_container->host_rot_data[B_T.z];

		float3 A_X = data_container->host_ObA_data[A_T.y], B_X = data_container->host_ObA_data[B_T.y];
		float3 A_Y = data_container->host_ObB_data[A_T.y], B_Y = data_container->host_ObB_data[B_T.y];
		float3 A_Z = data_container->host_ObC_data[A_T.y], B_Z = data_container->host_ObC_data[B_T.y];

		float4 A_R = rotA;// + obR[A_T.y];
		float4 B_R = rotB;// + obR[B_T.y];

		if (A_T.x == 0 || A_T.x == 1 || A_T.x == 2 || A_T.x == 3) {
			A_X = A_X + posA;
		} else if (A_T.x == 5) {
			A_X = quatRotate(A_X + posA, A_R);
			A_Y = quatRotate(A_Y + posA, A_R);
			A_Z = quatRotate(A_Z + posA, A_R);
		}

		if (B_T.x == 0 || B_T.x == 1 || B_T.x == 2 || B_T.x == 3) {
			B_X = B_X + posB;
		} else if (B_T.x == 5) {
			B_X = quatRotate(B_X + posB, B_R);
			B_Y = quatRotate(B_Y + posB, B_R);
			B_Z = quatRotate(B_Z + posB, B_R);
		}

		float3 N, p1, p2;
		float depth = 0;
		if (!CollideAndFindPoint(A_T.x, A_X, A_Y, A_Z, A_R, B_T.x, B_X, B_Y, B_Z, B_R, N, p1, p2, depth)) {
			continue;
		};

		p1 = (TransformSupportVert(A_T.x, A_X, A_Y, A_Z, A_R, -N) - p1) * N * N + p1;
		p2 = (TransformSupportVert(B_T.x, B_X, B_Y, B_Z, B_R, N) - p2) * N * N + p2;
		depth = sqrtf(dot((p2 - p1), (p2 - p1)));
		data_container->host_norm_data.push_back(-N);
		data_container->host_cpta_data.push_back(p1);
		data_container->host_cptb_data.push_back(p2);
		data_container->host_dpth_data.push_back(-depth);
		data_container->host_bids_data.push_back(I2(A_T.z, B_T.z));
		data_container->number_of_contacts++;
	}
	data_container->host_gam_data.resize(data_container->number_of_contacts);
	thrust::fill(data_container->host_gam_data.begin(), data_container->host_gam_data.end(), F3(0));
}
void ChCCollisionGPU::Narrowphase(gpu_container & gpu_data) {
	thrust::device_vector<uint> generic_counter(gpu_data.number_of_contacts_possible, 0xFFFFFFFF);
	uint number_of_contacts_possible=gpu_data.number_of_contacts_possible;
	gpu_data.device_norm_data.resize(gpu_data.number_of_contacts_possible);
	gpu_data.device_cpta_data.resize(gpu_data.number_of_contacts_possible);
	gpu_data.device_cptb_data.resize(gpu_data.number_of_contacts_possible);
	gpu_data.device_dpth_data.resize(gpu_data.number_of_contacts_possible);
	gpu_data.device_bids_data.resize(gpu_data.number_of_contacts_possible);

	MPR_GPU_Store CUDA_KERNEL_DIM(BLOCKS(number_of_contacts_possible),THREADS) (
			CASTF3(gpu_data.device_pos_data),
			CASTF4(gpu_data.device_rot_data),
			CASTF3(gpu_data.device_ObA_data),
			CASTF3(gpu_data.device_ObB_data),
			CASTF3(gpu_data.device_ObC_data),
			CASTF4(gpu_data.device_ObR_data),
			CASTI3(gpu_data.device_typ_data),
			CASTLL(gpu_data.device_pair_data),
			CASTU1(generic_counter),
			CASTF3(gpu_data.device_norm_data),
			CASTF3(gpu_data.device_cpta_data),
			CASTF3(gpu_data.device_cptb_data),
			CASTF1(gpu_data.device_dpth_data),
			CASTI2(gpu_data.device_bids_data),
			number_of_contacts_possible);

	thrust::sort_by_key(generic_counter.begin(), generic_counter.end(), thrust::make_zip_iterator(
					thrust::make_tuple(
							gpu_data.device_norm_data.begin(),
							gpu_data.device_cpta_data.begin(),
							gpu_data.device_cptb_data.begin(),
							gpu_data.device_dpth_data.begin(),
							gpu_data.device_bids_data.begin(),
							gpu_data.device_pair_data.begin())));

	gpu_data.number_of_contacts = number_of_contacts_possible- Thrust_Count(generic_counter,0xFFFFFFFF);

	//thrust::device_vector<float3> old_gamma = data_container->device_gam_data;

	gpu_data.device_norm_data.resize(gpu_data.number_of_contacts);
	gpu_data.device_cpta_data.resize(gpu_data.number_of_contacts);
	gpu_data.device_cptb_data.resize(gpu_data.number_of_contacts);
	gpu_data.device_dpth_data.resize(gpu_data.number_of_contacts);
	gpu_data.device_bids_data.resize(gpu_data.number_of_contacts);
	gpu_data.device_gam_data.resize(gpu_data.number_of_contacts);
	Thrust_Fill(gpu_data.device_gam_data,F3(0));
	//	contact_pair.resize(number_of_contacts);
	//
	//	thrust::sort_by_key(contact_pair.begin(), contact_pair.end(), thrust::make_zip_iterator(thrust::make_tuple(data_container->device_norm_data.begin(), data_container->device_cpta_data.begin(),
	//			data_container->device_cptb_data.begin(), data_container->device_dpth_data.begin(), data_container->device_bids_data.begin())));
	//	if (old_contact_pair.size() != 0) {
	//		thrust::device_vector<int> res(old_contact_pair.size());
	//
	//		thrust::binary_search(contact_pair.begin(), contact_pair.end(), old_contact_pair.begin(), old_contact_pair.end(), res.begin());//list of persistent contacts
	//		thrust::sort_by_key(res.begin(), res.end(), old_contact_pair.begin(),thrust::greater<int>() );//index of common contacts from old list
	//
	//		int numP = Thrust_Count(res,1);
	//		old_contact_pair.resize(numP);
	//		if (numP > 0) {cout<<numP<<"\t";
	//			thrust::device_vector<int> temporaryB(numP);
	//			thrust::lower_bound(contact_pair.begin(), contact_pair.end(), old_contact_pair.begin(), old_contact_pair.end(), temporaryB.begin());//return index of common new contact
	//CopyGamma		<<<BLOCKS(numP),THREADS>>>(
	//				CASTI1(temporaryB),
	//				CASTF3(old_gamma),
	//				CASTF3(data_container->device_gam_data),
	//				numP);
	//		//				for(int i=0; i<old_contact_pair.size(); i++){cout<<old_contact_pair[i]<<endl;}
	//		//				cout<<"------------------------"<<endl;
	//		//				for(int i=0; i<contact_pair.size(); i++){cout<<contact_pair[i]<<endl;}
	//		//				cout<<"------------------------"<<endl;
	//		//				for(int i=0; i<res1.size(); i++){cout<<res1[i]<<endl;}
	//		//				cout<<"------------------------"<<endl;
	//		//				for(int i=0; i<temporaryA.size(); i++){cout<<temporaryA[i]<<endl;}
	//		//				cout<<"------------------------"<<endl;
	//		//				for(int i=0; i<temporaryB.size(); i++){cout<<temporaryB[i]<<endl;}
	//		//
	//		//				exit(0);
	//	}
	//}
	//old_contact_pair = contact_pair;
}
